#include "hip/hip_runtime.h"
#ifndef WRAPPER_H_GUARD
#define WRAPPER_H_GUARD
void eval_surface(float*, float*,
                   float*, float*,
                   float,
                   float, float,
                   float, float,
                   float, float,
                   int, int,
                   float, float,
                   int, int, int, int, float, int, int);
#endif

/* eval surface wrapper (n = 0)
 *
 * wrapper function for the eval_surface device function.
 * THREADS: num_sides
 */
__global__ void eval_surface_wrapper0(float *c, float *left_riemann_rhs, float *right_riemann_rhs, 
                                      float *s_length, 
                                      float *V1x, float *V1y,
                                      float *V2x, float *V2y,
                                      float *V3x, float *V3y,
                                      int *left_elem, int *right_elem,
                                      int *left_side_number, int *right_side_number, 
                                      float *Nx, float *Ny, 
                                      int n_quad1d, int n_p, int num_sides, int num_elem, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        register float rho_left[1], u_left[1], v_left[1], E_left[1];
        register float rho_right[1], u_right[1], v_right[1], E_right[1];
        register float c_left[1], c_right[1];

        // grab the coefficients for the left & right elements
        // TODO: group all the boundary sides together so they are in the same warp;
        //       means no warp divergence
        if (right_elem[idx] != -1) {
            rho_left[0] = c[num_elem * n_p * 0 + left_elem[idx]];
            u_left[0]   = c[num_elem * n_p * 1 + left_elem[idx]];
            v_left[0]   = c[num_elem * n_p * 2 + left_elem[idx]];
            E_left[0]   = c[num_elem * n_p * 3 + left_elem[idx]];
            rho_right[0] = c[num_elem * n_p * 0 + right_elem[idx]];
            u_right[0]   = c[num_elem * n_p * 1 + right_elem[idx]];
            v_right[0]   = c[num_elem * n_p * 2 + right_elem[idx]];
            E_right[0]   = c[num_elem * n_p * 3 + right_elem[idx]];
        } else {
            rho_left[0] = c[num_elem * n_p * 0 + left_elem[idx]];
            u_left[0]   = c[num_elem * n_p * 1 + left_elem[idx]];
            v_left[0]   = c[num_elem * n_p * 2 + left_elem[idx]];
            E_left[0]   = c[num_elem * n_p * 3 + left_elem[idx]];
        }

        __syncthreads();

        eval_surface(rho_left, u_left, v_left, E_left,
                     rho_right, u_right, v_right, E_right,
                     left_riemann_rhs, right_riemann_rhs,
                     s_length[idx],
                     V1x[left_elem[idx]], V1y[left_elem[idx]],
                     V2x[left_elem[idx]], V2y[left_elem[idx]],
                     V3x[left_elem[idx]], V3y[left_elem[idx]],
                     left_elem[idx], right_elem[idx],
                     left_side_number[idx], right_side_number[idx],
                     Nx[idx], Ny[idx],
                     n_quad1d, n_p, num_sides, num_elem, t, idx, alpha);
    }
}


/* eval surface wrapper (n = 1)
 *
 * wrapper function for the eval_surface device function.
 * THREADS: num_sides
 */
__global__ void eval_surface_wrapper1(float *c, float *left_riemann_rhs, float *right_riemann_rhs, 
                                      float *s_length, 
                                      float *V1x, float *V1y,
                                      float *V2x, float *V2y,
                                      float *V3x, float *V3y,
                                      int *left_elem, int *right_elem,
                                      int *left_side_number, int *right_side_number, 
                                      float *Nx, float *Ny, 
                                      int n_quad1d, int n_p, int num_sides, int num_elem, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        register float rho_left[3], u_left[3], v_left[3], E_left[3];
        register float rho_right[3], u_right[3], v_right[3], E_right[3];
        register float r_c_left[3], r_c_right[3];

        // grab the coefficients for the left & right elements
        // TODO: group all the boundary sides together so they are in the same warp;
        //       means no warp divergence
        if (right_elem[idx] != -1) {
            for (i = 0; i < 3; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
                rho_right[i] = c[num_elem * n_p * 0 + i * num_elem + right_elem[idx]];
                u_right[i]   = c[num_elem * n_p * 1 + i * num_elem + right_elem[idx]];
                v_right[i]   = c[num_elem * n_p * 2 + i * num_elem + right_elem[idx]];
                E_right[i]   = c[num_elem * n_p * 3 + i * num_elem + right_elem[idx]];
            }
        } else {
            for (i = 0; i < 3; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
            }
        }

        __syncthreads();

        eval_surface(rho_left, u_left, v_left, E_left,
                     rho_right, u_right, v_right, E_right,
                     left_riemann_rhs, right_riemann_rhs,
                     s_length[idx],
                     V1x[left_elem[idx]], V1y[left_elem[idx]],
                     V2x[left_elem[idx]], V2y[left_elem[idx]],
                     V3x[left_elem[idx]], V3y[left_elem[idx]],
                     left_elem[idx], right_elem[idx],
                     left_side_number[idx], right_side_number[idx],
                     Nx[idx], Ny[idx],
                     n_quad1d, n_p, num_sides, num_elem, t, idx, alpha);
    }
}


/* eval surface wrapper (n = 2)
 *
 * wrapper function for the eval_surface device function.
 * THREADS: num_sides
 */
__global__ void eval_surface_wrapper2(float *c, float *left_riemann_rhs, float *right_riemann_rhs, 
                                      float *s_length, 
                                      float *V1x, float *V1y,
                                      float *V2x, float *V2y,
                                      float *V3x, float *V3y,
                                      int *left_elem, int *right_elem,
                                      int *left_side_number, int *right_side_number, 
                                      float *Nx, float *Ny, 
                                      int n_quad1d, int n_p, int num_sides, int num_elem, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        register float rho_left[6], u_left[6], v_left[6], E_left[6];
        register float rho_right[6], u_right[6], v_right[6], E_right[6];
        register float r_c_left[6], r_c_right[6];

        // grab the coefficients for the left & right elements
        // TODO: group all the boundary sides together so they are in the same warp;
        //       means no warp divergence
        if (right_elem[idx] != -1) {
            for (i = 0; i < 6; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
                rho_right[i] = c[num_elem * n_p * 0 + i * num_elem + right_elem[idx]];
                u_right[i]   = c[num_elem * n_p * 1 + i * num_elem + right_elem[idx]];
                v_right[i]   = c[num_elem * n_p * 2 + i * num_elem + right_elem[idx]];
                E_right[i]   = c[num_elem * n_p * 3 + i * num_elem + right_elem[idx]];
            }
        } else {
            for (i = 0; i < 6; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
            }
        }

        __syncthreads();

        eval_surface(rho_left, u_left, v_left, E_left,
                     rho_right, u_right, v_right, E_right,
                     left_riemann_rhs, right_riemann_rhs,
                     s_length[idx],
                     V1x[left_elem[idx]], V1y[left_elem[idx]],
                     V2x[left_elem[idx]], V2y[left_elem[idx]],
                     V3x[left_elem[idx]], V3y[left_elem[idx]],
                     left_elem[idx], right_elem[idx],
                     left_side_number[idx], right_side_number[idx],
                     Nx[idx], Ny[idx],
                     n_quad1d, n_p, num_sides, num_elem, t, idx, alpha);
    }
}


/* eval surface wrapper (n = 3)
 *
 * wrapper function for the eval_surface device function.
 * THREADS: num_sides
 */
__global__ void eval_surface_wrapper3(float *c, float *left_riemann_rhs, float *right_riemann_rhs, 
                                      float *s_length, 
                                      float *V1x, float *V1y,
                                      float *V2x, float *V2y,
                                      float *V3x, float *V3y,
                                      int *left_elem, int *right_elem,
                                      int *left_side_number, int *right_side_number, 
                                      float *Nx, float *Ny, 
                                      int n_quad1d, int n_p, int num_sides, int num_elem, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        register float rho_left[10], u_left[10], v_left[10], E_left[10];
        register float rho_right[10], u_right[10], v_right[10], E_right[10];
        register float r_c_left[10], r_c_right[10];

        // grab the coefficients for the left & right elements
        // TODO: group all the boundary sides together so they are in the same warp;
        //       means no warp divergence
        if (right_elem[idx] != -1) {
            for (i = 0; i < 10; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
                rho_right[i] = c[num_elem * n_p * 0 + i * num_elem + right_elem[idx]];
                u_right[i]   = c[num_elem * n_p * 1 + i * num_elem + right_elem[idx]];
                v_right[i]   = c[num_elem * n_p * 2 + i * num_elem + right_elem[idx]];
                E_right[i]   = c[num_elem * n_p * 3 + i * num_elem + right_elem[idx]];
            }
        } else {
            for (i = 0; i < 10; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
            }
        }

        __syncthreads();

        eval_surface(rho_left, u_left, v_left, E_left,
                     rho_right, u_right, v_right, E_right,
                     left_riemann_rhs, right_riemann_rhs,
                     s_length[idx],
                     V1x[left_elem[idx]], V1y[left_elem[idx]],
                     V2x[left_elem[idx]], V2y[left_elem[idx]],
                     V3x[left_elem[idx]], V3y[left_elem[idx]],
                     left_elem[idx], right_elem[idx],
                     left_side_number[idx], right_side_number[idx],
                     Nx[idx], Ny[idx],
                     n_quad1d, n_p, num_sides, num_elem, t, idx, alpha);
    }
}


/* eval surface wrapper (n = 4)
 *
 * wrapper function for the eval_surface device function.
 * THREADS: num_sides
 */
__global__ void eval_surface_wrapper4(float *c, float *left_riemann_rhs, float *right_riemann_rhs, 
                                      float *s_length, 
                                      float *V1x, float *V1y,
                                      float *V2x, float *V2y,
                                      float *V3x, float *V3y,
                                      int *left_elem, int *right_elem,
                                      int *left_side_number, int *right_side_number, 
                                      float *Nx, float *Ny, 
                                      int n_quad1d, int n_p, int num_sides, int num_elem, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        register float rho_left[15], u_left[15], v_left[15], E_left[15];
        register float rho_right[15], u_right[15], v_right[15], E_right[15];

        // grab the coefficients for the left & right elements
        // TODO: group all the boundary sides together so they are in the same warp;
        //       means no warp divergence
        if (right_elem[idx] != -1) {
            for (i = 0; i < 15; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
                rho_right[i] = c[num_elem * n_p * 0 + i * num_elem + right_elem[idx]];
                u_right[i]   = c[num_elem * n_p * 1 + i * num_elem + right_elem[idx]];
                v_right[i]   = c[num_elem * n_p * 2 + i * num_elem + right_elem[idx]];
                E_right[i]   = c[num_elem * n_p * 3 + i * num_elem + right_elem[idx]];
            }
        } else {
            for (i = 0; i < 15; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
            }
        }

        __syncthreads();

        eval_surface(rho_left, u_left, v_left, E_left,
                     rho_right, u_right, v_right, E_right,
                     left_riemann_rhs, right_riemann_rhs,
                     s_length[idx],
                     V1x[left_elem[idx]], V1y[left_elem[idx]],
                     V2x[left_elem[idx]], V2y[left_elem[idx]],
                     V3x[left_elem[idx]], V3y[left_elem[idx]],
                     left_elem[idx], right_elem[idx],
                     left_side_number[idx], right_side_number[idx],
                     Nx[idx], Ny[idx],
                     n_quad1d, n_p, num_sides, num_elem, t, idx, alpha);
    }
}


/* eval surface wrapper (n = 5)
 *
 * wrapper function for the eval_surface device function.
 * THREADS: num_sides
 */
__global__ void eval_surface_wrapper5(float *c, float *left_riemann_rhs, float *right_riemann_rhs, 
                                      float *s_length, 
                                      float *V1x, float *V1y,
                                      float *V2x, float *V2y,
                                      float *V3x, float *V3y,
                                      int *left_elem, int *right_elem,
                                      int *left_side_number, int *right_side_number, 
                                      float *Nx, float *Ny, 
                                      int n_quad1d, int n_p, int num_sides, int num_elem, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        register float rho_left[21], u_left[21], v_left[21], E_left[21];
        register float rho_right[21], u_right[21], v_right[21], E_right[21];

        // grab the coefficients for the left & right elements
        // TODO: group all the boundary sides together so they are in the same warp;
        //       means no warp divergence
        if (right_elem[idx] != -1) {
            for (i = 0; i < 21; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
                rho_right[i] = c[num_elem * n_p * 0 + i * num_elem + right_elem[idx]];
                u_right[i]   = c[num_elem * n_p * 1 + i * num_elem + right_elem[idx]];
                v_right[i]   = c[num_elem * n_p * 2 + i * num_elem + right_elem[idx]];
                E_right[i]   = c[num_elem * n_p * 3 + i * num_elem + right_elem[idx]];
            }
        } else {
            for (i = 0; i < 21; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
            }
        }

        __syncthreads();

        eval_surface(rho_left, u_left, v_left, E_left,
                     rho_right, u_right, v_right, E_right,
                     left_riemann_rhs, right_riemann_rhs,
                     s_length[idx],
                     V1x[left_elem[idx]], V1y[left_elem[idx]],
                     V2x[left_elem[idx]], V2y[left_elem[idx]],
                     V3x[left_elem[idx]], V3y[left_elem[idx]],
                     left_elem[idx], right_elem[idx],
                     left_side_number[idx], right_side_number[idx],
                     Nx[idx], Ny[idx],
                     n_quad1d, n_p, num_sides, num_elem, t, idx, alpha);
    }
}


//* eval volume wrapper (n = 0)
//*
//* wrapper function for the eval_volume device function.
//* THREADS: num_sides
 __global__ void eval_volume_wrapper0(float *c, float *quad_rhs, 
                                      float *xr, float *yr,
                                      float *xs, float *ys,
                                      int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float rho[1], u[1], v[1], E[1];

        // get the coefficients for this element
        rho[0] = c[num_elem * n_p * 0 + idx];
        u[0]   = c[num_elem * n_p * 1 + idx];
        v[0]   = c[num_elem * n_p * 2 + idx];
        E[0]   = c[num_elem * n_p * 3 + idx];

        eval_volume(rho, u, v, E, quad_rhs,
                    xr[idx], yr[idx],
                    xs[idx], ys[idx],
                    n_quad, n_p, num_elem, idx);
    }
}

//* eval volume wrapper (n = 1)
//*
//* wrapper function for the eval_volume device function.
//* THREADS: num_sides
 __global__ void eval_volume_wrapper1(float *c, float *quad_rhs, 
                                      float *xr, float *yr,
                                      float *xs, float *ys,
                                      int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float rho[3], u[3], v[3], E[3];

        // get the coefficients for this element
        for (i = 0; i < 3; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_volume(rho, u, v, E, quad_rhs,
                    xr[idx], yr[idx],
                    xs[idx], ys[idx],
                    n_quad, n_p, num_elem, idx);
         
    }
}

//* eval volume wrapper (n = 2)
//*
//* wrapper function for the eval_volume device function.
//* THREADS: num_sides
 __global__ void eval_volume_wrapper2(float *c, float *quad_rhs, 
                                      float *xr, float *yr,
                                      float *xs, float *ys,
                                      int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float rho[6], u[6], v[6], E[6];

        // get the coefficients for this element
        for (i = 0; i < 6; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_volume(rho, u, v, E, quad_rhs,
                    xr[idx], yr[idx],
                    xs[idx], ys[idx],
                    n_quad, n_p, num_elem, idx);
         
    }
}
//* eval volume wrapper (n = 3)
//*
//* wrapper function for the eval_volume device function.
//* THREADS: num_sides
 __global__ void eval_volume_wrapper3(float *c, float *quad_rhs, 
                                      float *xr, float *yr,
                                      float *xs, float *ys,
                                      int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
        
    }
}
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float rho[10], u[10], v[10], E[10];

        // get the coefficients for this element
        for (i = 0; i < 10; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_volume(rho, u, v, E, quad_rhs,
                    xr[idx], yr[idx],
                    xs[idx], ys[idx],
                    n_quad, n_p, num_elem, idx);
         
    }
}
//* eval volume wrapper (n = 4)
//*
//* wrapper function for the eval_volume device function.
//* THREADS: num_sides
 
 __global__ void eval_volume_wrapper4(float *c, float *quad_rhs, 
                                      float *xr, float *yr,
                                      float *xs, float *ys,
                                      int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float rho[15], u[15], v[15], E[15];

        // get the coefficients for this element
        for (i = 0; i < 15; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_volume(rho, u, v, E, quad_rhs,
                    xr[idx], yr[idx],
                    xs[idx], ys[idx],
                    n_quad, n_p, num_elem, idx);
         
    }
}
//* eval volume wrapper (n = 5)
//*
//* wrapper function for the eval_volume device function.
//* THREADS: num_sides
 __global__ void eval_volume_wrapper5(float *c, float *quad_rhs, 
                                      float *xr, float *yr,
                                      float *xs, float *ys,
                                      int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float rho[15], u[15], v[15], E[15];

        // get the coefficients for this element
        for (i = 0; i < 15; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_volume(rho, u, v, E, quad_rhs,
                    xr[idx], yr[idx],
                    xs[idx], ys[idx],
                    n_quad, n_p, num_elem, idx);
         
    }
}
//* eval u wrapper (n = 0)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_u_wrapper0(float *c,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[1];

        // get the coefficients for this element
        r_c[0] = c[idx];

        eval_u(r_c, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 1)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_u_wrapper1(float *c,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[3];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];

        eval_u(r_c, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 2)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_u_wrapper2(float *c,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[6];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];
        r_c[3] = c[3 * num_elem + idx];
        r_c[4] = c[4 * num_elem + idx];
        r_c[5] = c[5 * num_elem + idx];

        eval_u(r_c, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval u wrapper (n = 3)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_u_wrapper3(float *c,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[10];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];
        r_c[3] = c[3 * num_elem + idx];
        r_c[4] = c[4 * num_elem + idx];
        r_c[5] = c[5 * num_elem + idx];
        r_c[6] = c[6 * num_elem + idx];
        r_c[7] = c[7 * num_elem + idx];
        r_c[8] = c[8 * num_elem + idx];
        r_c[9] = c[9 * num_elem + idx];

        eval_u(r_c, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval u wrapper (n = 4)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
 
__global__ void eval_u_wrapper4(float *c, 
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[15];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];
        r_c[3] = c[3 * num_elem + idx];
        r_c[4] = c[4 * num_elem + idx];
        r_c[5] = c[5 * num_elem + idx];
        r_c[6] = c[6 * num_elem + idx];
        r_c[7] = c[7 * num_elem + idx];
        r_c[8] = c[8 * num_elem + idx];
        r_c[9] = c[9 * num_elem + idx];

        r_c[10] = c[10 * num_elem + idx];
        r_c[11] = c[11 * num_elem + idx];
        r_c[12] = c[12 * num_elem + idx];
        r_c[13] = c[13 * num_elem + idx];
        r_c[14] = c[14 * num_elem + idx];

        eval_u(r_c, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 5)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_u_wrapper5(float *c,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[21];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];
        r_c[3] = c[3 * num_elem + idx];
        r_c[4] = c[4 * num_elem + idx];
        r_c[5] = c[5 * num_elem + idx];
        r_c[6] = c[6 * num_elem + idx];
        r_c[7] = c[7 * num_elem + idx];
        r_c[8] = c[8 * num_elem + idx];
        r_c[9] = c[9 * num_elem + idx];

        r_c[10] = c[10 * num_elem + idx];
        r_c[11] = c[11 * num_elem + idx];
        r_c[12] = c[12 * num_elem + idx];
        r_c[13] = c[13 * num_elem + idx];
        r_c[14] = c[14 * num_elem + idx];
        r_c[15] = c[15 * num_elem + idx];
        r_c[16] = c[16 * num_elem + idx];
        r_c[17] = c[17 * num_elem + idx];
        r_c[18] = c[18 * num_elem + idx];
        r_c[19] = c[19 * num_elem + idx];

        r_c[20] = c[20 * num_elem + idx];

        eval_u(r_c, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval u wrapper (n = 6)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_u_wrapper6(float *c,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[28];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];
        r_c[3] = c[3 * num_elem + idx];
        r_c[4] = c[4 * num_elem + idx];
        r_c[5] = c[5 * num_elem + idx];
        r_c[6] = c[6 * num_elem + idx];
        r_c[7] = c[7 * num_elem + idx];
        r_c[8] = c[8 * num_elem + idx];
        r_c[9] = c[9 * num_elem + idx];

        r_c[10] = c[10 * num_elem + idx];
        r_c[11] = c[11 * num_elem + idx];
        r_c[12] = c[12 * num_elem + idx];
        r_c[13] = c[13 * num_elem + idx];
        r_c[14] = c[14 * num_elem + idx];
        r_c[15] = c[15 * num_elem + idx];
        r_c[16] = c[16 * num_elem + idx];
        r_c[17] = c[17 * num_elem + idx];
        r_c[18] = c[18 * num_elem + idx];
        r_c[19] = c[19 * num_elem + idx];

        r_c[20] = c[20 * num_elem + idx];
        r_c[21] = c[21 * num_elem + idx];
        r_c[22] = c[22 * num_elem + idx];
        r_c[23] = c[23 * num_elem + idx];
        r_c[24] = c[24 * num_elem + idx];
        r_c[25] = c[25 * num_elem + idx];
        r_c[26] = c[26 * num_elem + idx];
        r_c[27] = c[27 * num_elem + idx];

        eval_u(r_c, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 7)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_u_wrapper7(float *c,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[36];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];
        r_c[3] = c[3 * num_elem + idx];
        r_c[4] = c[4 * num_elem + idx];
        r_c[5] = c[5 * num_elem + idx];
        r_c[6] = c[6 * num_elem + idx];
        r_c[7] = c[7 * num_elem + idx];
        r_c[8] = c[8 * num_elem + idx];
        r_c[9] = c[9 * num_elem + idx];

        r_c[10] = c[10 * num_elem + idx];
        r_c[11] = c[11 * num_elem + idx];
        r_c[12] = c[12 * num_elem + idx];
        r_c[13] = c[13 * num_elem + idx];
        r_c[14] = c[14 * num_elem + idx];
        r_c[15] = c[15 * num_elem + idx];
        r_c[16] = c[16 * num_elem + idx];
        r_c[17] = c[17 * num_elem + idx];
        r_c[18] = c[18 * num_elem + idx];
        r_c[19] = c[19 * num_elem + idx];

        r_c[20] = c[20 * num_elem + idx];
        r_c[21] = c[21 * num_elem + idx];
        r_c[22] = c[22 * num_elem + idx];
        r_c[23] = c[23 * num_elem + idx];
        r_c[24] = c[24 * num_elem + idx];
        r_c[25] = c[25 * num_elem + idx];
        r_c[26] = c[26 * num_elem + idx];
        r_c[27] = c[27 * num_elem + idx];
        r_c[28] = c[28 * num_elem + idx];
        r_c[29] = c[29 * num_elem + idx];

        r_c[30] = c[30 * num_elem + idx];
        r_c[31] = c[31 * num_elem + idx];
        r_c[32] = c[32 * num_elem + idx];
        r_c[33] = c[33 * num_elem + idx];
        r_c[34] = c[34 * num_elem + idx];
        r_c[35] = c[35 * num_elem + idx];

        eval_u(r_c, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval error wrapper (n = 0)
//*
//* wrapper function for the eval_error device function.
//* THREADS: num_sides
__global__ void eval_error_wrapper0(float *c,
                       float *V1x, float *V1y,
                       float *V2x, float *V2y,
                       float *V3x, float *V3y,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[1];

        // get the coefficients for this element
        r_c[0] = c[idx];

        eval_error(r_c, V1x[idx], V1y[idx], V2x[idx], V2y[idx], V3x[idx], V3y[idx], 
                   Uv1, Uv2, Uv3,
                   num_elem, n_p, t, idx, alpha);
    }
}

//* eval error wrapper (n = 1)
//*
//* wrapper function for the eval_error device function.
//* THREADS: num_sides
__global__ void eval_error_wrapper1(float *c,
                       float *V1x, float *V1y,
                       float *V2x, float *V2y,
                       float *V3x, float *V3y,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[3];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];

        eval_error(r_c, V1x[idx], V1y[idx], V2x[idx], V2y[idx], V3x[idx], V3y[idx], 
                   Uv1, Uv2, Uv3,
                   num_elem, n_p, t, idx, alpha);
    }
}

//* eval error wrapper (n = 2)
//*
//* wrapper function for the eval_error device function.
//* THREADS: num_sides
__global__ void eval_error_wrapper2(float *c,
                       float *V1x, float *V1y,
                       float *V2x, float *V2y,
                       float *V3x, float *V3y,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[6];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];
        r_c[3] = c[3 * num_elem + idx];
        r_c[4] = c[4 * num_elem + idx];
        r_c[5] = c[5 * num_elem + idx];

        eval_error(r_c, V1x[idx], V1y[idx], V2x[idx], V2y[idx], V3x[idx], V3y[idx], 
                   Uv1, Uv2, Uv3,
                   num_elem, n_p, t, idx, alpha);
         
    }
}

//* eval error wrapper (n = 3)
//*
//* wrapper function for the eval_error device function.
//* THREADS: num_sides
__global__ void eval_error_wrapper3(float *c,
                       float *V1x, float *V1y,
                       float *V2x, float *V2y,
                       float *V3x, float *V3y,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[10];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];
        r_c[3] = c[3 * num_elem + idx];
        r_c[4] = c[4 * num_elem + idx];
        r_c[5] = c[5 * num_elem + idx];
        r_c[6] = c[6 * num_elem + idx];
        r_c[7] = c[7 * num_elem + idx];
        r_c[8] = c[8 * num_elem + idx];
        r_c[9] = c[9 * num_elem + idx];

        eval_error(r_c, V1x[idx], V1y[idx], V2x[idx], V2y[idx], V3x[idx], V3y[idx], 
                   Uv1, Uv2, Uv3,
                   num_elem, n_p, t, idx, alpha);
         
    }
}

//* eval error wrapper (n = 4)
//*
//* wrapper function for the eval_error device function.
//* THREADS: num_sides
 
__global__ void eval_error_wrapper4(float *c, 
                       float *V1x, float *V1y,
                       float *V2x, float *V2y,
                       float *V3x, float *V3y,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[15];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];
        r_c[3] = c[3 * num_elem + idx];
        r_c[4] = c[4 * num_elem + idx];
        r_c[5] = c[5 * num_elem + idx];
        r_c[6] = c[6 * num_elem + idx];
        r_c[7] = c[7 * num_elem + idx];
        r_c[8] = c[8 * num_elem + idx];
        r_c[9] = c[9 * num_elem + idx];

        r_c[10] = c[10 * num_elem + idx];
        r_c[11] = c[11 * num_elem + idx];
        r_c[12] = c[12 * num_elem + idx];
        r_c[13] = c[13 * num_elem + idx];
        r_c[14] = c[14 * num_elem + idx];

        eval_error(r_c, V1x[idx], V1y[idx], V2x[idx], V2y[idx], V3x[idx], V3y[idx], 
                   Uv1, Uv2, Uv3,
                   num_elem, n_p, t, idx, alpha);
    }
}

//* eval error wrapper (n = 5)
//*
//* wrapper function for the eval_error device function.
//* THREADS: num_sides
__global__ void eval_error_wrapper5(float *c,
                       float *V1x, float *V1y,
                       float *V2x, float *V2y,
                       float *V3x, float *V3y,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[21];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];
        r_c[3] = c[3 * num_elem + idx];
        r_c[4] = c[4 * num_elem + idx];
        r_c[5] = c[5 * num_elem + idx];
        r_c[6] = c[6 * num_elem + idx];
        r_c[7] = c[7 * num_elem + idx];
        r_c[8] = c[8 * num_elem + idx];
        r_c[9] = c[9 * num_elem + idx];

        r_c[10] = c[10 * num_elem + idx];
        r_c[11] = c[11 * num_elem + idx];
        r_c[12] = c[12 * num_elem + idx];
        r_c[13] = c[13 * num_elem + idx];
        r_c[14] = c[14 * num_elem + idx];
        r_c[15] = c[15 * num_elem + idx];
        r_c[16] = c[16 * num_elem + idx];
        r_c[17] = c[17 * num_elem + idx];
        r_c[18] = c[18 * num_elem + idx];
        r_c[19] = c[19 * num_elem + idx];

        r_c[20] = c[20 * num_elem + idx];

        eval_error(r_c, V1x[idx], V1y[idx], V2x[idx], V2y[idx], V3x[idx], V3y[idx], 
                   Uv1, Uv2, Uv3,
                   num_elem, n_p, t, idx, alpha);
         
    }
}

//* eval error wrapper (n = 6)
//*
//* wrapper function for the eval_error device function.
//* THREADS: num_sides
__global__ void eval_error_wrapper6(float *c,
                       float *V1x, float *V1y,
                       float *V2x, float *V2y,
                       float *V3x, float *V3y,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[28];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];
        r_c[3] = c[3 * num_elem + idx];
        r_c[4] = c[4 * num_elem + idx];
        r_c[5] = c[5 * num_elem + idx];
        r_c[6] = c[6 * num_elem + idx];
        r_c[7] = c[7 * num_elem + idx];
        r_c[8] = c[8 * num_elem + idx];
        r_c[9] = c[9 * num_elem + idx];

        r_c[10] = c[10 * num_elem + idx];
        r_c[11] = c[11 * num_elem + idx];
        r_c[12] = c[12 * num_elem + idx];
        r_c[13] = c[13 * num_elem + idx];
        r_c[14] = c[14 * num_elem + idx];
        r_c[15] = c[15 * num_elem + idx];
        r_c[16] = c[16 * num_elem + idx];
        r_c[17] = c[17 * num_elem + idx];
        r_c[18] = c[18 * num_elem + idx];
        r_c[19] = c[19 * num_elem + idx];

        r_c[20] = c[20 * num_elem + idx];
        r_c[21] = c[21 * num_elem + idx];
        r_c[22] = c[22 * num_elem + idx];
        r_c[23] = c[23 * num_elem + idx];
        r_c[24] = c[24 * num_elem + idx];
        r_c[25] = c[25 * num_elem + idx];
        r_c[26] = c[26 * num_elem + idx];
        r_c[27] = c[27 * num_elem + idx];

        eval_error(r_c, V1x[idx], V1y[idx], V2x[idx], V2y[idx], V3x[idx], V3y[idx], 
                   Uv1, Uv2, Uv3,
                   num_elem, n_p, t, idx, alpha);
    }
}

//* eval error wrapper (n = 7)
//*
//* wrapper function for the eval_error device function.
//* THREADS: num_sides
__global__ void eval_error_wrapper7(float *c,
                       float *V1x, float *V1y,
                       float *V2x, float *V2y,
                       float *V3x, float *V3y,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p, float t, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float r_c[36];

        // get the coefficients for this element
        r_c[0] = c[idx];
        r_c[1] = c[num_elem + idx];
        r_c[2] = c[2 * num_elem + idx];
        r_c[3] = c[3 * num_elem + idx];
        r_c[4] = c[4 * num_elem + idx];
        r_c[5] = c[5 * num_elem + idx];
        r_c[6] = c[6 * num_elem + idx];
        r_c[7] = c[7 * num_elem + idx];
        r_c[8] = c[8 * num_elem + idx];
        r_c[9] = c[9 * num_elem + idx];

        r_c[10] = c[10 * num_elem + idx];
        r_c[11] = c[11 * num_elem + idx];
        r_c[12] = c[12 * num_elem + idx];
        r_c[13] = c[13 * num_elem + idx];
        r_c[14] = c[14 * num_elem + idx];
        r_c[15] = c[15 * num_elem + idx];
        r_c[16] = c[16 * num_elem + idx];
        r_c[17] = c[17 * num_elem + idx];
        r_c[18] = c[18 * num_elem + idx];
        r_c[19] = c[19 * num_elem + idx];

        r_c[20] = c[20 * num_elem + idx];
        r_c[21] = c[21 * num_elem + idx];
        r_c[22] = c[22 * num_elem + idx];
        r_c[23] = c[23 * num_elem + idx];
        r_c[24] = c[24 * num_elem + idx];
        r_c[25] = c[25 * num_elem + idx];
        r_c[26] = c[26 * num_elem + idx];
        r_c[27] = c[27 * num_elem + idx];
        r_c[28] = c[28 * num_elem + idx];
        r_c[29] = c[29 * num_elem + idx];

        r_c[30] = c[30 * num_elem + idx];
        r_c[31] = c[31 * num_elem + idx];
        r_c[32] = c[32 * num_elem + idx];
        r_c[33] = c[33 * num_elem + idx];
        r_c[34] = c[34 * num_elem + idx];
        r_c[35] = c[35 * num_elem + idx];

        eval_error(r_c, V1x[idx], V1y[idx], V2x[idx], V2y[idx], V3x[idx], V3y[idx], 
                   Uv1, Uv2, Uv3,
                   num_elem, n_p, t, idx, alpha);
    }
}
