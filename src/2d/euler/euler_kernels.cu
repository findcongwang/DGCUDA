
#include <hip/hip_runtime.h>
/* 2dadvec_kernels_euler.cu
 *
 * This file contains the kernels for the 2D euler DG method.
 *
 * d_t [   rho   ] + d_x [     rho * u    ] + d_y [    rho * v     ] = 0
 * d_t [ rho * u ] + d_x [ rho * u^2 + p  ] + d_y [   rho * u * v  ] = 0
 * d_t [ rho * v ] + d_x [  rho * u * v   ] + d_y [  rho * v^2 + p ] = 0
 * d_t [    E    ] + d_x [ u * ( E +  p ) ] + d_y [ v * ( E +  p ) ] = 0
 *

 */

#define PI 3.14159
#define GAMMA 1.4
#define MACH 2.5

/***********************
 *
 * DEVICE VARIABLES
 *
 ***********************/
/* These are always prefixed with d_ for "device" */
double *d_c;                 // coefficients for [rho, rho * u, rho * v, E]
double *d_quad_rhs;          // the right hand side containing the quadrature contributions
double *d_left_riemann_rhs;  // the right hand side containing the left riemann contributions
double *d_right_riemann_rhs; // the right hand side containing the right riemann contributions

// TODO: switch to low storage runge-kutta
// runge kutta variables
double *d_kstar;
double *d_k1;
double *d_k2;
double *d_k3;
double *d_k4;

// precomputed basis functions 
// TODO: maybe making these 2^n makes sure the offsets are cached more efficiently? who knows...
// precomputed basis functions ordered like so
//
// [phi_1(r1, s1), phi_1(r2, s2), ... , phi_1(r_nq, s_nq)   ]
// [phi_2(r1, s1), phi_2(r2, s2), ... , phi_2(r_nq, s_nq)   ]
// [   .               .           .            .           ]
// [   .               .           .            .           ]
// [   .               .           .            .           ]
// [phi_np(r1, s1), phi_np(r2, s2), ... , phi_np(r_nq, s_nq)]
//
__device__ __constant__ double basis[2048];
// note: these are multiplied by the weights
__device__ __constant__ double basis_grad_x[2048]; 
__device__ __constant__ double basis_grad_y[2048]; 

// precomputed basis functions evaluated along the sides. ordered
// similarly to basis and basis_grad_{x,y} but with one "matrix" for each side
// starting with side 0. to get to each side, offset with:
//      side_number * n_p * num_quad1d.
__device__ __constant__ double basis_side[1024];
__device__ __constant__ double basis_vertex[256];

// weights for 2d and 1d quadrature rules
__device__ __constant__ double w[32];
__device__ __constant__ double w_oned[16];

__device__ __constant__ double r1[32];
__device__ __constant__ double r2[32];
__device__ __constant__ double r_oned[32];

void set_basis(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis"), value, size * sizeof(double));
}
void set_basis_grad_x(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_grad_x"), value, size * sizeof(double));
}
void set_basis_grad_y(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_grad_y"), value, size * sizeof(double));
}
void set_basis_side(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_side"), value, size * sizeof(double));
}
void set_basis_vertex(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_vertex"), value, size * sizeof(double));
}
void set_w(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("w"), value, size * sizeof(double));
}
void set_w_oned(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("w_oned"), value, size * sizeof(double));
}
void set_r1(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("r1"), value, size * sizeof(double));
}
void set_r2(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("r2"), value, size * sizeof(double));
}
void set_r_oned(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("r_oned"), value, size * sizeof(double));
}

// tells which side (1, 2, or 3) to evaluate this boundary integral over
int *d_left_side_number;
int *d_right_side_number;

double *d_J;         // jacobian determinant 
double *d_reduction; // for the min / maxes in the reductions 
double *d_lambda;    // stores computed lambda values for each element
double *d_s_length;  // length of sides

// the num_elem values of the x and y coordinates for the two vertices defining a side
// TODO: can i delete these after the lengths are precomputed?
//       maybe these should be in texture memory?
double *d_s_V1x;
double *d_s_V1y;
double *d_s_V2x;
double *d_s_V2y;

// the num_elem values of the x and y partials
double *d_xr;
double *d_yr;
double *d_xs;
double *d_ys;

// the K indices of the sides for each element ranged 0->H-1
int *d_elem_s1;
int *d_elem_s2;
int *d_elem_s3;

// vertex x and y coordinates on the mesh which define an element
// TODO: can i delete these after the jacobians are precomputed?
//       maybe these should be in texture memory?
double *d_V1x;
double *d_V1y;
double *d_V2x;
double *d_V2y;
double *d_V3x;
double *d_V3y;

// stores computed values at three vertices
double *d_Uv1;
double *d_Uv2;
double *d_Uv3;

// normal vectors for the sides
double *d_Nx;
double *d_Ny;

// index lists for sides
int *d_left_elem;  // index of left  element for side idx
int *d_right_elem; // index of right element for side idx

/***********************
 *
 * DEVICE FUNCTIONS
 *
 ***********************/
__device__ double pressure(double rho, double u, double v, double E) {
    return (GAMMA - 1.) * (E - (u*u + v*v) / 2. * rho);
}

/* evaluate c
 *
 * evaulates the speed of sound c
 */
__device__ double eval_c(double rho, double u, double v, double E) {
    double p = pressure(rho, u, v, E);

    return sqrtf(GAMMA * p / rho);
}    

/***********************
 *
 * INITIAL CONDITIONS
 *
 ***********************/

/* initial condition function
 *
 * returns the value of the intial condition at point x
 */
__device__ double rho0(double x, double y) {
    double r = x*x + y*y;
    return powf(1 + (GAMMA - 1)/ 2. * MACH * (1 - powf(1. / r, 2)), 1./(GAMMA - 1));
}
__device__ double u0(double x, double y) {
    double r = x*x + y*y;
    return cos(PI/2. * x/1.384) * MACH / r;
}
__device__ double v0(double x, double y) {
    double r = x*x + y*y;
    return -cos(PI/2. * y/1.384) * MACH / r;
}
__device__ double E0(double x, double y) {
    return powf(rho0(x,y),GAMMA) / (GAMMA * (GAMMA - 1)) + (powf(u0(x, y), 2) + powf(v0(x, y), 2)) / 2. * rho0(x, y);
}

/* boundary exact
 *
 * returns the exact boundary conditions
 */
__device__ double boundary_exact_rho(double x, double y, double t) {
    return rho0(x, y);
}
__device__ double boundary_exact_u(double x, double y, double t) {
    return u0(x, y);
}
__device__ double boundary_exact_v(double x, double y, double t) {
    return v0(x, y);
}
__device__ double boundary_exact_E(double x, double y, double t) {
    return E0(x, y);
}

__device__ void reflecting_boundary(double rho_left, double *rho_right,
                                    double u_left,   double *u_right,
                                    double v_left,   double *v_right,
                                    double E_left,   double *E_right,
                                    double nx,       double ny) {
    // set the sides to reflect
    *rho_right = rho_left;
    *E_right   = E_left;

    // make the velocities reflect wrt the normal
    // -2 (V dot N) * N + V
    double dot = u_left * (-ny) + v_left * nx;
    *u_right   = u_left - 2 * dot * (-ny);
    *v_right   = v_left - 2 * dot * nx;
}

__device__ void outflow_boundary(double rho_left, double *rho_right,
                                 double u_left,   double *u_right,
                                 double v_left,   double *v_right,
                                 double E_left,   double *E_right) {
    *rho_right = rho_left;
    *u_right   = u_left;
    *v_right   = v_left;
    *E_right   = E_left;
}

__device__ void inflow_boundary(double *rho_right, double *u_right, double *v_right, double *E_right,
                                double v1x, double v1y, 
                                double v2x, double v2y,
                                double v3x, double v3y,
                                int j,
                                int left_side, int n_quad1d) {

    double r1_eval, r2_eval;
    double x, y;

    // we need the mapping back to the grid space
    switch (left_side) {
        case 0: 
            r1_eval = 0.5 + 0.5 * r_oned[j];
            r2_eval = 0.;
            break;
        case 1: 
            r1_eval = (1. - r_oned[j]) / 2.;
            r2_eval = (1. + r_oned[j]) / 2.;
            break;
        case 2: 
            r1_eval = 0.;
            r2_eval = 0.5 + 0.5 * r_oned[n_quad1d - 1 - j];
            break;
    }

    // x = x2 * r + x3 * s + x1 * (1 - r - s)
    x = v2x * r1_eval + v3x * r2_eval + v1x * (1 - r1_eval - r2_eval);
    y = v2y * r1_eval + v3y * r2_eval + v1y * (1 - r1_eval - r2_eval);
        
    *rho_right = rho0(x, y);
    *u_right   = u0(x, y);
    *v_right   = v0(x, y);
    *E_right   = E0(x, y);
}

/* u exact
 *
 * returns the exact value of u for error measurement.
 */
__device__ double uexact(double x, double y, double t) {
    return u0(x, y);
}

/* initial conditions
 *
 * computes the coefficients for the initial conditions
 * THREADS: num_elem
 */
__global__ void init_conditions(double *c, double *J,
                                double *V1x, double *V1y,
                                double *V2x, double *V2y,
                                double *V3x, double *V3y,
                                int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i, j;
    double x, y, rho, u, v, E;

    if (idx < num_elem) {
        for (i = 0; i < n_p; i++) {
            rho = 0.;
            u   = 0.;
            v   = 0.;
            E   = 0.;
            // perform quadrature
            for (j = 0; j < n_quad; j++) {
                // map from the canonical element to the actual point on the mesh
                // x = x2 * r + x3 * s + x1 * (1 - r - s)
                x = r1[j] * V2x[idx] + r2[j] * V3x[idx] + (1 - r1[j] - r2[j]) * V1x[idx];
                y = r1[j] * V2y[idx] + r2[j] * V3y[idx] + (1 - r1[j] - r2[j]) * V1y[idx];

                // evaluate rho, u, v, E there
                rho += w[j] * rho0(x, y) * basis[i * n_quad + j];
                u   += w[j] * u0(x, y) * rho0(x, y) * basis[i * n_quad + j];
                v   += w[j] * v0(x, y) * rho0(x, y) * basis[i * n_quad + j];
                E   += w[j] * E0(x, y) * basis[i * n_quad + j];
            }

            c[num_elem * n_p * 0 + i * num_elem + idx] = rho;
            c[num_elem * n_p * 1 + i * num_elem + idx] = u; // we actually calculate and store rho * u
            c[num_elem * n_p * 2 + i * num_elem + idx] = v; // we actually calculate and store rho * v
            c[num_elem * n_p * 3 + i * num_elem + idx] = E;
        } 
    }
}

/* min reduction function
 *
 * returns the min value from the global data J and stores in min_J
 * each block computes the min jacobian inside of that block and stores it in the
 * blockIdx.x spot of the shared min_J variable.
 * NOTE: this is fixed for 256 threads.
 */
__global__ void min_reduction(double *D, double *min_D, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;
    int i   = (blockIdx.x * 256 * 2) + threadIdx.x;

    __shared__ double s_min[256];

    if (idx < num_elem) {
        // set all of min to D[idx] initially
        s_min[tid] = D[idx];
        __syncthreads();

        // test a few
        while (i < num_elem) {
            s_min[tid] = (s_min[tid] < D[i]) ? s_min[tid] : D[i];
            s_min[tid] = (s_min[tid] < D[i + 256]) ? s_min[tid] : D[i];
            i += gridDim.x * 256 * 2;
            __syncthreads();
        }

        // first half of the warps
        __syncthreads();
        if (tid < 128) {
            s_min[tid] = (s_min[tid] < s_min[tid + 128]) ? s_min[tid] : s_min[tid + 128];
        }

        // first and second warps
        __syncthreads();
        if (tid < 64) {
            s_min[tid] = (s_min[tid] < s_min[tid + 64]) ? s_min[tid] : s_min[tid + 64];
        }

        // unroll last warp
        __syncthreads();
        if (tid < 32) {
            if (blockDim.x >= 64) {
                s_min[tid] = (s_min[tid] < s_min[tid + 32]) ? s_min[tid] : s_min[tid + 32];
            }
            if (blockDim.x >= 32) {
                s_min[tid] = (s_min[tid] < s_min[tid + 16]) ? s_min[tid] : s_min[tid + 16];
            }
            if (blockDim.x >= 16) {
                s_min[tid] = (s_min[tid] < s_min[tid + 8]) ? s_min[tid] : s_min[tid + 8];
            }
            if (blockDim.x >= 8) {
                s_min[tid] = (s_min[tid] < s_min[tid + 4]) ? s_min[tid] : s_min[tid + 4];
            }
            if (blockDim.x >= 4) {
                s_min[tid] = (s_min[tid] < s_min[tid + 2]) ? s_min[tid] : s_min[tid + 2];
            }
            if (blockDim.x >= 2) {
                s_min[tid] = (s_min[tid] < s_min[tid + 1]) ? s_min[tid] : s_min[tid + 1];
            }
        }

        __syncthreads();
        if (tid == 0) {
            min_D[blockIdx.x] = s_min[0];
        }
    }
}

/* max reduction function
 *
 * returns the max value from the global data D and stores in max
 * each block computes the max jacobian inside of that block and stores it in the
 * blockIdx.x spot of the shared max variable.
 * NOTE: this is fixed for 256 threads.
 */
__global__ void max_reduction(double *D, double *max_D, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;
    int i   = (blockIdx.x * 256 * 2) + threadIdx.x;

    __shared__ double s_max[256];

    if (idx < num_elem) {
        // set all of max to D[idx] initially
        s_max[tid] = D[idx];
        __syncthreads();

        // test a few
        while (i + 256 < num_elem) {
            s_max[tid] = (s_max[tid] > D[i]) ? s_max[tid] : D[i];
            s_max[tid] = (s_max[tid] > D[i + 256]) ? s_max[tid] : D[i];
            i += gridDim.x * 256 * 2;
            __syncthreads();
        }

        // first half of the warps
        __syncthreads();
        if (tid < 128) {
            s_max[tid] = (s_max[tid] > s_max[tid + 128]) ? s_max[tid] : s_max[tid + 128];
        }

        // first and second warps
        __syncthreads();
        if (tid < 64) {
            s_max[tid] = (s_max[tid] > s_max[tid + 64]) ? s_max[tid] : s_max[tid + 64];
        }

        // unroll last warp
        __syncthreads();
        if (tid < 32) {
            if (blockDim.x >= 64) {
                s_max[tid] = (s_max[tid] > s_max[tid + 32]) ? s_max[tid] : s_max[tid + 32];
            }
            if (blockDim.x >= 32) {
                s_max[tid] = (s_max[tid] > s_max[tid + 16]) ? s_max[tid] : s_max[tid + 16];
            }
            if (blockDim.x >= 16) {
                s_max[tid] = (s_max[tid] > s_max[tid + 8]) ? s_max[tid] : s_max[tid + 8];
            }
            if (blockDim.x >= 8) {
                s_max[tid] = (s_max[tid] > s_max[tid + 4]) ? s_max[tid] : s_max[tid + 4];
            }
            if (blockDim.x >= 4) {
                s_max[tid] = (s_max[tid] > s_max[tid + 2]) ? s_max[tid] : s_max[tid + 2];
            }
            if (blockDim.x >= 2) {
                s_max[tid] = (s_max[tid] > s_max[tid + 1]) ? s_max[tid] : s_max[tid + 1];
            }
        }

        __syncthreads();
        if (tid == 0) {
            max_D[blockIdx.x] = s_max[0];
        }
    }
}

/***********************
 *
 * PRECOMPUTING
 *
 ***********************/

/* side length computer
 *
 * precomputes the length of each side.
 * THREADS: num_sides
 */ 
__global__ void preval_side_length(double *s_length, 
                              double *s_V1x, double *s_V1y, 
                              double *s_V2x, double *s_V2y,
                              int num_sides) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        // compute and store the length of the side
        s_length[idx] = sqrtf(powf(s_V1x[idx] - s_V2x[idx],2) + powf(s_V1y[idx] - s_V2y[idx],2));
    }
}

/* inscribed circle radius computing
 *
 * computes the radius of each inscribed circle. stores in d_J to find the minumum,
 * then we reuse d_J.
 */
__global__ void preval_inscribed_circles(double *J,
                                    double *V1x, double *V1y,
                                    double *V2x, double *V2y,
                                    double *V3x, double *V3y,
                                    int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double a, b, c, k;
        a = sqrtf(powf(V1x[idx] - V2x[idx], 2) + powf(V1y[idx] - V2y[idx], 2));
        b = sqrtf(powf(V2x[idx] - V3x[idx], 2) + powf(V2y[idx] - V3y[idx], 2));
        c = sqrtf(powf(V1x[idx] - V3x[idx], 2) + powf(V1y[idx] - V3y[idx], 2));

        k = 0.5 * (a + b + c);

        // for the diameter, we multiply by 2
        J[idx] = 2 * sqrtf(k * (k - a) * (k - b) * (k - c)) / k;
    }
}

/* jacobian computing
 *
 * precomputes the jacobian determinant for each element.
 * THREADS: num_elem
 */
__global__ void preval_jacobian(double *J, 
                           double *V1x, double *V1y, 
                           double *V2x, double *V2y, 
                           double *V3x, double *V3y,
                           int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double x1, y1, x2, y2, x3, y3;

        // read vertex points
        x1 = V1x[idx];
        y1 = V1y[idx];
        x2 = V2x[idx];
        y2 = V2y[idx];
        x3 = V3x[idx];
        y3 = V3y[idx];

        // calculate jacobian determinant
        // x = x2 * r + x3 * s + x1 * (1 - r - s)
        J[idx] = (x2 - x1) * (y3 - y1) - (x3 - x1) * (y2 - y1);
    }
}

/* evaluate normal vectors
 *
 * computes the normal vectors for each element along each side.
 * THREADS: num_sides
 *
 */
__global__ void preval_normals(double *Nx, double *Ny, 
                          double *s_V1x, double *s_V1y, 
                          double *s_V2x, double *s_V2y,
                          double *V1x, double *V1y, 
                          double *V2x, double *V2y, 
                          double *V3x, double *V3y,
                          int *left_side_number, int num_sides) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        double x, y, length;
        double sv1x, sv1y, sv2x, sv2y;
    
        sv1x = s_V1x[idx];
        sv1y = s_V1y[idx];
        sv2x = s_V2x[idx];
        sv2y = s_V2y[idx];
    
        // lengths of the vector components
        x = sv2x - sv1x;
        y = sv2y - sv1y;
    
        // normalize
        length = sqrtf(powf(x, 2) + powf(y, 2));

        // store the result
        Nx[idx] = -y / length;
        Ny[idx] =  x / length;
    }
}

__global__ void preval_normals_direction(double *Nx, double *Ny, 
                          double *V1x, double *V1y, 
                          double *V2x, double *V2y, 
                          double *V3x, double *V3y,
                          int *left_elem, int *left_side_number, int num_sides) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        double new_x, new_y, dot;
        double initial_x, initial_y, target_x, target_y;
        double x, y;
        int left_idx, side;

        // get left side's vertices
        left_idx = left_elem[idx];
        side     = left_side_number[idx];

        // get the normal vector
        x = Nx[idx];
        y = Ny[idx];
    
        // make it point the correct direction by learning the third vertex point
        switch (side) {
            case 0: 
                target_x = V3x[left_idx];
                target_y = V3y[left_idx];
                initial_x = (V1x[left_idx] + V2x[left_idx]) / 2.;
                initial_y = (V1y[left_idx] + V2y[left_idx]) / 2.;
                break;
            case 1:
                target_x = V1x[left_idx];
                target_y = V1y[left_idx];
                initial_x = (V2x[left_idx] + V3x[left_idx]) / 2.;
                initial_y = (V2y[left_idx] + V3y[left_idx]) / 2.;
                break;
            case 2:
                target_x = V2x[left_idx];
                target_y = V2y[left_idx];
                initial_x = (V1x[left_idx] + V3x[left_idx]) / 2.;
                initial_y = (V1y[left_idx] + V3y[left_idx]) / 2.;
                break;
        }

        // create the vector pointing towards the third vertex point
        new_x = target_x - initial_x;
        new_y = target_y - initial_y;

        // find the dot product between the normal and new vectors
        dot = x * new_x + y * new_y;
        
        if (dot > 0) {
            Nx[idx] *= -1;
            Ny[idx] *= -1;
        }
    }
}

__global__ void preval_partials(double *V1x, double *V1y,
                                double *V2x, double *V2y,
                                double *V3x, double *V3y,
                                double *xr,  double *yr,
                                double *xs,  double *ys, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num_elem) {
        // evaulate the jacobians of the mappings for the chain rule
        // x = x2 * r + x3 * s + x1 * (1 - r - s)
        xr[idx] = V2x[idx] - V1x[idx];
        yr[idx] = V2y[idx] - V1y[idx];
        xs[idx] = V3x[idx] - V1x[idx];
        ys[idx] = V3y[idx] - V1y[idx];
    }
}

/***********************
 *
 * MAIN FUNCTIONS
 *
 ***********************/

/* limiter
 *
 * the standard limiter for coefficient values
 */
__global__ void limit_c(double *c_inner, 
                   double *c_s1, double *c_s2, double *c_s3,
                   int n_p, int num_elem) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // get cell averages
    //avg_inner = c_inner[0];
    //avg_s1 = c_s1[0];
    //avg_s2 = c_s2[0];
    //avg_s3 = c_s3[0];

    // determine if this is a "troubled" cell

    //for (i = n_p; i > 1; i++) {
        //c_prev = c[i - 1];
    //}
}

/* global lambda evaluation
 *
 * computes the max value of |u + c|, |u|, |u - c|.
 */
__device__ void eval_global_lambda(double *c_rho, double *c_u, double *c_v, double *c_E, double *lambda,
                            int n_quad, int n_p, int idx) {
    int i, j;
    double rho, u, v, E, c;
    double sum;

    // get cell averages
    rho = c_rho[0];
    u   = c_u[0];
    v   = c_v[0];
    E   = c_E[0];

    u = u / rho;
    v = v / rho;

    // evaluate c
    c = eval_c(rho, u, v, E);

    // norm
    sum = sqrtf(u*u + v*v);

    if (sum > 0) {
        lambda[idx] = sum + c;
    } else {
        lambda[idx] = -sum + c;
    }
}

/* riemann evaluation
 *
 * device function to solve the riemann problem.
 */
__device__ void eval_left_right(double *c_rho_left, double *c_rho_right,
                             double *c_u_left,   double *c_u_right,
                             double *c_v_left,   double *c_v_right,
                             double *c_E_left,   double *c_E_right,
                             double *rho_left, double *u_left, double *v_left, double *E_left,
                             double *rho_right, double *u_right, double *v_right, double *E_right,
                             double nx, double ny,
                             double v1x, double v1y,
                             double v2x, double v2y,
                             double v3x, double v3y,
                             int j, // j, as usual, is the index of the integration point
                             int left_side, int right_side,
                             int left_idx, int right_idx,
                             int n_p, int n_quad1d,
                             int num_sides, double t) { 

    int i;

    // evaluate rho, u, v, E at the integration points
    *rho_left  = 0.;
    *u_left    = 0.;
    *v_left    = 0.;
    *E_left    = 0.;
    *rho_right = 0.;
    *u_right   = 0.;
    *v_right   = 0.;
    *E_right   = 0.;
    
    for (i = 0; i < n_p; i++) {
        *rho_left += c_rho_left[i] * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
        *u_left   += c_u_left[i]   * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
        *v_left   += c_v_left[i]   * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
        *E_left   += c_E_left[i]   * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
    }

    // unphysical rho
    if (*rho_left <= 0) {
        *rho_left = c_rho_left[0];
    }

    // since we actually have coefficients for rho * u and rho * v
    *u_left = *u_left / *rho_left;
    *v_left = *v_left / *rho_left;

    // TODO: make all threads in the first warps be boundary sides
    ///////////////////////
    // reflecting 
    ///////////////////////
    if (right_idx == -1) {
        reflecting_boundary(*rho_left, rho_right, 
                            *u_left,   u_right, 
                            *v_left,   v_right, 
                            *E_left,   E_right,
                            nx, ny);

    ///////////////////////
    // outflow 
    ///////////////////////
    } else if (right_idx == -2) {
        outflow_boundary(*rho_left, rho_right,
                         *u_left,   u_right,
                         *v_left,   v_right,
                         *E_left,   E_right);

    ///////////////////////
    // inflow 
    ///////////////////////
    } else if (right_idx == -3) {
        inflow_boundary(rho_right, u_right, v_right, E_right,
                        v1x, v1y, v2x, v2y, v3x, v3y, 
                        j, 
                        left_side, n_quad1d);
    ///////////////////////
    // not a boundary
    ///////////////////////
    } else {
        // evaluate the right side at the integration point
        for (i = 0; i < n_p; i++) {
            *rho_right += c_rho_right[i] * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
            *u_right   += c_u_right[i]   * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
            *v_right   += c_v_right[i]   * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
            *E_right   += c_E_right[i]   * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
        }

        // unphysical rho
        if (*rho_right <= 0) {
            *rho_right = c_rho_right[0];
        }

        // again, since we have coefficients for rho * u and rho * v
        *u_right = *u_right / *rho_right;
        *v_right = *v_right / *rho_right;
    }
}

/* surface integral evaluation
 *
 * evaluate all the riemann problems for each element.
 * THREADS: num_sides
 */
/*
 * d_t [   rho   ] + d_x [     rho * u    ] + d_y [    rho * v     ] = 0
 * d_t [ rho * u ] + d_x [ rho * u^2 + p  ] + d_y [   rho * u * v  ] = 0
 * d_t [ rho * v ] + d_x [  rho * u * v   ] + d_y [  rho * v^2 + p ] = 0
 * d_t [    E    ] + d_x [ u * ( E +  p ) ] + d_y [ v * ( E +  p ) ] = 0
 */

/* evaluate lambda
 *
 * finds the max absolute value of the jacobian for F(u).
 *  |u - c|, |u|, |u + c|
 */
__device__ double eval_lambda(double rho_left, double rho_right,
                              double u_left,   double u_right,
                              double v_left,   double v_right,
                              double E_left,   double E_right,
                              double nx,       double ny) {
                              
    double s_left, s_right;
    double left_max, right_max;
    double c_left, c_right;
    
    c_left  = eval_c(rho_left, u_left, v_left, E_left);
    c_right = eval_c(rho_right, u_right, v_right, E_right);

    s_left  = nx * u_left  + ny * v_left;
    s_right = nx * u_right + ny * v_right; 
    
    if (s_left > 0.) {
        left_max = s_left + c_left;
    } else {
        left_max = -s_left + c_left;
    }

    if (s_right > 0.) {
        right_max = s_right + c_right;
    } else {
        right_max = -s_right + c_right;
    }

    return (abs(left_max) > abs(right_max)) ? abs(left_max) : abs(right_max);

    ////////////////
    // left element 
    ////////////////

    /*
    // evaluate u - c
    sum1_l = 0.;
    for (j = 0; j < n_quad; j++) {
        // evaluate rho,u,v,E at the integration point
        rho = 0.;
        u   = 0.;
        v   = 0.;
        E   = 0.;
        for (i = 0; i < n_p; i++) {
            rho += c_rho_left[i] * basis[n_quad * i + j];
            u   += c_u_left[i]   * basis[n_quad * i + j];
            v   += c_v_left[i]   * basis[n_quad * i + j];
            E   += c_E_left[i]   * basis[n_quad * i + j];
        }
        u = u / rho;
        v = v / rho;
        // evaluate c at the integration point
        c = eval_c(rho, u, v, E);

        sum1_l += w[j] * (sqrtf(u*u + v*v) - c);
    }
    sum1_l = abs(sum1_l);

    // evaluate u
    sum2_l = 0.;
    for (j = 0; j < n_quad; j++) {
        // evaluate u at the integration point
        rho = 0.;
        u   = 0.;
        v   = 0.;
        for (i = 0; i < n_p; i++) {
            rho += c_rho_left[i] * basis[n_quad * i + j];
            u   += c_u_left[i]   * basis[n_quad * i + j];
            v   += c_v_left[i]   * basis[n_quad * i + j];
        }
        u = u / rho;
        v = v / rho;

        sum2_l += w[j] * sqrtf(u*u + v*v);
    }

    sum2_l = abs(sum2_l);

    // evaluate u + c
    sum3_l = 0;
    for (j = 0; j < n_quad; j++) {
        // evaluate rho,u,v,E at the integration point
        rho = 0.;
        u   = 0.;
        v   = 0.;
        E   = 0.;
        for (i = 0; i < n_p; i++) {
            rho += c_rho_left[i] * basis[n_quad * i + j];
            u   += c_u_left[i]   * basis[n_quad * i + j];
            v   += c_v_left[i]   * basis[n_quad * i + j];
            E   += c_E_left[i]   * basis[n_quad * i + j];
        }
        u = u / rho;
        v = v / rho;
        // evaluate c at the integration point
        c = eval_c(rho, u, v, E);

        sum3_l += w[j] * (sqrtf(u*u + v*v) + c);
    }
    sum3_l = abs(sum3_l);

    ////////////////
    // right element
    ////////////////
    // TODO: big bug here. c_*_right may not be defined if we're on a boundary element.

    if (right_idx != -1) {
        // evaluate u - c
        sum1_r = 0;
        for (j = 0; j < n_quad; j++) {
            // evaluate rho,u,v,E at the integration point
            rho = 0.;
            u   = 0.;
            v   = 0.;
            E   = 0.;
            for (i = 0; i < n_p; i++) {
                rho += c_rho_right[i] * basis[n_quad * i + j];
                u   += c_u_right[i]   * basis[n_quad * i + j];
                v   += c_v_right[i]   * basis[n_quad * i + j];
                E   += c_E_right[i]   * basis[n_quad * i + j];
            }
            u = u / rho;
            v = v / rho;
            // evaluate c at the integration point
            c = eval_c(rho, u, v, E);

            sum1_r += w[j] * (sqrtf(u*u + v*v) - c);
        }
        sum1_r = abs(sum1_r);

        // evaluate u
        sum2_r = 0;
        for (j = 0; j < n_quad; j++) {
            // evaluate u at the integration point
            rho = 0.;
            u   = 0.;
            v   = 0.;
            for (i = 0; i < n_p; i++) {
                rho += c_rho_right[i] * basis[n_quad * i + j];
                u   += c_u_right[i]   * basis[n_quad * i + j];
                v   += c_v_right[i]   * basis[n_quad * i + j];
            }
            u = u / rho;
            v = v / rho;

            sum2_r += w[j] * sqrtf(u*u + v*v);
        }

        sum2_r = abs(sum2_r);

        // evaluate u + c
        sum3_r = 0;
        for (j = 0; j < n_quad; j++) {
            // evaluate rho,u,v,E at the integration point
            rho = 0.;
            u   = 0.;
            v   = 0.;
            E   = 0.;
            for (i = 0; i < n_p; i++) {
                rho += c_rho_right[i] * basis[n_quad * i + j];
                u   += c_u_right[i]   * basis[n_quad * i + j];
                v   += c_v_right[i]   * basis[n_quad * i + j];
                E   += c_E_right[i]   * basis[n_quad * i + j];
            }
            u = u / rho;
            v = v / rho;
            // evaluate c at the integration point
            c = eval_c(rho, u, v, E);

            sum3_r += w[j] * (sqrtf(u*u + v*v) + c);
        }
        sum3_r = abs(sum3_r);
    }

    max = 0;
    if (sum1_l > max) {
        max = sum1_l;
    }
    if (sum2_l > max) {
        max = sum2_l;
    }
    if (sum3_l > max) {
        max = sum3_l;
    }

    if (right_idx != -1) {
        if (sum1_r > max) {
            max = sum1_r;
        }
        if (sum2_r > max) {
            max = sum2_r;
        }
        if (sum3_r > max) {
            max = sum3_r;
        }
    }

    return max;
    */
}

/* evaluate flux
 *
 * takes the actual values of rho, u, v, and E and returns the flux 
 * x and y components. 
 * NOTE: this needs the ACTUAL values for u and v, NOT rho * u, rho * v.
 */
__device__ void eval_flux(double rho, double u, double v, double E, 
                     double *flux_x1, double *flux_y1,
                     double *flux_x2, double *flux_y2,
                     double *flux_x3, double *flux_y3,
                     double *flux_x4, double *flux_y4) {

    // evaluate pressure
    double p = pressure(rho, u, v, E);

    // flux_1 
    *flux_x1 = rho * u;
    *flux_y1 = rho * v;

    // flux_2
    *flux_x2 = rho * u * u + p;
    *flux_y2 = rho * u * v;

    // flux_3
    *flux_x3 = rho * u * v;
    *flux_y3 = rho * v * v + p;

    // flux_4
    *flux_x4 = u * (E + p);
    *flux_y4 = v * (E + p);
}

__device__ void eval_surface(double *c_rho_left, double *c_u_left, double *c_v_left, double *c_E_left,
                             double *c_rho_right, double *c_u_right, double *c_v_right, double *c_E_right,
                             double *left_riemann_rhs, double *right_riemann_rhs, 
                             double len, double J,
                             double v1x, double v1y,
                             double v2x, double v2y,
                             double v3x, double v3y,
                             int left_idx,  int right_idx,
                             int left_side, int right_side, 
                             double nx, double ny, 
                             int n_quad1d, int n_quad, int n_p, int num_sides, 
                             int num_elem, double t, int idx) {
    int i, j;
    double s;
    double lambda;
    double left_sum1, right_sum1;
    double left_sum2, right_sum2;
    double left_sum3, right_sum3;
    double left_sum4, right_sum4;
    double flux_x1_l, flux_x2_l, flux_x3_l, flux_x4_l;
    double flux_x1_r, flux_x2_r, flux_x3_r, flux_x4_r;
    double flux_y1_l, flux_y2_l, flux_y3_l, flux_y4_l;
    double flux_y1_r, flux_y2_r, flux_y3_r, flux_y4_r;
    double rho_left, u_left, v_left, E_left;
    double rho_right, u_right, v_right, E_right;

    // multiply across by the i'th basis function
    for (i = 0; i < n_p; i++) {

        left_sum1  = 0.;
        left_sum2  = 0.;
        left_sum3  = 0.;
        left_sum4  = 0.;
        right_sum1 = 0.;
        right_sum2 = 0.;
        right_sum3 = 0.;
        right_sum4 = 0.;

        for (j = 0; j < n_quad1d; j++) {
            // calculate the left and right values along the surface
            eval_left_right(c_rho_left, c_rho_right,
                            c_u_left,   c_u_right,
                            c_v_left,   c_v_right,
                            c_E_left,   c_E_right,
                            &rho_left,  &u_left,  &v_left,  &E_left,
                            &rho_right, &u_right, &v_right, &E_right,
                            nx, ny,
                            v1x, v1y, v2x, v2y, v3x, v3y,
                            j, left_side, right_side,
                            left_idx, right_idx,
                            n_p, n_quad1d, num_sides, t);

            // calculate the left fluxes
            eval_flux(rho_left, u_left, v_left, E_left,
                      &flux_x1_l, &flux_y1_l, &flux_x2_l, &flux_y2_l,
                      &flux_x3_l, &flux_y3_l, &flux_x4_l, &flux_y4_l);

            // calculate the right fluxes
            eval_flux(rho_right, u_right, v_right, E_right,
                      &flux_x1_r, &flux_y1_r, &flux_x2_r, &flux_y2_r,
                      &flux_x3_r, &flux_y3_r, &flux_x4_r, &flux_y4_r);

            // need these local max values
            lambda = eval_lambda(rho_left, rho_right,
                                 u_left, u_right, 
                                 v_left, v_right, 
                                 E_left, E_right,
                                 nx, ny);

            // 1st equation
            s = 0.5 * ((flux_x1_l + flux_x1_r) * nx + (flux_y1_l + flux_y1_r) * ny 
                        + lambda * (rho_left - rho_right));
            left_sum1  += w_oned[j] * s * basis_side[left_side  * n_p * n_quad1d + i * n_quad1d + j];
            right_sum1 += w_oned[j] * s * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];

            // 2nd equation
            s = 0.5 * ((flux_x2_l + flux_x2_r) * nx + (flux_y2_l + flux_y2_r) * ny 
                        + lambda * (u_left - u_right));
            left_sum2  += w_oned[j] * s * basis_side[left_side  * n_p * n_quad1d + i * n_quad1d + j];
            right_sum2 += w_oned[j] * s * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];

            // 3rd equation
            s = 0.5 * ((flux_x3_l + flux_x3_r) * nx + (flux_y3_l + flux_y3_r) * ny 
                        + lambda * (v_left - v_right));
            left_sum3  += w_oned[j] * s * basis_side[left_side  * n_p * n_quad1d + i * n_quad1d + j];
            right_sum3 += w_oned[j] * s * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];

            // 4th equation
            s = 0.5 * ((flux_x4_l + flux_x4_r) * nx + (flux_y4_l + flux_y4_r) * ny 
                        + lambda * (E_left - E_right));
            left_sum4  += w_oned[j] * s * basis_side[left_side  * n_p * n_quad1d + i * n_quad1d + j];
            right_sum4 += w_oned[j] * s * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
        }

        // store this side's contribution in the riemann rhs vectors
        left_riemann_rhs[num_sides * n_p * 0 + i * num_sides + idx]  = -len / 2. * left_sum1;
        left_riemann_rhs[num_sides * n_p * 1 + i * num_sides + idx]  = -len / 2. * left_sum2;
        left_riemann_rhs[num_sides * n_p * 2 + i * num_sides + idx]  = -len / 2. * left_sum3;
        left_riemann_rhs[num_sides * n_p * 3 + i * num_sides + idx]  = -len / 2. * left_sum4;
        right_riemann_rhs[num_sides * n_p * 0 + i * num_sides + idx] =  len / 2. * right_sum1;
        right_riemann_rhs[num_sides * n_p * 1 + i * num_sides + idx] =  len / 2. * right_sum2;
        right_riemann_rhs[num_sides * n_p * 2 + i * num_sides + idx] =  len / 2. * right_sum3;
        right_riemann_rhs[num_sides * n_p * 3 + i * num_sides + idx] =  len / 2. * right_sum4;
    }
}

/* volume integrals
 *
 * evaluates and adds the volume integral to the rhs vector
 * THREADS: num_elem
 */
__device__ void eval_volume(double *c_rho, double *c_u, double *c_v,   double *c_E,
                            double *quad_rhs, 
                            double x_r, double y_r, double x_s, double y_s,
                            int n_quad, int n_p, int num_elem, int idx) {
    int i, j, k;
    double rho, u, v, E;
    double flux_x1, flux_y1, flux_x2, flux_y2;
    double flux_x3, flux_y3, flux_x4, flux_y4;
    double sum1, sum2, sum3, sum4;

    // evaluate the volume integral for each coefficient
    for (i = 0; i < n_p; i++) {
        sum1 = 0.;
        sum2 = 0.;
        sum3 = 0.;
        sum4 = 0.;
        for (j = 0; j < n_quad; j++) {
            // evaluate rho, u, v, E at the integration point.
            rho = 0.;
            u   = 0.;
            v   = 0.;
            E   = 0.;
            for (k = 0; k < n_p; k++) {
                rho += c_rho[k] * basis[n_quad * k + j];
                u   += c_u[k]   * basis[n_quad * k + j];
                v   += c_v[k]   * basis[n_quad * k + j];
                E   += c_E[k]   * basis[n_quad * k + j];
            }

            // unphysical rho
            if (rho <= 0) {
                rho = c_rho[0];
            }

            // since we actually have coefficients for rho * u, rho * v
            u = u / rho;
            v = v / rho;

            // evaluate flux
            eval_flux(rho, u, v, E,
                 &flux_x1, &flux_y1, &flux_x2, &flux_y2,
                 &flux_x3, &flux_y3, &flux_x4, &flux_y4);
                 
            // Add to the sum
            // [fx fy] * [y_s, -y_r; -x_s, x_r] * [phi_x phi_y]

            // 1st equation
            sum1 +=   flux_x1 * ( basis_grad_x[n_quad * i + j] * y_s
                                 -basis_grad_y[n_quad * i + j] * y_r)
                    + flux_y1 * (-basis_grad_x[n_quad * i + j] * x_s 
                                + basis_grad_y[n_quad * i + j] * x_r);

            // 2nd equation
            sum2 +=   flux_x2 * ( basis_grad_x[n_quad * i + j] * y_s
                                 -basis_grad_y[n_quad * i + j] * y_r)
                    + flux_y2 * (-basis_grad_x[n_quad * i + j] * x_s 
                                + basis_grad_y[n_quad * i + j] * x_r);

            // 3rd equation
            sum3 +=   flux_x3 * ( basis_grad_x[n_quad * i + j] * y_s
                                 -basis_grad_y[n_quad * i + j] * y_r)
                    + flux_y3 * (-basis_grad_x[n_quad * i + j] * x_s 
                                + basis_grad_y[n_quad * i + j] * x_r);

            // 4th equation
            sum4 +=   flux_x4 * ( basis_grad_x[n_quad * i + j] * y_s
                                 -basis_grad_y[n_quad * i + j] * y_r)
                    + flux_y4 * (-basis_grad_x[n_quad * i + j] * x_s 
                                + basis_grad_y[n_quad * i + j] * x_r);
        }

        // store the result
        quad_rhs[num_elem * n_p * 0 + i * num_elem + idx] = sum1;
        quad_rhs[num_elem * n_p * 1 + i * num_elem + idx] = sum2;
        quad_rhs[num_elem * n_p * 2 + i * num_elem + idx] = sum3;
        quad_rhs[num_elem * n_p * 3 + i * num_elem + idx] = sum4;
    }
}

/* evaluate u
 * 
 * evaluates rho and E at the three vertex points for output
 * THREADS: num_elem
 */
__device__ void eval_u(double *c, 
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p, int idx) {
    int i;
    double uv1, uv2, uv3;

    // calculate values at the integration points
    uv1 = 0.;
    uv2 = 0.;
    uv3 = 0.;
    for (i = 0; i < n_p; i++) {
        uv1 += c[i] * basis_vertex[i * 3 + 0];
        uv2 += c[i] * basis_vertex[i * 3 + 1];
        uv3 += c[i] * basis_vertex[i * 3 + 2];
    }

    // store result
    Uv1[idx] = uv1;
    Uv2[idx] = uv2;
    Uv3[idx] = uv3;
}

/* evaluate u velocity
 * 
 * evaluates u and v at the three vertex points for output
 * THREADS: num_elem
 */
__device__ void eval_u_velocity(double *c, double *c_rho,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p, int idx) {
    int i;
    double uv1, uv2, uv3;
    double rhov1, rhov2, rhov3;

    // calculate values at the integration points
    rhov1 = 0.;
    rhov2 = 0.;
    rhov3 = 0.;
    for (i = 0; i < n_p; i++) {
        rhov1 += c_rho[i] * basis_vertex[i * 3 + 0];
        rhov2 += c_rho[i] * basis_vertex[i * 3 + 1];
        rhov3 += c_rho[i] * basis_vertex[i * 3 + 2];
    }

    uv1 = 0.;
    uv2 = 0.;
    uv3 = 0.;
    for (i = 0; i < n_p; i++) {
        uv1 += c[i] * basis_vertex[i * 3 + 0];
        uv2 += c[i] * basis_vertex[i * 3 + 1];
        uv3 += c[i] * basis_vertex[i * 3 + 2];
    }

    uv1 = uv1 / rhov1;
    uv2 = uv2 / rhov2;
    uv3 = uv3 / rhov3;

    // store result
    Uv1[idx] = uv1;
    Uv2[idx] = uv2;
    Uv3[idx] = uv3;
}
