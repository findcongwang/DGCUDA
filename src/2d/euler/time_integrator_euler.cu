#include "hip/hip_runtime.h"
/* time_integrator.cu
 *
 * time integration functions.
 */
#ifndef TIMEINTEGRATOR_H_GUARD
#define TIMEINTEGRATOR_H_GUARD
void checkCudaError(const char*);
#endif

/***********************
 * RK4 
 ***********************/

/* tempstorage for RK4
 * 
 * I need to store u + alpha * k_i into some temporary variable called k*.
 */
__global__ void rk4_tempstorage(double *c, double *kstar, double*k, double alpha, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < n_p * num_elem) {
        kstar[idx] = c[idx] + alpha * k[idx];
    }
}

/* rk4
 *
 * computes the runge-kutta solution 
 * u_n+1 = u_n + k1/6 + k2/3 + k3/3 + k4/6
 */
__global__ void rk4(double *c, double *k1, double *k2, double *k3, double *k4, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < n_p * num_elem) {
        c[num_elem * n_p * 0 + idx] += k1[idx]/6. + k2[idx]/3. + k3[idx]/3. + k4[idx]/6.;
        c[num_elem * n_p * 1 + idx] += k1[idx]/6. + k2[idx]/3. + k3[idx]/3. + k4[idx]/6.;
        c[num_elem * n_p * 2 + idx] += k1[idx]/6. + k2[idx]/3. + k3[idx]/3. + k4[idx]/6.;
        c[num_elem * n_p * 3 + idx] += k1[idx]/6. + k2[idx]/3. + k3[idx]/3. + k4[idx]/6.;
    }
}

/* right hand side
 *
 * computes the sum of the quadrature and the riemann flux for the 
 * coefficients for each element
 * THREADS: num_elem
 */
__global__ void eval_rhs_rk4(double *c, double *quad_rhs, double *left_riemann_rhs, double *right_riemann_rhs, 
                         int *elem_s1, int *elem_s2, int *elem_s3,
                         int *left_elem, double *J, 
                         double dt, int n_p, int num_sides, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    double s1_eqn1, s2_eqn1, s3_eqn1;
    double s1_eqn2, s2_eqn2, s3_eqn2;
    double s1_eqn3, s2_eqn3, s3_eqn3;
    double s1_eqn4, s2_eqn4, s3_eqn4;
    double register_J;
    int i, s1_idx, s2_idx, s3_idx;

    if (idx < num_elem) {

        register_J = J[idx];

        // get the indicies for the riemann contributions for this element
        s1_idx = elem_s1[idx];
        s2_idx = elem_s2[idx];
        s3_idx = elem_s3[idx];

        for (i = 0; i < n_p; i++) {

            // determine left or right pointing
            if (idx == left_elem[s1_idx]) {
                s1_eqn1 = left_riemann_rhs[num_sides * n_p * 0 + i * num_sides + s1_idx];
                s1_eqn2 = left_riemann_rhs[num_sides * n_p * 1 + i * num_sides + s1_idx];
                s1_eqn3 = left_riemann_rhs[num_sides * n_p * 2 + i * num_sides + s1_idx];
                s1_eqn4 = left_riemann_rhs[num_sides * n_p * 3 + i * num_sides + s1_idx];
            } else {
                s1_eqn1 = right_riemann_rhs[num_sides * n_p * 0 + i * num_sides + s1_idx];
                s1_eqn2 = right_riemann_rhs[num_sides * n_p * 1 + i * num_sides + s1_idx];
                s1_eqn3 = right_riemann_rhs[num_sides * n_p * 2 + i * num_sides + s1_idx];
                s1_eqn4 = right_riemann_rhs[num_sides * n_p * 3 + i * num_sides + s1_idx];
            }

            if (idx == left_elem[s2_idx]) {
                s2_eqn1 = left_riemann_rhs[num_sides * n_p * 0 + i * num_sides + s2_idx];
                s2_eqn2 = left_riemann_rhs[num_sides * n_p * 1 + i * num_sides + s2_idx];
                s2_eqn3 = left_riemann_rhs[num_sides * n_p * 2 + i * num_sides + s2_idx];
                s2_eqn4 = left_riemann_rhs[num_sides * n_p * 3 + i * num_sides + s2_idx];
            } else {
                s2_eqn1 = right_riemann_rhs[num_sides * n_p * 0 + i * num_sides + s2_idx];
                s2_eqn2 = right_riemann_rhs[num_sides * n_p * 1 + i * num_sides + s2_idx];
                s2_eqn3 = right_riemann_rhs[num_sides * n_p * 2 + i * num_sides + s2_idx];
                s2_eqn4 = right_riemann_rhs[num_sides * n_p * 3 + i * num_sides + s2_idx];
            }

            if (idx == left_elem[s3_idx]) {
                s3_eqn1 = left_riemann_rhs[num_sides * n_p * 0 + i * num_sides + s3_idx];
                s3_eqn2 = left_riemann_rhs[num_sides * n_p * 1 + i * num_sides + s3_idx];
                s3_eqn3 = left_riemann_rhs[num_sides * n_p * 2 + i * num_sides + s3_idx];
                s3_eqn4 = left_riemann_rhs[num_sides * n_p * 3 + i * num_sides + s3_idx];
            } else {
                s3_eqn1 = right_riemann_rhs[num_sides * n_p * 0 + i * num_sides + s3_idx];
                s3_eqn2 = right_riemann_rhs[num_sides * n_p * 1 + i * num_sides + s3_idx];
                s3_eqn3 = right_riemann_rhs[num_sides * n_p * 2 + i * num_sides + s3_idx];
                s3_eqn4 = right_riemann_rhs[num_sides * n_p * 3 + i * num_sides + s3_idx];
            }

            // calculate the coefficient c
            c[num_elem * n_p * 0 + i * num_elem + idx] = 1. / register_J * dt * (quad_rhs[num_elem * n_p * 0 + i * num_elem + idx] + s1_eqn1 + s2_eqn1 + s3_eqn1);
            c[num_elem * n_p * 1 + i * num_elem + idx] = 1. / register_J * dt * (quad_rhs[num_elem * n_p * 1 + i * num_elem + idx] + s1_eqn2 + s2_eqn2 + s3_eqn2);
            c[num_elem * n_p * 2 + i * num_elem + idx] = 1. / register_J * dt * (quad_rhs[num_elem * n_p * 2 + i * num_elem + idx] + s1_eqn3 + s2_eqn3 + s3_eqn3);
            c[num_elem * n_p * 3 + i * num_elem + idx] = 1. / register_J * dt * (quad_rhs[num_elem * n_p * 3 + i * num_elem + idx] + s1_eqn4 + s2_eqn4 + s3_eqn4);
        }
    }
}

void time_integrate_rk4(double dt, int n_quad, int n_quad1d, int n_p, int n, 
                    int num_elem, int num_sides, int timesteps) {
    int n_threads = 256;
    int i;
    double t;

    int n_blocks_elem     = (num_elem  / n_threads) + ((num_elem  % n_threads) ? 1 : 0);
    int n_blocks_sides    = (num_sides / n_threads) + ((num_sides % n_threads) ? 1 : 0);
    int n_blocks_rk4      = ((n_p * num_elem) / n_threads) + (((n_p * num_elem) % n_threads) ? 1 : 0);
    int n_blocks_rk4_temp = ((4 * n_p * num_elem) / n_threads) + (((4 * n_p * num_elem) % n_threads) ? 1 : 0);

    void (*eval_surface_ftn)(double*, double*, double*, 
                         double*, double*,
                         double*, double*,
                         double*, double*,
                         double*, double*,
                         int*, int*,
                         int*, int*,
                         double*, double*,
                         int, int, int, int, int, double) = NULL;
    void (*eval_volume_ftn)(double*, double*, 
                        double*, double*, 
                        double*, double*,
                        int, int, int) = NULL;
    switch (n) {
        case 0: eval_surface_ftn = eval_surface_wrapper0;
                eval_volume_ftn  = eval_volume_wrapper0;
                break;
        case 1: eval_surface_ftn = eval_surface_wrapper1;
                eval_volume_ftn  = eval_volume_wrapper1;
                break;
        case 2: eval_surface_ftn = eval_surface_wrapper2;
                eval_volume_ftn  = eval_volume_wrapper2;
                break;
        case 3: eval_surface_ftn = eval_surface_wrapper3;
                eval_volume_ftn  = eval_volume_wrapper3;
                break;
        case 4: eval_surface_ftn = eval_surface_wrapper4;
                eval_volume_ftn  = eval_volume_wrapper4;
                break;
        case 5: eval_surface_ftn = eval_surface_wrapper5;
                eval_volume_ftn  = eval_volume_wrapper5;
                break;
    }

    if ((eval_surface_ftn == NULL) || (eval_volume_ftn == NULL)) {
        printf("ERROR: dispatched kernel functions in rk4 were NULL.\n");
        exit(0);
    }

    for (i = 0; i < timesteps; i++) {
        t = i * dt;
        // stage 1
        checkCudaError("error before stage 1: eval_surface_ftn");
        eval_surface_ftn<<<n_blocks_sides, n_threads>>>
                        (d_c, d_left_riemann_rhs, d_right_riemann_rhs, 
                         d_s_length, d_J,
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         d_left_elem, d_right_elem,
                         d_left_side_number, d_right_side_number,
                         d_Nx, d_Ny, 
                         n_quad1d, n_quad, n_p, num_sides, num_elem, t);

        checkCudaError("error after stage 1: eval_surface_ftn");

        eval_volume_ftn<<<n_blocks_elem, n_threads>>>
                        (d_c, d_quad_rhs, 
                         d_xr, d_yr, d_xs, d_ys,
                         n_quad, n_p, num_elem);
        hipDeviceSynchronize();

        eval_rhs_rk4<<<n_blocks_elem, n_threads>>>(d_k1, d_quad_rhs, d_left_riemann_rhs, d_right_riemann_rhs, 
                                              d_elem_s1, d_elem_s2, d_elem_s3, 
                                              d_left_elem, d_J, dt, n_p, num_sides, num_elem);
        hipDeviceSynchronize();

        rk4_tempstorage<<<n_blocks_rk4_temp, n_threads>>>(d_c, d_kstar, d_k1, 0.5, n_p, num_elem);
        hipDeviceSynchronize();

        checkCudaError("error after stage 1.");

        // stage 2
        eval_surface_ftn<<<n_blocks_sides, n_threads>>>
                        (d_kstar, d_left_riemann_rhs, d_right_riemann_rhs, 
                         d_s_length, d_J,
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         d_left_elem, d_right_elem,
                         d_left_side_number, d_right_side_number,
                         d_Nx, d_Ny, 
                         n_quad1d, n_quad, n_p, num_sides, num_elem, t);

        eval_volume_ftn<<<n_blocks_elem, n_threads>>>
                        (d_kstar, d_quad_rhs, 
                         d_xr, d_yr, d_xs, d_ys,
                         n_quad, n_p, num_elem);
        hipDeviceSynchronize();

        eval_rhs_rk4<<<n_blocks_elem, n_threads>>>(d_k2, d_quad_rhs, d_left_riemann_rhs, d_right_riemann_rhs,
                                              d_elem_s1, d_elem_s2, d_elem_s3, 
                                              d_left_elem, d_J, dt, n_p, num_sides, num_elem);
        hipDeviceSynchronize();

        rk4_tempstorage<<<n_blocks_rk4_temp, n_threads>>>(d_c, d_kstar, d_k2, 0.5, n_p, num_elem);
        hipDeviceSynchronize();

        checkCudaError("error after stage 2.");

        // stage 3
        eval_surface_ftn<<<n_blocks_sides, n_threads>>>
                        (d_kstar, d_left_riemann_rhs, d_right_riemann_rhs, 
                         d_s_length, d_J,
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         d_left_elem, d_right_elem,
                         d_left_side_number, d_right_side_number,
                         d_Nx, d_Ny, 
                         n_quad1d, n_quad, n_p, num_sides, num_elem, t);

        eval_volume_ftn<<<n_blocks_elem, n_threads>>>
                        (d_kstar, d_quad_rhs, 
                         d_xr, d_yr, d_xs, d_ys,
                         n_quad, n_p, num_elem);
        hipDeviceSynchronize();

        eval_rhs_rk4<<<n_blocks_elem, n_threads>>>(d_k3, d_quad_rhs, d_left_riemann_rhs, d_right_riemann_rhs, 
                                              d_elem_s1, d_elem_s2, d_elem_s3, 
                                              d_left_elem, d_J, dt, n_p, num_sides, num_elem);
        hipDeviceSynchronize();

        rk4_tempstorage<<<n_blocks_rk4_temp, n_threads>>>(d_c, d_kstar, d_k3, 1.0, n_p, num_elem);
        hipDeviceSynchronize();

        checkCudaError("error after stage 3.");

        // stage 4
        eval_surface_ftn<<<n_blocks_sides, n_threads>>>
                        (d_kstar, d_left_riemann_rhs, d_right_riemann_rhs, 
                         d_s_length, d_J,
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         d_left_elem, d_right_elem,
                         d_left_side_number, d_right_side_number,
                         d_Nx, d_Ny, 
                         n_quad1d, n_quad, n_p, num_sides, num_elem, t);

        eval_volume_ftn<<<n_blocks_elem, n_threads>>>
                        (d_kstar, d_quad_rhs, 
                         d_xr, d_yr, d_xs, d_ys,
                         n_quad, n_p, num_elem);
        hipDeviceSynchronize();

        eval_rhs_rk4<<<n_blocks_elem, n_threads>>>(d_k4, d_quad_rhs, d_left_riemann_rhs, d_right_riemann_rhs, 
                                              d_elem_s1, d_elem_s2, d_elem_s3, 
                                              d_left_elem, d_J, dt, n_p, num_sides, num_elem);
        hipDeviceSynchronize();

        checkCudaError("error after stage 4.");
        
        // final stage
        rk4<<<n_blocks_rk4, n_threads>>>(d_c, d_k1, d_k2, d_k3, d_k4, n_p, num_elem);
        hipDeviceSynchronize();

        checkCudaError("error after final stage.");
    }
}

/***********************
 * FORWARD EULER
 ***********************/

__global__ void eval_rhs_fe(double *c, double *quad_rhs, double *left_riemann_rhs, double *right_riemann_rhs, 
                         int *elem_s1, int *elem_s2, int *elem_s3,
                         int *left_elem, double *J, 
                         double dt, int n_p, int num_sides, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    double s1_eqn1, s2_eqn1, s3_eqn1;
    double s1_eqn2, s2_eqn2, s3_eqn2;
    double s1_eqn3, s2_eqn3, s3_eqn3;
    double s1_eqn4, s2_eqn4, s3_eqn4;
    double register_J;
    int i, s1_idx, s2_idx, s3_idx;

    if (idx < num_elem) {

        register_J = J[idx];

        // get the indicies for the riemann contributions for this element
        s1_idx = elem_s1[idx];
        s2_idx = elem_s2[idx];
        s3_idx = elem_s3[idx];

        for (i = 0; i < n_p; i++) {

            // determine left or right pointing
            if (idx == left_elem[s1_idx]) {
                s1_eqn1 = left_riemann_rhs[num_sides * n_p * 0 + i * num_sides + s1_idx];
                s1_eqn2 = left_riemann_rhs[num_sides * n_p * 1 + i * num_sides + s1_idx];
                s1_eqn3 = left_riemann_rhs[num_sides * n_p * 2 + i * num_sides + s1_idx];
                s1_eqn4 = left_riemann_rhs[num_sides * n_p * 3 + i * num_sides + s1_idx];
            } else {
                s1_eqn1 = right_riemann_rhs[num_sides * n_p * 0 + i * num_sides + s1_idx];
                s1_eqn2 = right_riemann_rhs[num_sides * n_p * 1 + i * num_sides + s1_idx];
                s1_eqn3 = right_riemann_rhs[num_sides * n_p * 2 + i * num_sides + s1_idx];
                s1_eqn4 = right_riemann_rhs[num_sides * n_p * 3 + i * num_sides + s1_idx];
            }

            if (idx == left_elem[s2_idx]) {
                s2_eqn1 = left_riemann_rhs[num_sides * n_p * 0 + i * num_sides + s2_idx];
                s2_eqn2 = left_riemann_rhs[num_sides * n_p * 1 + i * num_sides + s2_idx];
                s2_eqn3 = left_riemann_rhs[num_sides * n_p * 2 + i * num_sides + s2_idx];
                s2_eqn4 = left_riemann_rhs[num_sides * n_p * 3 + i * num_sides + s2_idx];
            } else {
                s2_eqn1 = right_riemann_rhs[num_sides * n_p * 0 + i * num_sides + s2_idx];
                s2_eqn2 = right_riemann_rhs[num_sides * n_p * 1 + i * num_sides + s2_idx];
                s2_eqn3 = right_riemann_rhs[num_sides * n_p * 2 + i * num_sides + s2_idx];
                s2_eqn4 = right_riemann_rhs[num_sides * n_p * 3 + i * num_sides + s2_idx];
            }

            if (idx == left_elem[s3_idx]) {
                s3_eqn1 = left_riemann_rhs[num_sides * n_p * 0 + i * num_sides + s3_idx];
                s3_eqn2 = left_riemann_rhs[num_sides * n_p * 1 + i * num_sides + s3_idx];
                s3_eqn3 = left_riemann_rhs[num_sides * n_p * 2 + i * num_sides + s3_idx];
                s3_eqn4 = left_riemann_rhs[num_sides * n_p * 3 + i * num_sides + s3_idx];
            } else {
                s3_eqn1 = right_riemann_rhs[num_sides * n_p * 0 + i * num_sides + s3_idx];
                s3_eqn2 = right_riemann_rhs[num_sides * n_p * 1 + i * num_sides + s3_idx];
                s3_eqn3 = right_riemann_rhs[num_sides * n_p * 2 + i * num_sides + s3_idx];
                s3_eqn4 = right_riemann_rhs[num_sides * n_p * 3 + i * num_sides + s3_idx];
            }

            // calculate the coefficient c
            c[num_elem * n_p * 0 + i * num_elem + idx] += 1. / register_J * dt * (quad_rhs[num_elem * n_p * 0 + i * num_elem + idx] + s1_eqn1 + s2_eqn1 + s3_eqn1);
            c[num_elem * n_p * 1 + i * num_elem + idx] += 1. / register_J * dt * (quad_rhs[num_elem * n_p * 1 + i * num_elem + idx] + s1_eqn2 + s2_eqn2 + s3_eqn2);
            c[num_elem * n_p * 2 + i * num_elem + idx] += 1. / register_J * dt * (quad_rhs[num_elem * n_p * 2 + i * num_elem + idx] + s1_eqn3 + s2_eqn3 + s3_eqn3);
            c[num_elem * n_p * 3 + i * num_elem + idx] += 1. / register_J * dt * (quad_rhs[num_elem * n_p * 3 + i * num_elem + idx] + s1_eqn4 + s2_eqn4 + s3_eqn4);
        }
    }
}

// forward eulers
void time_integrate_fe(double dt, int n_quad, int n_quad1d, int n_p, int n, 
              int num_elem, int num_sides, int timesteps) {
    int n_threads = 128;
    int i;
    double t;

    int n_blocks_elem    = (num_elem  / n_threads) + ((num_elem  % n_threads) ? 1 : 0);
    int n_blocks_sides   = (num_sides / n_threads) + ((num_sides % n_threads) ? 1 : 0);

    void (*eval_surface_ftn)(double*, double*, double*, 
                         double*, double*,
                         double*, double*,
                         double*, double*,
                         double*, double*,
                         int*, int*,
                         int*, int*,
                         double*, double*,
                         int, int, int, int, int, double) = NULL;
    void (*eval_volume_ftn)(double*, double*, 
                        double*, double*, 
                        double*, double*,
                        int, int, int) = NULL;
    switch (n) {
        case 0: eval_surface_ftn = eval_surface_wrapper0;
                eval_volume_ftn  = eval_volume_wrapper0;
                break;
        case 1: eval_surface_ftn = eval_surface_wrapper1;
                eval_volume_ftn  = eval_volume_wrapper1;
                break;
        case 2: eval_surface_ftn = eval_surface_wrapper2;
                eval_volume_ftn  = eval_volume_wrapper2;
                break;
        case 3: eval_surface_ftn = eval_surface_wrapper3;
                eval_volume_ftn  = eval_volume_wrapper3;
                break;
        case 4: eval_surface_ftn = eval_surface_wrapper4;
                eval_volume_ftn  = eval_volume_wrapper4;
                break;
        case 5: eval_surface_ftn = eval_surface_wrapper5;
                eval_volume_ftn  = eval_volume_wrapper5;
                break;
    }
    if ((eval_surface_ftn == NULL) || (eval_volume_ftn == NULL)) {
        printf("ERROR: dispatched kernel functions in fe were NULL.\n");
        exit(0);
    }

    for (i = 0; i < timesteps; i++) {
        t = i * dt;
        eval_surface_ftn<<<n_blocks_sides, n_threads>>>
                        (d_c, d_left_riemann_rhs, d_right_riemann_rhs, 
                         d_s_length, d_J,
                         d_V1x, d_V1y,
                         d_V2x, d_V2y,
                         d_V3x, d_V3y,
                         d_left_elem, d_right_elem,
                         d_left_side_number, d_right_side_number,
                         d_Nx, d_Ny, 
                         n_quad1d, n_quad, n_p, num_sides, num_elem, t);
        hipDeviceSynchronize();

        checkCudaError("error after eval_surface_ftn");

        eval_volume_ftn<<<n_blocks_elem, n_threads>>>
                        (d_c, d_quad_rhs, 
                         d_xr, d_yr, d_xs, d_ys,
                         n_quad, n_p, num_elem);
        hipDeviceSynchronize();

        eval_rhs_fe<<<n_blocks_elem, n_threads>>>(d_c, d_quad_rhs, d_left_riemann_rhs, d_right_riemann_rhs, 
                                              d_elem_s1, d_elem_s2, d_elem_s3, 
                                              d_left_elem, d_J, dt, n_p, num_sides, num_elem);
        hipDeviceSynchronize();
    }
}
