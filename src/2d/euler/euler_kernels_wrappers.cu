#include "hip/hip_runtime.h"
/* eval surface wrapper (n = 0)
 *
 * wrapper function for the eval_surface device function.
 * THREADS: num_sides
 */
__global__ void eval_surface_wrapper0(double *c, double *left_riemann_rhs, double *right_riemann_rhs, 
                                      double *s_length, double *J,
                                      double *V1x, double *V1y,
                                      double *V2x, double *V2y,
                                      double *V3x, double *V3y,
                                      int *left_elem, int *right_elem,
                                      int *left_side_number, int *right_side_number, 
                                      double *Nx, double *Ny, 
                                      int n_quad1d, int n_quad, int n_p, int num_sides, int num_elem, double t) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        register double rho_left[1], u_left[1], v_left[1], E_left[1];
        register double rho_right[1], u_right[1], v_right[1], E_right[1];

        // grab the coefficients for the left & right elements
        // TODO: group all the boundary sides together so they are in the same warp;
        //       means no warp divergence
        if (right_elem[idx] != -1) {
            rho_left[0] = c[num_elem * n_p * 0 + left_elem[idx]];
            u_left[0]   = c[num_elem * n_p * 1 + left_elem[idx]];
            v_left[0]   = c[num_elem * n_p * 2 + left_elem[idx]];
            E_left[0]   = c[num_elem * n_p * 3 + left_elem[idx]];
            rho_right[0] = c[num_elem * n_p * 0 + right_elem[idx]];
            u_right[0]   = c[num_elem * n_p * 1 + right_elem[idx]];
            v_right[0]   = c[num_elem * n_p * 2 + right_elem[idx]];
            E_right[0]   = c[num_elem * n_p * 3 + right_elem[idx]];
        } else {
            rho_left[0] = c[num_elem * n_p * 0 + left_elem[idx]];
            u_left[0]   = c[num_elem * n_p * 1 + left_elem[idx]];
            v_left[0]   = c[num_elem * n_p * 2 + left_elem[idx]];
            E_left[0]   = c[num_elem * n_p * 3 + left_elem[idx]];
        }

        __syncthreads();

        eval_surface(rho_left, u_left, v_left, E_left,
                rho_right, u_right, v_right, E_right,
                left_riemann_rhs, right_riemann_rhs,
                s_length[idx], J[left_elem[idx]],
                V1x[left_elem[idx]], V1y[left_elem[idx]],
                V2x[left_elem[idx]], V2y[left_elem[idx]],
                V3x[left_elem[idx]], V3y[left_elem[idx]],
                left_elem[idx], right_elem[idx],
                left_side_number[idx], right_side_number[idx],
                Nx[idx], Ny[idx],
                n_quad1d, n_quad, n_p, num_sides, num_elem, t, idx);
    }
}


/* eval surface wrapper (n = 1)
 *
 * wrapper function for the eval_surface device function.
 * THREADS: num_sides
 */
__global__ void eval_surface_wrapper1(double *c, double *left_riemann_rhs, double *right_riemann_rhs, 
        double *s_length, double *J,
        double *V1x, double *V1y,
        double *V2x, double *V2y,
        double *V3x, double *V3y,
        int *left_elem, int *right_elem,
        int *left_side_number, int *right_side_number, 
        double *Nx, double *Ny, 
        int n_quad1d, int n_quad, int n_p, int num_sides, int num_elem, double t) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        register double rho_left[3], u_left[3], v_left[3], E_left[3];
        register double rho_right[3], u_right[3], v_right[3], E_right[3];
        int i;

        // grab the coefficients for the left & right elements
        // TODO: group all the boundary sides together so they are in the same warp;
        //       means no warp divergence
        if (right_elem[idx] != -1) {
            for (i = 0; i < 3; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
                rho_right[i] = c[num_elem * n_p * 0 + i * num_elem + right_elem[idx]];
                u_right[i]   = c[num_elem * n_p * 1 + i * num_elem + right_elem[idx]];
                v_right[i]   = c[num_elem * n_p * 2 + i * num_elem + right_elem[idx]];
                E_right[i]   = c[num_elem * n_p * 3 + i * num_elem + right_elem[idx]];
            }
        } else {
            for (i = 0; i < 3; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
            }
        }

        __syncthreads();

        eval_surface(rho_left, u_left, v_left, E_left,
                rho_right, u_right, v_right, E_right,
                left_riemann_rhs, right_riemann_rhs,
                s_length[idx], J[left_elem[idx]],
                V1x[left_elem[idx]], V1y[left_elem[idx]],
                V2x[left_elem[idx]], V2y[left_elem[idx]],
                V3x[left_elem[idx]], V3y[left_elem[idx]],
                left_elem[idx], right_elem[idx],
                left_side_number[idx], right_side_number[idx],
                Nx[idx], Ny[idx],
                n_quad1d, n_quad, n_p, num_sides, num_elem, t, idx);
    }
}

/* eval surface wrapper (n = 2)
 *
 * wrapper function for the eval_surface device function.
 * THREADS: num_sides
 */
__global__ void eval_surface_wrapper2(double *c, double *left_riemann_rhs, double *right_riemann_rhs, 
        double *s_length, double *J,
        double *V1x, double *V1y,
        double *V2x, double *V2y,
        double *V3x, double *V3y,
        int *left_elem, int *right_elem,
        int *left_side_number, int *right_side_number, 
        double *Nx, double *Ny, 
        int n_quad1d, int n_quad, int n_p, int num_sides, int num_elem, double t) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        register double rho_left[6], u_left[6], v_left[6], E_left[6];
        register double rho_right[6], u_right[6], v_right[6], E_right[6];
        int i;

        // grab the coefficients for the left & right elements
        // TODO: group all the boundary sides together so they are in the same warp;
        //       means no warp divergence
        if (right_elem[idx] != -1) {
            for (i = 0; i < 6; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
                rho_right[i] = c[num_elem * n_p * 0 + i * num_elem + right_elem[idx]];
                u_right[i]   = c[num_elem * n_p * 1 + i * num_elem + right_elem[idx]];
                v_right[i]   = c[num_elem * n_p * 2 + i * num_elem + right_elem[idx]];
                E_right[i]   = c[num_elem * n_p * 3 + i * num_elem + right_elem[idx]];
            }
        } else {
            for (i = 0; i < 6; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
            }
        }

        __syncthreads();

        eval_surface(rho_left, u_left, v_left, E_left,
                rho_right, u_right, v_right, E_right,
                left_riemann_rhs, right_riemann_rhs,
                s_length[idx], J[left_elem[idx]],
                V1x[left_elem[idx]], V1y[left_elem[idx]],
                V2x[left_elem[idx]], V2y[left_elem[idx]],
                V3x[left_elem[idx]], V3y[left_elem[idx]],
                left_elem[idx], right_elem[idx],
                left_side_number[idx], right_side_number[idx],
                Nx[idx], Ny[idx],
                n_quad1d, n_quad, n_p, num_sides, num_elem, t, idx);
    }
}


/* eval surface wrapper (n = 3)
 *
 * wrapper function for the eval_surface device function.
 * THREADS: num_sides
 */
__global__ void eval_surface_wrapper3(double *c, double *left_riemann_rhs, double *right_riemann_rhs, 
        double *s_length, double *J,
        double *V1x, double *V1y,
        double *V2x, double *V2y,
        double *V3x, double *V3y,
        int *left_elem, int *right_elem,
        int *left_side_number, int *right_side_number, 
        double *Nx, double *Ny, 
        int n_quad1d, int n_quad, int n_p, int num_sides, int num_elem, double t) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        register double rho_left[10], u_left[10], v_left[10], E_left[10];
        register double rho_right[10], u_right[10], v_right[10], E_right[10];
        int i;

        // grab the coefficients for the left & right elements
        // TODO: group all the boundary sides together so they are in the same warp;
        //       means no warp divergence
        if (right_elem[idx] != -1) {
            for (i = 0; i < 10; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
                rho_right[i] = c[num_elem * n_p * 0 + i * num_elem + right_elem[idx]];
                u_right[i]   = c[num_elem * n_p * 1 + i * num_elem + right_elem[idx]];
                v_right[i]   = c[num_elem * n_p * 2 + i * num_elem + right_elem[idx]];
                E_right[i]   = c[num_elem * n_p * 3 + i * num_elem + right_elem[idx]];
            }
        } else {
            for (i = 0; i < 10; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
            }
        }

        __syncthreads();

        eval_surface(rho_left, u_left, v_left, E_left,
                rho_right, u_right, v_right, E_right,
                left_riemann_rhs, right_riemann_rhs,
                s_length[idx], J[left_elem[idx]],
                V1x[left_elem[idx]], V1y[left_elem[idx]],
                V2x[left_elem[idx]], V2y[left_elem[idx]],
                V3x[left_elem[idx]], V3y[left_elem[idx]],
                left_elem[idx], right_elem[idx],
                left_side_number[idx], right_side_number[idx],
                Nx[idx], Ny[idx],
                n_quad1d, n_quad, n_p, num_sides, num_elem, t, idx);
    }
}


/* eval surface wrapper (n = 4)
 *
 * wrapper function for the eval_surface device function.
 * THREADS: num_sides
 */
__global__ void eval_surface_wrapper4(double *c, double *left_riemann_rhs, double *right_riemann_rhs, 
        double *s_length, double *J,
        double *V1x, double *V1y,
        double *V2x, double *V2y,
        double *V3x, double *V3y,
        int *left_elem, int *right_elem,
        int *left_side_number, int *right_side_number, 
        double *Nx, double *Ny, 
        int n_quad1d, int n_quad, int n_p, int num_sides, int num_elem, double t) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        register double rho_left[15], u_left[15], v_left[15], E_left[15];
        register double rho_right[15], u_right[15], v_right[15], E_right[15];
        int i;

        // grab the coefficients for the left & right elements
        // TODO: group all the boundary sides together so they are in the same warp;
        //       means no warp divergence
        if (right_elem[idx] != -1) {
            for (i = 0; i < 15; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
                rho_right[i] = c[num_elem * n_p * 0 + i * num_elem + right_elem[idx]];
                u_right[i]   = c[num_elem * n_p * 1 + i * num_elem + right_elem[idx]];
                v_right[i]   = c[num_elem * n_p * 2 + i * num_elem + right_elem[idx]];
                E_right[i]   = c[num_elem * n_p * 3 + i * num_elem + right_elem[idx]];
            }
        } else {
            for (i = 0; i < 15; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
            }
        }

        __syncthreads();

        eval_surface(rho_left, u_left, v_left, E_left,
                rho_right, u_right, v_right, E_right,
                left_riemann_rhs, right_riemann_rhs,
                s_length[idx], J[left_elem[idx]],
                V1x[left_elem[idx]], V1y[left_elem[idx]],
                V2x[left_elem[idx]], V2y[left_elem[idx]],
                V3x[left_elem[idx]], V3y[left_elem[idx]],
                left_elem[idx], right_elem[idx],
                left_side_number[idx], right_side_number[idx],
                Nx[idx], Ny[idx],
                n_quad1d, n_quad, n_p, num_sides, num_elem, t, idx);
    }
}


/* eval surface wrapper (n = 5)
 *
 * wrapper function for the eval_surface device function.
 * THREADS: num_sides
 */
__global__ void eval_surface_wrapper5(double *c, double *left_riemann_rhs, double *right_riemann_rhs, 
        double *s_length, double *J,
        double *V1x, double *V1y,
        double *V2x, double *V2y,
        double *V3x, double *V3y,
        int *left_elem, int *right_elem,
        int *left_side_number, int *right_side_number, 
        double *Nx, double *Ny, 
        int n_quad1d, int n_quad, int n_p, int num_sides, int num_elem, double t) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        register double rho_left[21], u_left[21], v_left[21], E_left[21];
        register double rho_right[21], u_right[21], v_right[21], E_right[21];
        int i;

        // grab the coefficients for the left & right elements
        // TODO: group all the boundary sides together so they are in the same warp;
        //       means no warp divergence
        if (right_elem[idx] != -1) {
            for (i = 0; i < 21; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
                rho_right[i] = c[num_elem * n_p * 0 + i * num_elem + right_elem[idx]];
                u_right[i]   = c[num_elem * n_p * 1 + i * num_elem + right_elem[idx]];
                v_right[i]   = c[num_elem * n_p * 2 + i * num_elem + right_elem[idx]];
                E_right[i]   = c[num_elem * n_p * 3 + i * num_elem + right_elem[idx]];
            }
        } else {
            for (i = 0; i < 21; i++) {
                rho_left[i] = c[num_elem * n_p * 0 + i * num_elem + left_elem[idx]];
                u_left[i]   = c[num_elem * n_p * 1 + i * num_elem + left_elem[idx]];
                v_left[i]   = c[num_elem * n_p * 2 + i * num_elem + left_elem[idx]];
                E_left[i]   = c[num_elem * n_p * 3 + i * num_elem + left_elem[idx]];
            }
        }

        __syncthreads();

        eval_surface(rho_left, u_left, v_left, E_left,
                rho_right, u_right, v_right, E_right,
                left_riemann_rhs, right_riemann_rhs,
                s_length[idx], J[left_elem[idx]],
                V1x[left_elem[idx]], V1y[left_elem[idx]],
                V2x[left_elem[idx]], V2y[left_elem[idx]],
                V3x[left_elem[idx]], V3y[left_elem[idx]],
                left_elem[idx], right_elem[idx],
                left_side_number[idx], right_side_number[idx],
                Nx[idx], Ny[idx],
                n_quad1d, n_quad, n_p, num_sides, num_elem, t, idx);
    }
}

//* eval volume wrapper (n = 0)
//*
//* wrapper function for the eval_volume device function.
//* THREADS: num_sides
__global__ void eval_volume_wrapper0(double *c, double *quad_rhs, 
        double *xr, double *yr,
        double *xs, double *ys,
        int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double rho[1], u[1], v[1], E[1];

        // get the coefficients for this element
        rho[0] = c[num_elem * n_p * 0 + idx];
        u[0]   = c[num_elem * n_p * 1 + idx];
        v[0]   = c[num_elem * n_p * 2 + idx];
        E[0]   = c[num_elem * n_p * 3 + idx];

        eval_volume(rho, u, v, E, quad_rhs,
                xr[idx], yr[idx],
                xs[idx], ys[idx],
                n_quad, n_p, num_elem, idx);
    }
}

//* eval volume wrapper (n = 1)
//*
//* wrapper function for the eval_volume device function.
//* THREADS: num_sides
__global__ void eval_volume_wrapper1(double *c, double *quad_rhs, 
        double *xr, double *yr,
        double *xs, double *ys,
        int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double rho[3], u[3], v[3], E[3];
        int i;

        // get the coefficients for this element
        for (i = 0; i < 3; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_volume(rho, u, v, E, quad_rhs,
                xr[idx], yr[idx],
                xs[idx], ys[idx],
                n_quad, n_p, num_elem, idx);

    }
}

//* eval volume wrapper (n = 2)
//*
//* wrapper function for the eval_volume device function.
//* THREADS: num_sides
__global__ void eval_volume_wrapper2(double *c, double *quad_rhs, 
        double *xr, double *yr,
        double *xs, double *ys,
        int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double rho[6], u[6], v[6], E[6];
        int i;

        // get the coefficients for this element
        for (i = 0; i < 6; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_volume(rho, u, v, E, quad_rhs,
                xr[idx], yr[idx],
                xs[idx], ys[idx],
                n_quad, n_p, num_elem, idx);

    }
}
//* eval volume wrapper (n = 3)
//*
//* wrapper function for the eval_volume device function.
//* THREADS: num_sides
__global__ void eval_volume_wrapper3(double *c, double *quad_rhs, 
        double *xr, double *yr,
        double *xs, double *ys,
        int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double rho[10], u[10], v[10], E[10];
        int i;

        // get the coefficients for this element
        for (i = 0; i < 10; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_volume(rho, u, v, E, quad_rhs,
                xr[idx], yr[idx],
                xs[idx], ys[idx],
                n_quad, n_p, num_elem, idx);

    }
}
//* eval volume wrapper (n = 4)
//*
//* wrapper function for the eval_volume device function.
//* THREADS: num_sides

__global__ void eval_volume_wrapper4(double *c, double *quad_rhs, 
        double *xr, double *yr,
        double *xs, double *ys,
        int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double rho[15], u[15], v[15], E[15];
        int i;

        // get the coefficients for this element
        for (i = 0; i < 15; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_volume(rho, u, v, E, quad_rhs,
                xr[idx], yr[idx],
                xs[idx], ys[idx],
                n_quad, n_p, num_elem, idx);

    }
}
//* eval volume wrapper (n = 5)
//*
//* wrapper function for the eval_volume device function.
//* THREADS: num_sides
__global__ void eval_volume_wrapper5(double *c, double *quad_rhs, 
        double *xr, double *yr,
        double *xs, double *ys,
        int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double rho[21], u[21], v[21], E[21];
        int i;

        // get the coefficients for this element
        for (i = 0; i < 21; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_volume(rho, u, v, E, quad_rhs,
                xr[idx], yr[idx],
                xs[idx], ys[idx],
                n_quad, n_p, num_elem, idx);

    }
}

//* eval lambda wrapper (n = 0)
//*
//* wrapper function for the eval_global_lambda device function.
//* THREADS: num_sides
 __global__ void eval_global_lambda_wrapper0(double *c, double *lambda, int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double rho[1], u[1], v[1], E[1];

        // get the coefficients for this element
        rho[0] = c[num_elem * n_p * 0 + idx];
        u[0]   = c[num_elem * n_p * 1 + idx];
        v[0]   = c[num_elem * n_p * 2 + idx];
        E[0]   = c[num_elem * n_p * 3 + idx];

        eval_global_lambda(rho, u, v, E, lambda, n_quad, n_p, idx);
    }
}

//* eval lambda wrapper (n = 1)
//*
//* wrapper function for the eval_global_lambda device function.
//* THREADS: num_sides
 __global__ void eval_global_lambda_wrapper1(double *c, double *lambda, int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double rho[3], u[3], v[3], E[3];
        int i;

        // get the coefficients for this element
        for (i = 0; i < 3; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_global_lambda(rho, u, v, E, lambda, n_quad, n_p, idx);
    }
}

//* eval lambda wrapper (n = 2)
//*
//* wrapper function for the eval_global_lambda device function.
//* THREADS: num_sides
 __global__ void eval_global_lambda_wrapper2(double *c, double *lambda, int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double rho[6], u[6], v[6], E[6];
        int i;

        // get the coefficients for this element
        for (i = 0; i < 6; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_global_lambda(rho, u, v, E, lambda, n_quad, n_p, idx);
         
    }
}
//* eval lambda wrapper (n = 3)
//*
//* wrapper function for the eval_global_lambda device function.
//* THREADS: num_sides
 __global__ void eval_global_lambda_wrapper3(double *c, double *lambda, int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
        
    if (idx < num_elem) {
        double rho[10], u[10], v[10], E[10];
        int i;

        // get the coefficients for this element
        for (i = 0; i < 10; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_global_lambda(rho, u, v, E, lambda, n_quad, n_p, idx);
         
    }
}
//* eval lambda wrapper (n = 4)
//*
//* wrapper function for the eval_global_lambda device function.
//* THREADS: num_sides
 
 __global__ void eval_global_lambda_wrapper4(double *c, double *lambda, int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double rho[15], u[15], v[15], E[15];
        int i;

        // get the coefficients for this element
        for (i = 0; i < 15; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_global_lambda(rho, u, v, E, lambda, n_quad, n_p, idx);
         
    }
}
//* eval lambda wrapper (n = 5)
//*
//* wrapper function for the eval_global_lambda device function.
//* THREADS: num_sides
 __global__ void eval_global_lambda_wrapper5(double *c, double *lambda, int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double rho[21], u[21], v[21], E[21];
        int i;

        // get the coefficients for this element
        for (i = 0; i < 21; i++) {
            rho[i] = c[num_elem * n_p * 0 + i * num_elem + idx];
            u[i]   = c[num_elem * n_p * 1 + i * num_elem + idx];
            v[i]   = c[num_elem * n_p * 2 + i * num_elem + idx];
            E[i]   = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_global_lambda(rho, u, v, E, lambda, n_quad, n_p, idx);
         
    }
}
//* eval u wrapper (n = 0)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_rho_wrapper0(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double c_rho[1];

        // get the coefficients for this element
        c_rho[0] = c[idx];

        eval_u(c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 1)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_rho_wrapper1(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double c_rho[3];

        // get the coefficients for this element
        c_rho[0] = c[idx];
        c_rho[1] = c[num_elem + idx];
        c_rho[2] = c[2 * num_elem + idx];

        eval_u(c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 2)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_rho_wrapper2(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double c_rho[6];

        // get the coefficients for this element
        c_rho[0] = c[idx];
        c_rho[1] = c[num_elem + idx];
        c_rho[2] = c[2 * num_elem + idx];
        c_rho[3] = c[3 * num_elem + idx];
        c_rho[4] = c[4 * num_elem + idx];
        c_rho[5] = c[5 * num_elem + idx];

        eval_u(c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval u wrapper (n = 3)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_rho_wrapper3(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double c_rho[10];

        // get the coefficients for this element
        c_rho[0] = c[idx];
        c_rho[1] = c[num_elem + idx];
        c_rho[2] = c[2 * num_elem + idx];
        c_rho[3] = c[3 * num_elem + idx];
        c_rho[4] = c[4 * num_elem + idx];
        c_rho[5] = c[5 * num_elem + idx];
        c_rho[6] = c[6 * num_elem + idx];
        c_rho[7] = c[7 * num_elem + idx];
        c_rho[8] = c[8 * num_elem + idx];
        c_rho[9] = c[9 * num_elem + idx];

        eval_u(c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval u wrapper (n = 4)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
 
__global__ void eval_rho_wrapper4(double *c, 
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double c_rho[15];

        // get the coefficients for this element
        c_rho[0] = c[idx];
        c_rho[1] = c[num_elem + idx];
        c_rho[2] = c[2 * num_elem + idx];
        c_rho[3] = c[3 * num_elem + idx];
        c_rho[4] = c[4 * num_elem + idx];
        c_rho[5] = c[5 * num_elem + idx];
        c_rho[6] = c[6 * num_elem + idx];
        c_rho[7] = c[7 * num_elem + idx];
        c_rho[8] = c[8 * num_elem + idx];
        c_rho[9] = c[9 * num_elem + idx];

        c_rho[10] = c[10 * num_elem + idx];
        c_rho[11] = c[11 * num_elem + idx];
        c_rho[12] = c[12 * num_elem + idx];
        c_rho[13] = c[13 * num_elem + idx];
        c_rho[14] = c[14 * num_elem + idx];

        eval_u(c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 5)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_rho_wrapper5(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double c_rho[21];

        // get the coefficients for this element
        c_rho[0] = c[idx];
        c_rho[1] = c[num_elem + idx];
        c_rho[2] = c[2 * num_elem + idx];
        c_rho[3] = c[3 * num_elem + idx];
        c_rho[4] = c[4 * num_elem + idx];
        c_rho[5] = c[5 * num_elem + idx];
        c_rho[6] = c[6 * num_elem + idx];
        c_rho[7] = c[7 * num_elem + idx];
        c_rho[8] = c[8 * num_elem + idx];
        c_rho[9] = c[9 * num_elem + idx];

        c_rho[10] = c[10 * num_elem + idx];
        c_rho[11] = c[11 * num_elem + idx];
        c_rho[12] = c[12 * num_elem + idx];
        c_rho[13] = c[13 * num_elem + idx];
        c_rho[14] = c[14 * num_elem + idx];
        c_rho[15] = c[15 * num_elem + idx];
        c_rho[16] = c[16 * num_elem + idx];
        c_rho[17] = c[17 * num_elem + idx];
        c_rho[18] = c[18 * num_elem + idx];
        c_rho[19] = c[19 * num_elem + idx];

        c_rho[20] = c[20 * num_elem + idx];

        eval_u(c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

__global__ void eval_u_wrapper0(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double c_u[1];
        double c_rho[1];

        // get the coefficients for this element
        c_u[0]   = c[n_p * num_elem + idx];
        c_rho[0] = c[idx];

        eval_u_velocity(c_u, c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 1)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_u_wrapper1(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double c_u[3];
        double c_rho[3];

        // get the coefficients for this element
        c_u[0] = c[n_p * num_elem + idx];
        c_u[1] = c[n_p * num_elem + num_elem + idx];
        c_u[2] = c[n_p * num_elem + 2 * num_elem + idx];
        c_rho[0] = c[idx];
        c_rho[1] = c[num_elem + idx];
        c_rho[2] = c[2 * num_elem + idx];

        eval_u_velocity(c_u, c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 2)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_u_wrapper2(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        double c_u[6];
        double c_rho[6];

        // get the coefficients for this element
        for (i = 0; i < 6; i++) {
            c_u[i]   = c[num_elem * n_p + i * num_elem + idx];
            c_rho[i] = c[i * num_elem + idx];
        }

        eval_u_velocity(c_u, c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval u wrapper (n = 3)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_u_wrapper3(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        double c_u[10];
        double c_rho[10];

        // get the coefficients for this element
        for (i = 0; i < 10; i++) {
            c_u[i] = c[num_elem * n_p + i * num_elem + idx];
            c_rho[i] = c[i * num_elem + idx];
        }

        eval_u_velocity(c_u, c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval u wrapper (n = 4)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
 
__global__ void eval_u_wrapper4(double *c, 
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        double c_u[15];
        double c_rho[15];

        // get the coefficients for this element
        for (i = 0; i < 15; i++) {
            c_u[i] = c[num_elem * n_p + i * num_elem + idx];
            c_rho[i] = c[i * num_elem + idx];
        }

        eval_u_velocity(c_u, c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 5)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_u_wrapper5(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        double c_u[21];
        double c_rho[21];

        // get the coefficients for this element
        for (i = 0; i < 21; i++) {
            c_u[i] = c[num_elem * n_p + i * num_elem + idx];
            c_rho[i] = c[i * num_elem + idx];
        }

        eval_u_velocity(c_u, c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval v wrapper (n = 0)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_v_wrapper0(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double c_v[1];
        double c_rho[1];

        // get the coefficients for this element
        c_v[0] = c[num_elem * n_p * 2 + idx];
        c_rho[0] = c[idx];

        eval_u_velocity(c_v, c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 1)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_v_wrapper1(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double c_v[3];
        double c_rho[3];

        // get the coefficients for this element
        c_v[0] = c[num_elem * n_p * 2 + idx];
        c_v[1] = c[num_elem * n_p * 2 + num_elem + idx];
        c_v[2] = c[num_elem * n_p * 2 + 2 * num_elem + idx];
        c_rho[0] = c[idx];
        c_rho[1] = c[num_elem + idx];
        c_rho[2] = c[2 * num_elem + idx];

        eval_u_velocity(c_v, c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 2)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_v_wrapper2(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        double c_v[6];
        double c_rho[6];

        // get the coefficients for this element
        for (i = 0; i < 6; i++) {
            c_v[i] = c[num_elem * n_p * 2 + i * num_elem + idx];
            c_rho[i] = c[i * num_elem + idx];
        }

        eval_u_velocity(c_v, c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval u wrapper (n = 3)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_v_wrapper3(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        double c_v[10];
        double c_rho[10];

        // get the coefficients for this element
        for (i = 0; i < 6; i++) {
            c_v[i] = c[num_elem * n_p * 2 + i * num_elem + idx];
            c_rho[i] = c[i * num_elem + idx];
        }

        eval_u_velocity(c_v, c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval u wrapper (n = 4)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
 
__global__ void eval_v_wrapper4(double *c, 
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        double c_v[15];
        double c_rho[15];

        // get the coefficients for this element
        for (i = 0; i < 6; i++) {
            c_v[i] = c[num_elem * n_p * 2 + i * num_elem + idx];
            c_rho[i] = c[i * num_elem + idx];
        }

        eval_u_velocity(c_v, c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 5)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_v_wrapper5(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        double c_v[21];
        double c_rho[21];

        // get the coefficients for this element
        for (i = 0; i < 21; i++) {
            c_v[i] = c[num_elem * n_p * 2 + i * num_elem + idx];
            c_rho[i] = c[i * num_elem + idx];
        }

        eval_u_velocity(c_v, c_rho, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval u wrapper (n = 0)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_E_wrapper0(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double c_E[1];

        // get the coefficients for this element
        c_E[0] = c[num_elem * n_p * 3 + idx];

        eval_u(c_E, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 1)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_E_wrapper1(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double c_E[3];

        // get the coefficients for this element
        c_E[0] = c[num_elem * n_p * 3 + idx];
        c_E[1] = c[num_elem * n_p * 3 + num_elem + idx];
        c_E[2] = c[num_elem * n_p * 3 + 2 * num_elem + idx];

        eval_u(c_E, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 2)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_E_wrapper2(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        double c_E[6];

        // get the coefficients for this element
        for (i = 0; i < 6; i++) {
            c_E[i] = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_u(c_E, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval u wrapper (n = 3)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_E_wrapper3(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        double c_E[10];

        // get the coefficients for this element
        for (i = 0; i < 10; i++) {
            c_E[i] = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_u(c_E, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}

//* eval u wrapper (n = 4)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
 
__global__ void eval_E_wrapper4(double *c, 
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        double c_E[15];

        // get the coefficients for this element
        for (i = 0; i < 15; i++) {
            c_E[i] = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_u(c_E, Uv1, Uv2, Uv3, num_elem, n_p, idx);
    }
}

//* eval u wrapper (n = 5)
//*
//* wrapper function for the eval_u device function.
//* THREADS: num_sides
__global__ void eval_E_wrapper5(double *c,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        double c_E[21];

        // get the coefficients for this element
        for (i = 0; i < 21; i++) {
            c_E[i] = c[num_elem * n_p * 3 + i * num_elem + idx];
        }

        eval_u(c_E, Uv1, Uv2, Uv3, num_elem, n_p, idx);
         
    }
}
