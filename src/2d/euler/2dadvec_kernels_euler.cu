
#include <hip/hip_runtime.h>
/* 2dadvec_kernels_euler.cu
 *
 * This file contains the kernels for the 2D euler DG method.
 *
 * d_t [   rho   ] + d_x [     rho * u    ] + d_y [    rho * v     ] = 0
 * d_t [ rho * u ] + d_x [ rho * u^2 + p  ] + d_y [   rho * u * v  ] = 0
 * d_t [ rho * v ] + d_x [  rho * u * v   ] + d_y [  rho * v^2 + p ] = 0
 * d_t [    E    ] + d_x [ u * ( E +  p ) ] + d_y [ v * ( E +  p ) ] = 0
 *

 */

#define PI 3.14159
#define GAMMA 1.4

/***********************
 *
 * DEVICE VARIABLES
 *
 ***********************/
/* These are always prefixed with d_ for "device" */
float *d_c;                 // coefficients for [rho, u, v, E]
float *d_quad_rhs;          // the right hand side containing the quadrature contributions
float *d_left_riemann_rhs;  // the right hand side containing the left riemann contributions
float *d_right_riemann_rhs; // the right hand side containing the right riemann contributions

// TODO: switch to low storage runge-kutta
// runge kutta variables
float *d_kstar;
float *d_k1;
float *d_k2;
float *d_k3;
float *d_k4;

// precomputed basis functions 
// TODO: maybe making these 2^n makes sure the offsets are cached more efficiently? who knows...
// precomputed basis functions ordered like so
//
// [phi_1(r1, s1), phi_1(r2, s2), ... , phi_1(r_nq, s_nq)   ]
// [phi_2(r1, s1), phi_2(r2, s2), ... , phi_2(r_nq, s_nq)   ]
// [   .               .           .            .           ]
// [   .               .           .            .           ]
// [   .               .           .            .           ]
// [phi_np(r1, s1), phi_np(r2, s2), ... , phi_np(r_nq, s_nq)]
//
__device__ __constant__ float basis[2048];
// note: these are multiplied by the weights
__device__ __constant__ float basis_grad_x[2048]; 
__device__ __constant__ float basis_grad_y[2048]; 

// precomputed basis functions evaluated along the sides. ordered
// similarly to basis and basis_grad_{x,y} but with one "matrix" for each side
// starting with side 0. to get to each side, offset with:
//      side_number * n_p * num_quad1d.
__device__ __constant__ float basis_side[1024];
__device__ __constant__ float basis_vertex[256];

// weights for 2d and 1d quadrature rules
__device__ __constant__ float w[32];
__device__ __constant__ float w_oned[16];

__device__ __constant__ float r1[32];
__device__ __constant__ float r2[32];
__device__ __constant__ float r_oned[32];

void set_basis(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis"), value, size * sizeof(float));
}
void set_basis_grad_x(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_grad_x"), value, size * sizeof(float));
}
void set_basis_grad_y(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_grad_y"), value, size * sizeof(float));
}
void set_basis_side(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_side"), value, size * sizeof(float));
}
void set_basis_vertex(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_vertex"), value, size * sizeof(float));
}
void set_w(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("w"), value, size * sizeof(float));
}
void set_w_oned(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("w_oned"), value, size * sizeof(float));
}
void set_r1(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("r1"), value, size * sizeof(float));
}
void set_r2(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("r2"), value, size * sizeof(float));
}
void set_r_oned(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("r_oned"), value, size * sizeof(float));
}

// tells which side (1, 2, or 3) to evaluate this boundary integral over
int *d_left_side_number;
int *d_right_side_number;

float *d_J;        // jacobian determinant 
float *d_min_J;      // for the min sized jacobian
float *d_s_length; // length of sides

// the num_elem values of the x and y coordinates for the two vertices defining a side
// TODO: can i delete these after the lengths are precomputed?
//       maybe these should be in texture memory?
float *d_s_V1x;
float *d_s_V1y;
float *d_s_V2x;
float *d_s_V2y;

// the num_elem values of the x and y partials
float *d_xr;
float *d_yr;
float *d_xs;
float *d_ys;

// the K indices of the sides for each element ranged 0->H-1
int *d_elem_s1;
int *d_elem_s2;
int *d_elem_s3;

// vertex x and y coordinates on the mesh which define an element
// TODO: can i delete these after the jacobians are precomputed?
//       maybe these should be in texture memory?
float *d_V1x;
float *d_V1y;
float *d_V2x;
float *d_V2y;
float *d_V3x;
float *d_V3y;

// stores computed values at three vertices
float *d_Uv1;
float *d_Uv2;
float *d_Uv3;

// normal vectors for the sides
float *d_Nx;
float *d_Ny;

// index lists for sides
int *d_left_elem;  // index of left  element for side idx
int *d_right_elem; // index of right element for side idx

/***********************
 *
 * DEVICE FUNCTIONS
 *
 ***********************/
/* riemann solver
 *
 * evaluates the riemann problem over the boundary using Gaussian quadrature
 * with Legendre polynomials as basis functions.
 */
__device__ float riemann(float u_left, float u_right) {
    return 0.5 * (u_left + u_right);
}

__device__ float pressure(float rho, float u, float v, float E) {
    return (GAMMA - 1) * (E - (u*u + v*v) / 2 * rho);
}

/***********************
 *
 * INITIAL CONDITIONS
 *
 ***********************/

/* initial condition function
 *
 * returns the value of the intial condition at point x
 */
__device__ float rho0(float x, float y) {
    return x;
}
__device__ float u0(float x, float y) {
    return 0.;
}
__device__ float v0(float x, float y) {
    return 1.;
}
__device__ float E0(float x, float y) {
    return 0.;
}

/* boundary exact
 *
 * returns the exact boundary conditions
 */
__device__ float boundary_exact_rho(float x, float y, float t) {
    return 0;
}
__device__ float boundary_exact_u(float x, float y, float t) {
    return 0;
}
__device__ float boundary_exact_v(float x, float y, float t) {
    return 1;
}
__device__ float boundary_exact_E(float x, float y, float t) {
    return 0;
}

/* u exact
 *
 * returns the exact value of u for error measurement.
 */
__device__ float uexact(float x, float y, float t) {
    return u0(x, y);
}

/* initial conditions
 *
 * computes the coefficients for the initial conditions
 * THREADS: num_elem
 */
__global__ void init_conditions(float *c, float *J,
                                float *V1x, float *V1y,
                                float *V2x, float *V2y,
                                float *V3x, float *V3y,
                                int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i, j;
    float x, y, rho, u, v, E;

    if (idx < num_elem) {
        for (i = 0; i < n_p; i++) {
            u = 0.;
            // perform quadrature
            for (j = 0; j < n_quad; j++) {
                // map from the canonical element to the actual point on the mesh
                // x = x2 * r + x3 * s + x1 * (1 - r - s)
                x = r1[j] * V2x[idx] + r2[j] * V3x[idx] + (1 - r1[j] - r2[j]) * V1x[idx];
                y = r1[j] * V2y[idx] + r2[j] * V3y[idx] + (1 - r1[j] - r2[j]) * V1y[idx];

                // evaluate rho, u, v, E there
                rho += w[j] * rho0(x, y) * basis[i * n_quad + j];
                u   += w[j] * u0(x, y)   * basis[i * n_quad + j];
                v   += w[j] * v0(x, y)   * basis[i * n_quad + j];
                E   += w[j] * E0(x, y)   * basis[i * n_quad + j];
            }

            c[num_elem * n_p * 0 + i * num_elem + idx] = rho;
            c[num_elem * n_p * 1 + i * num_elem + idx] = u;
            c[num_elem * n_p * 2 + i * num_elem + idx] = v;
            c[num_elem * n_p * 3 + i * num_elem + idx] = E;
        } 
    }
}

/* find min jacobian
 *
 * returns the min jacobian inside of min_J. 
 * each block computes the min jacobian inside of that block and stores it in the
 * blockIdx.x spot of the shared min_J variable.
 * NOTE: this is fixed for 256 threads.
 */
__global__ void min_jacobian(float *J, float *min_J, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;
    int i   = blockDim.x * (blockIdx.x * 256 * 2) + threadIdx.x;

    __shared__ float s_min[256];

    if (idx < num_elem) {
        // set all of min to J[idx] initially
        s_min[tid] = J[idx];
        __syncthreads();

        // test a few
        while (i < num_elem) {
            s_min[tid] = (s_min[tid] < s_min[i]) ? s_min[tid] : s_min[i];
            s_min[tid] = (s_min[tid] < s_min[i + 256]) ? s_min[tid] : s_min[i];
            i += gridDim.x * 256 * 2;
            __syncthreads();
        }

        // first half of the warps
        __syncthreads();
        if (tid < 128) {
            s_min[tid] = (s_min[tid] < s_min[tid + 128]) ? s_min[tid] : s_min[tid + 128];
        }

        // first and second warps
        __syncthreads();
        if (tid < 64) {
            s_min[tid] = (s_min[tid] < s_min[tid + 64]) ? s_min[tid] : s_min[tid + 64];
        }

        // unroll last warp
        __syncthreads();
        if (tid < 32) {
            if (blockDim.x >= 64) {
                s_min[tid] = (s_min[tid] < s_min[tid + 32]) ? s_min[tid] : s_min[tid + 32];
            }
            if (blockDim.x >= 32) {
                s_min[tid] = (s_min[tid] < s_min[tid + 16]) ? s_min[tid] : s_min[tid + 16];
            }
            if (blockDim.x >= 16) {
                s_min[tid] = (s_min[tid] < s_min[tid + 8]) ? s_min[tid] : s_min[tid + 8];
            }
            if (blockDim.x >= 8) {
                s_min[tid] = (s_min[tid] < s_min[tid + 4]) ? s_min[tid] : s_min[tid + 4];
            }
            if (blockDim.x >= 4) {
                s_min[tid] = (s_min[tid] < s_min[tid + 2]) ? s_min[tid] : s_min[tid + 2];
            }
            if (blockDim.x >= 2) {
                s_min[tid] = (s_min[tid] < s_min[tid + 1]) ? s_min[tid] : s_min[tid + 1];
            }
        }

        __syncthreads();
        if (tid == 0) {
            min_J[blockIdx.x] = s_min[0];
        }
    }
}

/***********************
 *
 * PRECOMPUTING
 *
 ***********************/

/* side length computer
 *
 * precomputes the length of each side.
 * THREADS: num_sides
 */ 
__global__ void preval_side_length(float *s_length, 
                              float *s_V1x, float *s_V1y, 
                              float *s_V2x, float *s_V2y,
                              int num_sides) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        // compute and store the length of the side
        s_length[idx] = sqrtf(pow(s_V1x[idx] - s_V2x[idx],2) + pow(s_V1y[idx] - s_V2y[idx],2));
    }
}

/* jacobian computing
 *
 * precomputes the jacobian determinant for each element.
 * THREADS: num_elem
 */
__global__ void preval_jacobian(float *J, 
                           float *V1x, float *V1y, 
                           float *V2x, float *V2y, 
                           float *V3x, float *V3y,
                           int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        float x1, y1, x2, y2, x3, y3;

        // read vertex points
        x1 = V1x[idx];
        y1 = V1y[idx];
        x2 = V2x[idx];
        y2 = V2y[idx];
        x3 = V3x[idx];
        y3 = V3y[idx];

        // calculate jacobian determinant
        // x = x2 * r + x3 * s + x1 * (1 - r - s)
        J[idx] = (x2 - x1) * (y3 - y1) - (x3 - x1) * (y2 - y1);
    }
}

/* evaluate normal vectors
 *
 * computes the normal vectors for each element along each side.
 * THREADS: num_sides
 *
 */
__global__ void preval_normals(float *Nx, float *Ny, 
                          float *s_V1x, float *s_V1y, 
                          float *s_V2x, float *s_V2y,
                          float *V1x, float *V1y, 
                          float *V2x, float *V2y, 
                          float *V3x, float *V3y,
                          int *left_side_number, int num_sides) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        float x, y, length;
        float sv1x, sv1y, sv2x, sv2y;
    
        sv1x = s_V1x[idx];
        sv1y = s_V1y[idx];
        sv2x = s_V2x[idx];
        sv2y = s_V2y[idx];
    
        // lengths of the vector components
        x = sv2x - sv1x;
        y = sv2y - sv1y;
    
        // normalize
        length = sqrtf(pow(x, 2) + pow(y, 2));

        // store the result
        Nx[idx] = -y / length;
        Ny[idx] =  x / length;
    }
}

__global__ void preval_normals_direction(float *Nx, float *Ny, 
                          float *V1x, float *V1y, 
                          float *V2x, float *V2y, 
                          float *V3x, float *V3y,
                          int *left_elem, int *left_side_number, int num_sides) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        float new_x, new_y, dot;
        float initial_x, initial_y, target_x, target_y;
        float x, y;
        int left_idx, side;

        // get left side's vertices
        left_idx = left_elem[idx];
        side     = left_side_number[idx];

        // get the normal vector
        x = Nx[idx];
        y = Ny[idx];
    
        // make it point the correct direction by learning the third vertex point
        switch (side) {
            case 0: 
                target_x = V3x[left_idx];
                target_y = V3y[left_idx];
                initial_x = (V1x[left_idx] + V2x[left_idx]) / 2.;
                initial_y = (V1y[left_idx] + V2y[left_idx]) / 2.;
                break;
            case 1:
                target_x = V1x[left_idx];
                target_y = V1y[left_idx];
                initial_x = (V2x[left_idx] + V3x[left_idx]) / 2.;
                initial_y = (V2y[left_idx] + V3y[left_idx]) / 2.;
                break;
            case 2:
                target_x = V2x[left_idx];
                target_y = V2y[left_idx];
                initial_x = (V1x[left_idx] + V3x[left_idx]) / 2.;
                initial_y = (V1y[left_idx] + V3y[left_idx]) / 2.;
                break;
        }

        // create the vector pointing towards the third vertex point
        new_x = target_x - initial_x;
        new_y = target_y - initial_y;

        // find the dot product between the normal and new vectors
        dot = x * new_x + y * new_y;
        
        if (dot > 0) {
            Nx[idx] *= -1;
            Ny[idx] *= -1;
        }
    }
}

__global__ void preval_partials(float *V1x, float *V1y,
                                float *V2x, float *V2y,
                                float *V3x, float *V3y,
                                float *xr,  float *yr,
                                float *xs,  float *ys, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num_elem) {
        // evaulate the jacobians of the mappings for the chain rule
        // x = x2 * r + x3 * s + x1 * (1 - r - s)
        xr[idx] = V2x[idx] - V1x[idx];
        yr[idx] = V2y[idx] - V1y[idx];
        xs[idx] = V3x[idx] - V1x[idx];
        ys[idx] = V3y[idx] - V1y[idx];
    }
}

/***********************
 *
 * MAIN FUNCTIONS
 *
 ***********************/

/* riemann evaluation
 *
 * device function to solve the riemann problem.
 */
__device__ void eval_riemann(float *c_rho_left, float *c_rho_right,
                              float *c_u_left,   float *c_u_right,
                              float *c_v_left,   float *c_v_right,
                              float *c_E_left,   float *c_E_right,
                              float v1x, float v1y,
                              float v2x, float v2y,
                              float v3x, float v3y,
                              int j, // j, as usual, is the index of the integration point
                              int left_side, int right_side,
                              int left_idx, int right_idx,
                              int n_p, int n_quad1d,
                              int num_sides, float t, 
                              float *rho, float *u, float *v, float *E) {

    int i;

    float rho_left, u_left, v_left, E_left;
    float rho_right, u_right, v_right, E_right;

    rho_left = 0.;
    u_left   = 0.;
    v_left   = 0.;
    E_left   = 0.;
    rho_right = 0.;
    u_right   = 0.;
    v_right   = 0.;
    E_right   = 0.;
    
    for (i = 0; i < n_p; i++) {
        rho_left += c_rho_left[i] * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
        u_left   += c_u_left[i]   * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
        v_left   += c_v_left[i]   * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
        E_left   += c_E_left[i]   * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
    }

    // make all threads in the first warps be boundary sides
    if (right_idx == -1) {
        float r1_eval, r2_eval;
        float x, y;

        // we need the mapping back to the grid space
        switch (left_side) {
            case 0: 
                r1_eval = 0.5 + 0.5 * r_oned[j];
                r2_eval = 0.;
                break;
            case 1: 
                r1_eval = (1. - r_oned[j]) / 2.;
                r2_eval = (1. + r_oned[j]) / 2.;
                break;
            case 2: 
                r1_eval = 0.;
                r2_eval = 0.5 + 0.5 * r_oned[n_quad1d - 1 - j];
                break;
        }

        // x = x2 * r + x3 * s + x1 * (1 - r - s)
        x = v2x * r1_eval + v3x * r2_eval + v1x * (1 - r1_eval - r2_eval);
        y = v2y * r1_eval + v3y * r2_eval + v1y * (1 - r1_eval - r2_eval);
            
        // deal with the boundary element here
        rho_right = boundary_exact_rho(x, y, t);
        u_right   = boundary_exact_u(x, y, t);
        v_right   = boundary_exact_v(x, y, t);
        E_right   = boundary_exact_E(x, y, t);

    } else {
        // evaluate the right side at the integration point
        for (i = 0; i < n_p; i++) {
            rho_right += c_rho_right[i] * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
            u_right   += c_u_right[i]   * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
            v_right   += c_v_right[i]   * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
            E_right   += c_E_right[i]   * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
        }
    }

    *rho =  riemann(rho_left, rho_right);
    *u   =  riemann(rho_left, rho_right);
    *v   =  riemann(rho_left, rho_right);
    *E   =  riemann(rho_left, rho_right);
}

/* surface integral evaluation
 *
 * evaluate all the riemann problems for each element.
 * THREADS: num_sides
 */
/*
 * d_t [   rho   ] + d_x [     rho * u    ] + d_y [    rho * v     ] = 0
 * d_t [ rho * u ] + d_x [ rho * u^2 + p  ] + d_y [   rho * u * v  ] = 0
 * d_t [ rho * v ] + d_x [  rho * u * v   ] + d_y [  rho * v^2 + p ] = 0
 * d_t [    E    ] + d_x [ u * ( E +  p ) ] + d_y [ v * ( E +  p ) ] = 0
 */

__device__ void eval_flux(float rho, float u, float v, float E, 
                     float *flux_x1, float *flux_y1,
                     float *flux_x2, float *flux_y2,
                     float *flux_x3, float *flux_y3,
                     float *flux_x4, float *flux_y4) {
    // evaluate pressure

    float p = pressure(rho, u, v, E);

    // flux_1 
    *flux_x1 = rho * u;
    *flux_y1 = rho * v;

    // flux_2
    *flux_x2 = rho * u * u + p;
    *flux_y2 = rho * u * v;

    // flux_3
    *flux_x3 = rho * u * v;
    *flux_y3 = rho * v * v + p;

    // flux_4
    *flux_x4 = u * (E + p);
    *flux_y4 = v * (E + p);
}

__device__ void eval_surface(float *rho_left, float *u_left, float *v_left, float *E_left,
                             float *rho_right, float *u_right, float *v_right, float *E_right,
                             float *left_riemann_rhs, float *right_riemann_rhs, 
                             float len,
                             float v1x, float v1y,
                             float v2x, float v2y,
                             float v3x, float v3y,
                             int left_idx,  int right_idx,
                             int left_side, int right_side, 
                             float nx, float ny, 
                             int n_quad1d, int n_p, int num_sides, 
                             int num_elem, float t, int idx) {
    int i, j;
    float rho, u, v, E;
    float left_sum1, right_sum1, flux_x1, flux_y1;
    float left_sum2, right_sum2, flux_x2, flux_y2;
    float left_sum3, right_sum3, flux_x3, flux_y3;
    float left_sum4, right_sum4, flux_x4, flux_y4;

    // multiply across by the i'th basis function
    for (i = 0; i < n_p; i++) {

        left_sum1  = 0.;
        left_sum2  = 0.;
        left_sum3  = 0.;
        left_sum4  = 0.;
        right_sum1 = 0.;
        right_sum2 = 0.;
        right_sum3 = 0.;
        right_sum4 = 0.;

        for (j = 0; j < n_quad1d; j++) {

            // calculate the riemann problems
            eval_riemann(rho_left, rho_right,
                         u_left,   u_right,
                         v_left,   v_right,
                         E_left,   E_right,
                         v1x, v1y, v2x, v2y, v3x, v3y,
                         j, left_side, right_side,
                         left_idx, right_idx,
                         n_p, n_quad1d, num_sides, t,
                         &rho, &u, &v, &E);

            // calculate the fluxes
            eval_flux(rho, u, v, E,
                 &flux_x1, &flux_y1, &flux_x2, &flux_y2,
                 &flux_x3, &flux_y3, &flux_x4, &flux_y4);

            // 1st row
            left_sum1  += (nx * flux_x1 + ny * flux_y1) * w_oned[j] * 
                           basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
            right_sum1 += (nx * flux_x1 + ny * flux_y1) * w_oned[j] * 
                           basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
            // 2nd row
            left_sum2  += (nx * flux_x2 + ny * flux_y2) * w_oned[j] * 
                           basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
            right_sum2 += (nx * flux_x2 + ny * flux_y2) * w_oned[j] * 
                           basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];

            // 3rd row
            left_sum3  += (nx * flux_x3 + ny * flux_y3) * w_oned[j] * 
                           basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
            right_sum3 += (nx * flux_x3 + ny * flux_y3) * w_oned[j] * 
                           basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];

            // 4th row
            left_sum4  += (nx * flux_x4 + ny * flux_y4) * w_oned[j] * 
                           basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
            right_sum4 += (nx * flux_x4 + ny * flux_y4) * w_oned[j] * 
                           basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
        }

        // store this side's contribution in the riemann rhs vectors
        left_riemann_rhs[num_elem * n_p * 0 + i * num_sides + idx]  = -len / 2 * left_sum1;
        left_riemann_rhs[num_elem * n_p * 1 + i * num_sides + idx]  = -len / 2 * left_sum2;
        left_riemann_rhs[num_elem * n_p * 2 + i * num_sides + idx]  = -len / 2 * left_sum3;
        left_riemann_rhs[num_elem * n_p * 3 + i * num_sides + idx]  = -len / 2 * left_sum4;
        right_riemann_rhs[num_elem * n_p * 0 + i * num_sides + idx] =  len / 2 * right_sum1;
        right_riemann_rhs[num_elem * n_p * 1 + i * num_sides + idx] =  len / 2 * right_sum2;
        right_riemann_rhs[num_elem * n_p * 2 + i * num_sides + idx] =  len / 2 * right_sum3;
        right_riemann_rhs[num_elem * n_p * 3 + i * num_sides + idx] =  len / 2 * right_sum4;
    }
}
/* flux boundary evaluation 
 *
 * evaulates the flux at the boundaries by handling them somehow.
 * THREADS: num_boundary
 */

/* volume integrals
 *
 * evaluates and adds the volume integral to the rhs vector
 * THREADS: num_elem
 */
__device__ void eval_volume(float *c_rho, float *c_u,
                            float *c_v,   float *c_E,
                            float *quad_rhs, 
                            float x_r, float y_r,
                            float x_s, float y_s,
                            int n_quad, int n_p, int num_elem, int idx) {
    int i, j, k;
    float rho, u, v, E;
    float flux_x1, flux_y1, flux_x2, flux_y2;
    float flux_x3, flux_y3, flux_x4, flux_y4;
    float sum1, sum2, sum3, sum4;

    // evaluate the volume integral for each coefficient
    for (i = 0; i < n_p; i++) {
        sum1 = 0.;
        sum2 = 0.;
        sum3 = 0.;
        sum4 = 0.;
        for (j = 0; j < n_quad; j++) {

            // evaluate rho, u, v, E at the integration point.
            rho = 0.;
            u   = 0.;
            v   = 0.;
            E   = 0.;
            for (k = 0; k < n_p; k++) {
                rho += c_rho[k] * basis[n_quad * k + j];
                u   += c_u[k]   * basis[n_quad * k + j];
                v   += c_v[k]   * basis[n_quad * k + j];
                E   += c_E[k]   * basis[n_quad * k + j];
            }

            // evaluate flux
            eval_flux(rho, u, v, E,
                 &flux_x1, &flux_y1, &flux_x2, &flux_y2,
                 &flux_x3, &flux_y3, &flux_x4, &flux_y4);
                 
            // Add to the sum
            // [fx fy] * [y_s, -y_r; -x_s, x_r] * [phi_x phi_y]
            sum1 += (  flux_x1 * ( basis_grad_x[n_quad * i + j] * y_s
                                  -basis_grad_y[n_quad * i + j] * y_r)
                     + flux_y1 * (-basis_grad_x[n_quad * i + j] * x_s 
                                 + basis_grad_y[n_quad * i + j] * x_r));
            sum2 += (  flux_x2 * ( basis_grad_x[n_quad * i + j] * y_s
                                  -basis_grad_y[n_quad * i + j] * y_r)
                     + flux_y2 * (-basis_grad_x[n_quad * i + j] * x_s 
                                 + basis_grad_y[n_quad * i + j] * x_r));
            sum3 += (  flux_x3 * ( basis_grad_x[n_quad * i + j] * y_s
                                  -basis_grad_y[n_quad * i + j] * y_r)
                     + flux_y3 * (-basis_grad_x[n_quad * i + j] * x_s 
                                 + basis_grad_y[n_quad * i + j] * x_r));
            sum4 += (  flux_x4 * ( basis_grad_x[n_quad * i + j] * y_s
                                  -basis_grad_y[n_quad * i + j] * y_r)
                     + flux_y4 * (-basis_grad_x[n_quad * i + j] * x_s 
                                 + basis_grad_y[n_quad * i + j] * x_r));
        }

        // store the result
        quad_rhs[num_elem * n_p * 0 + i * num_elem + idx] = sum1;
        quad_rhs[num_elem * n_p * 1 + i * num_elem + idx] = sum2;
        quad_rhs[num_elem * n_p * 2 + i * num_elem + idx] = sum3;
        quad_rhs[num_elem * n_p * 3 + i * num_elem + idx] = sum4;
    }
}

/* evaluate error
 * 
 * evaluates u at the three vertex points for output
 * THREADS: num_elem
 */
__device__ void eval_error(float *c, 
                       float v1x, float v1y,
                       float v2x, float v2y,
                       float v3x, float v3y,
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p, float t, int idx) {

    int i;
    float uv1, uv2, uv3;

    // calculate values at three vertex points
    uv1 = 0.;
    uv2 = 0.;
    uv3 = 0.;
    for (i = 0; i < n_p; i++) {
        uv1 += c[i] * basis_vertex[i * 3 + 0];
        uv2 += c[i] * basis_vertex[i * 3 + 1];
        uv3 += c[i] * basis_vertex[i * 3 + 2];
    }

    // store result
    Uv1[idx] = uv1 - uexact(v1x, v1y, t);
    Uv2[idx] = uv2 - uexact(v2x, v2y, t);
    Uv3[idx] = uv3 - uexact(v3x, v3y, t);
}

/* evaluate u
 * 
 * evaluates u at the three vertex points for output
 * THREADS: num_elem
 */
__device__ void eval_u(float *c, 
                       float *Uv1, float *Uv2, float *Uv3,
                       int num_elem, int n_p, int idx) {
    int i;
    float uv1, uv2, uv3;

    // calculate values at the integration points
    uv1 = 0.;
    uv2 = 0.;
    uv3 = 0.;
    for (i = 0; i < n_p; i++) {
        uv1 += c[i] * basis_vertex[i * 3 + 0];
        uv2 += c[i] * basis_vertex[i * 3 + 1];
        uv3 += c[i] * basis_vertex[i * 3 + 2];
    }

    // store result
    Uv1[idx] = uv1;
    Uv2[idx] = uv2;
    Uv3[idx] = uv3;
}
