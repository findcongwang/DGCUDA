#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "2dadvec_kernels.cu"
#include "2dadvec_kernels_wrappers.cu"
#include "quadrature.cu"
#include "basis.cu"
#include "time_integrator.cu"

/* 2dadvec.cu
 * 
 * This file calls the kernels in 2dadvec_kernels.cu for the 2D advection
 * DG method.
 */

/* set quadrature 
 *
 * sets the 1d quadrature integration points and weights for the boundary integrals
 * and the 2d quadrature integration points and weights for the volume intergrals.
 */
void set_quadrature(int n,
                    double **r1_local, double **r2_local, double **w_local,
                    double **s_r, double **oned_w_local, 
                    int *n_quad, int *n_quad1d) {
    int i;
    /*
     * The sides are mapped to the canonical element, so we want the integration points
     * for the boundary integrals for sides s1, s2, and s3 as shown below:

     s (r2) |\
     ^      | \
     |      |  \
     |      |   \
     |   s3 |    \ s2
     |      |     \
     |      |      \
     |      |       \
     |      |________\
     |         s1
     |
     ------------------------> r (r1)

    *
    */
    switch (n) {
        case 0: *n_quad = 1;
                *n_quad1d = 1;
                break;
        case 1: *n_quad = 3;
                *n_quad1d = 2;
                break;
        case 2: *n_quad = 6;
                *n_quad1d = 3;
                break;
        case 3: *n_quad = 12 ;
                *n_quad1d = 4;
                break;
        case 4: *n_quad = 16;
                *n_quad1d = 5;
                break;
        case 5: *n_quad = 25;
                *n_quad1d = 6;
                break;
        //case 6: *n_quad = 13;
                //*n_quad1d = 7;
                //break;
        //case 7: *n_quad = 16;
                //*n_quad1d = 8;
                //break;
        //case 8: *n_quad = 19;
                //*n_quad1d = 9;
                //break;
        //case 9: *n_quad = 25;
                //*n_quad1d = 10;
                //break;
    }
    // allocate integration points
    *r1_local = (double *)  malloc(*n_quad * sizeof(double));
    *r2_local = (double *)  malloc(*n_quad * sizeof(double));
    *w_local  =  (double *) malloc(*n_quad * sizeof(double));

    *s_r = (double *) malloc(*n_quad1d * sizeof(double));
    *oned_w_local = (double *) malloc(*n_quad1d * sizeof(double));

    // set 2D quadrature rules
    for (i = 0; i < *n_quad; i++) {
        if (n > 0) {
            (*r1_local)[i] = quad_2d[2 * n - 1][3*i];
            (*r2_local)[i] = quad_2d[2 * n - 1][3*i+1];
            (*w_local) [i] = quad_2d[2 * n - 1][3*i+2] / 2.; //weights are 2 times too big for some reason
        } else {
            (*r1_local)[i] = quad_2d[0][3*i];
            (*r2_local)[i] = quad_2d[0][3*i+1];
            (*w_local) [i] = quad_2d[0][3*i+2] / 2.; //weights are 2 times too big for some reason
        }
    }

    // set 1D quadrature rules
    for (i = 0; i < *n_quad1d; i++) {
        (*s_r)[i] = quad_1d[n][2*i];
        (*oned_w_local)[i] = quad_1d[n][2*i+1];
    }
}

void checkCudaError(const char *message)
{
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
        exit(-1);
    }
}

void read_mesh(FILE *mesh_file, 
              int *num_sides,
              int num_elem,
              double *V1x, double *V1y,
              double *V2x, double *V2y,
              double *V3x, double *V3y,
              int *left_side_number, int *right_side_number,
              double *sides_x1, double *sides_y1,
              double *sides_x2, double *sides_y2,
              int *elem_s1,  int *elem_s2, int *elem_s3,
              int *left_elem, int *right_elem) {

    int i, j, s1, s2, s3, numsides;
    double J, tmpx, tmpy;
    char line[100];
    numsides = 0;
    // stores the number of sides this element has.
    int *total_sides = (int *) malloc(num_elem * sizeof(int));
    for (i = 0; i < num_elem; i++) {
        total_sides[i] = 0;
    }

    i = 0;
    while(fgets(line, sizeof(line), mesh_file) != NULL) {
        // these three vertices define the element
        sscanf(line, "%lf %lf %lf %lf %lf %lf", &V1x[i], &V1y[i], &V2x[i], &V2y[i], &V3x[i], &V3y[i]);
        //printf("(%lf, %lf, %lf, %lf, %lf, %lf)\n", V1x[i], V1y[i], V2x[i], V2y[i], V3x[i], V3y[i]);

        // determine whether we should add these three sides or not
        s1 = 1;
        s2 = 1;
        s3 = 1;

        // enforce strictly positive jacobian
        J = (V2x[i] - V1x[i]) * (V3y[i] - V1y[i]) - (V3x[i] - V1x[i]) * (V2y[i] - V1y[i]);
        if (J < 0) {
            tmpx = V1x[i];
            tmpy = V1y[i];
            V1x[i] = V2x[i];
            V1y[i] = V2y[i];
            V2x[i] = tmpx;
            V2y[i] = tmpy;
        }

        // scan through the existing sides to see if we already added it
        // TODO: yeah, there's a better way to do this.
        // TODO: Also, this is super sloppy. should be checking indices instead of double values.
        for (j = 0; j < numsides; j++) {
            // side 1
            if (s1 && ((sides_x1[j] == V1x[i] && sides_y1[j] == V1y[i]
             && sides_x2[j] == V2x[i] && sides_y2[j] == V2y[i]) 
            || (sides_x2[j] == V1x[i] && sides_y2[j] == V1y[i]
             && sides_x1[j] == V2x[i] && sides_y1[j] == V2y[i]))) {
                s1 = 0;
                // OK, we've added this side to element i
                right_elem[j] = i;
                // link the added side j to this element
                elem_s1[i] = j;
                right_side_number[j] = 0;
                break;
            }
        }
        for (j = 0; j < numsides; j++) {
            // side 2
            if (s2 && ((sides_x1[j] == V2x[i] && sides_y1[j] == V2y[i]
             && sides_x2[j] == V3x[i] && sides_y2[j] == V3y[i]) 
            || (sides_x2[j] == V2x[i] && sides_y2[j] == V2y[i]
             && sides_x1[j] == V3x[i] && sides_y1[j] == V3y[i]))) {
                s2 = 0;
                // OK, we've added this side to some element before; which one?
                right_elem[j] = i;
                elem_s2[i] = j;
                // link the added side to this element
                right_side_number[j] = 1;
                break;
            }
        }
        for (j = 0; j < numsides; j++) {
            // side 3
            if (s3 && ((sides_x1[j] == V1x[i] && sides_y1[j] == V1y[i]
             && sides_x2[j] == V3x[i] && sides_y2[j] == V3y[i]) 
            || (sides_x2[j] == V1x[i] && sides_y2[j] == V1y[i]
             && sides_x1[j] == V3x[i] && sides_y1[j] == V3y[i]))) {
                s3 = 0;
                // OK, we've added this side to some element before; which one?
                right_elem[j] = i;
                elem_s3[i] = j;
                // link the added side to this element
                right_side_number[j] = 2;
                break;
            }
        }
        // if we haven't added the side already, add it
        if (s1) {
            sides_x1[numsides] = V1x[i];
            sides_y1[numsides] = V1y[i];
            sides_x2[numsides] = V2x[i];
            sides_y2[numsides] = V2y[i];

            // link the added side to this element
            left_side_number[numsides] = 0;
            // and link the element to this side
            elem_s1[i] = numsides;

            // make this the left element
            left_elem[numsides] = i;
            numsides++;
        }
        if (s2) {
            sides_x1[numsides] = V2x[i];
            sides_y1[numsides] = V2y[i];
            sides_x2[numsides] = V3x[i];
            sides_y2[numsides] = V3y[i];

            // link the added side to this element
            left_side_number[numsides] = 1;
            // and link the element to this side
            elem_s2[i] = numsides;

            // make this the left element
            left_elem[numsides] = i;
            numsides++;
        }
        if (s3) {
            sides_x1[numsides] = V3x[i];
            sides_y1[numsides] = V3y[i];
            sides_x2[numsides] = V1x[i];
            sides_y2[numsides] = V1y[i];

            // link the added side to this element
            left_side_number[numsides] = 2;
            // and link the element to this side
            elem_s3[i] = numsides;

            // make this the left element
            left_elem[numsides] = i;
            numsides++;
        }
        i++;
    }
    //free(total_sides);
    *num_sides = numsides;
}

void init_gpu(int num_elem, int num_sides, int n_p,
              double *V1x, double *V1y, 
              double *V2x, double *V2y, 
              double *V3x, double *V3y, 
              int *left_side_number, int *right_side_number,
              double *sides_x1, double *sides_y1,
              double *sides_x2, double *sides_y2,
              int *elem_s1, int *elem_s2, int *elem_s3,
              int *left_elem, int *right_elem) {
    checkCudaError("error before init.");
    hipDeviceReset();

    hipMalloc((void **) &d_c,        num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_quad_rhs, num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_left_riemann_rhs,  num_sides * n_p * sizeof(double));
    hipMalloc((void **) &d_right_riemann_rhs, num_sides * n_p * sizeof(double));

    hipMalloc((void **) &d_kstar, num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_k1, num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_k2, num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_k3, num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_k4, num_elem * n_p * sizeof(double));

    hipMalloc((void **) &d_J, num_elem * sizeof(double));
    hipMalloc((void **) &d_s_length, num_sides * sizeof(double));

    hipMalloc((void **) &d_s_V1x, num_sides * sizeof(double));
    hipMalloc((void **) &d_s_V2x, num_sides * sizeof(double));
    hipMalloc((void **) &d_s_V1y, num_sides * sizeof(double));
    hipMalloc((void **) &d_s_V2y, num_sides * sizeof(double));

    hipMalloc((void **) &d_elem_s1, num_elem * sizeof(int));
    hipMalloc((void **) &d_elem_s2, num_elem * sizeof(int));
    hipMalloc((void **) &d_elem_s3, num_elem * sizeof(int));

    hipMalloc((void **) &d_Uv1, num_elem * sizeof(double));
    hipMalloc((void **) &d_Uv2, num_elem * sizeof(double));
    hipMalloc((void **) &d_Uv3, num_elem * sizeof(double));

    hipMalloc((void **) &d_V1x, num_elem * sizeof(double));
    hipMalloc((void **) &d_V1y, num_elem * sizeof(double));
    hipMalloc((void **) &d_V2x, num_elem * sizeof(double));
    hipMalloc((void **) &d_V2y, num_elem * sizeof(double));
    hipMalloc((void **) &d_V3x, num_elem * sizeof(double));
    hipMalloc((void **) &d_V3y, num_elem * sizeof(double));

    hipMalloc((void **) &d_xr, num_elem * sizeof(double));
    hipMalloc((void **) &d_yr, num_elem * sizeof(double));
    hipMalloc((void **) &d_xs, num_elem * sizeof(double));
    hipMalloc((void **) &d_ys, num_elem * sizeof(double));

    hipMalloc((void **) &d_left_side_number , num_sides * sizeof(int));
    hipMalloc((void **) &d_right_side_number, num_sides * sizeof(int));

    hipMalloc((void **) &d_Nx, num_sides * sizeof(double));
    hipMalloc((void **) &d_Ny, num_sides * sizeof(double));

    hipMalloc((void **) &d_right_elem, num_sides * sizeof(int));
    hipMalloc((void **) &d_left_elem , num_sides * sizeof(int));

    // set d_c to 0 not necessary
    //hipMemset(d_c, 0., num_elem * n_p * sizeof(double));
    hipMemset(d_quad_rhs, 0., num_elem * n_p * sizeof(double));

    // copy over data
    hipMemcpy(d_s_V1x, sides_x1, num_sides * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_s_V1y, sides_y1, num_sides * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_s_V2x, sides_x2, num_sides * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_s_V2y, sides_y2, num_sides * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_left_side_number , left_side_number , num_sides * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_right_side_number, right_side_number, num_sides * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_elem_s1, elem_s1, num_elem * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_elem_s2, elem_s2, num_elem * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_elem_s3, elem_s3, num_elem * sizeof(int), hipMemcpyHostToDevice);
    checkCudaError("error inside gpu init.");

    hipMemcpy(d_V1x, V1x, num_elem * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V1y, V1y, num_elem * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V2x, V2x, num_elem * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V2y, V2y, num_elem * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V3x, V3x, num_elem * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V3y, V3y, num_elem * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_left_elem , left_elem , num_sides * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_right_elem, right_elem, num_sides * sizeof(int), hipMemcpyHostToDevice);
}

void free_gpu() {
    hipFree(d_c);
    hipFree(d_quad_rhs);
    hipFree(d_left_riemann_rhs);
    hipFree(d_right_riemann_rhs);

    hipFree(d_kstar);
    hipFree(d_k1);
    hipFree(d_k2);
    hipFree(d_k3);
    hipFree(d_k4);

    hipFree(d_J);
    hipFree(d_s_length);

    hipFree(d_s_V1x);
    hipFree(d_s_V2x);
    hipFree(d_s_V1y);
    hipFree(d_s_V2y);

    hipFree(d_elem_s1);
    hipFree(d_elem_s2);
    hipFree(d_elem_s3);

    hipFree(d_Uv1);
    hipFree(d_Uv2);
    hipFree(d_Uv3);

    hipFree(d_V1x);
    hipFree(d_V1y);
    hipFree(d_V2x);
    hipFree(d_V2y);
    hipFree(d_V3x);
    hipFree(d_V3y);

    hipFree(d_xr);
    hipFree(d_yr);
    hipFree(d_xs);
    hipFree(d_ys);

    hipFree(d_left_side_number);
    hipFree(d_right_side_number);

    hipFree(d_Nx);
    hipFree(d_Ny);

    hipFree(d_right_elem);
    hipFree(d_left_elem);
}

void usage_error() {
    printf("\nUsage: dgcuda [OPTIONS] [MESH] [OUTFILE]\n");
    printf(" Options: [-n] Order of polynomial approximation.\n");
    printf("          [-t] Number of timesteps.\n");
    printf("          [-d] Debug.\n");
}

int get_input(int argc, char *argv[],
               int *n, int *debug, int *timesteps, int *alpha,
               char **mesh_filename, char **out_filename) {

    int i;

    *timesteps = 1;
    *debug     = 0;
    // read command line input
    if (argc < 5) {
        usage_error();
        return 1;
    }
    for (i = 0; i < argc; i++) {
        // order of polynomial
        if (strcmp(argv[i], "-n") == 0) {
            if (i + 1 < argc) {
                *n = atoi(argv[i+1]);
                if (*n < 0 || *n > 5) {
                    usage_error();
                    return 1;
                }
            } else {
                usage_error();
                return 1;
            }
        }
        if (strcmp(argv[i], "-t") == 0) {
            if (i + 1 < argc) {
                *timesteps = atoi(argv[i+1]);
                if (*timesteps < 0) {
                    usage_error();
                    return 1;
                }
            } else {
                usage_error();
                return 1;
            }
        }
        if (strcmp(argv[i], "-d") == 0) {
            *debug = 1;
        }
        if (strcmp(argv[i], "-a") == 0) {
            if (i + 1 < argc) {
                *alpha = atoi(argv[i+1]);
                if (*alpha < 0) {
                    usage_error();
                    return 1;
                }
            } else {
                usage_error();
                return 1;
            }
        }
    } 

    // second last argument is filename
    *mesh_filename = argv[argc - 2];
    // last argument is outfilename
    *out_filename  = argv[argc - 1];

    return 0;
}

void test_initial_condition(double *c,
                            double *V1x, double *V1y,
                            double *V2x, double *V2y,
                            double *V3x, double *V3y,
                            double *r1_local, double *r2_local,
                            double *w_local, double *basis_local, int n_quad, int n_p) {   

    int i, j;
    double u, x, y;

    for (i = 0; i < n_p; i++) {
        u = 0.;
        // perform quadrature
        for (j = 0; j < n_quad; j++) {
            // map from the canonical element to the actual point on the mesh
            // x = x2 * r + x3 * s + x1 * (1 - r - s)
            x = r1_local[j] * V2x[0] + r2_local[j] * V3x[0] + (1 - r1_local[j] - r2_local[j]) * V1x[0];
            y = r1_local[j] * V2y[0] + r2_local[j] * V3y[0] + (1 - r1_local[j] - r2_local[j]) * V1y[0];

                // evaluate u there
            u += w_local[j] * pow(x - y, 2) * basis_local[i * n_quad + j];
        }
        c[i] = u;
        printf("c[%i] = %lf\n", i, c[i]);
    }
}

