#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "2dadvec_kernels.cu"
#include "quadrature.h"
#include "basis.h"

/* 2dadvec.cu
 * 
 * This file calls the kernels in 2dadvec_kernels.cu for the 2D advection
 * DG method.
 */

/* set quadrature 
 *
 * sets the 1d quadrature integration points and weights for the boundary integrals
 * and the 2d quadrature integration points and weights for the volume intergrals.
 */
void set_quadrature(int p,
                    float **r1_local, float **r2_local, float **w_local,
                    float **s_r, float **oned_w_local, 
                    int *n_quad, int *n_quad1d) {
    int i;
    /*
     * The sides are mapped to the canonical element, so we want the integration points
     * for the boundary integrals for sides s1, s2, and s3 as shown below:

     s (r2) |\
     ^      | \
     |      |  \
     |      |   \
     |   s3 |    \ s2
     |      |     \
     |      |      \
     |      |       \
     |      |________\
     |         s1
     |
     ------------------------> r (r1)

    *
    */
    switch (p) {
        case 0: *n_quad = 1;
                *n_quad1d = 1;
                break;
        case 1: *n_quad = 3;
                *n_quad1d = 2;
                break;
        case 2: *n_quad = 4;
                *n_quad1d = 3;
                break;
        case 3: *n_quad = 6 ;
                *n_quad1d = 4;
                break;
        case 4: *n_quad = 7;
                *n_quad1d = 5;
                break;
        case 5: *n_quad = 12;
                *n_quad1d = 6;
                break;
        case 6: *n_quad = 13;
                *n_quad1d = 7;
                break;
        case 7: *n_quad = 16;
                *n_quad1d = 8;
                break;
        case 8: *n_quad = 19;
                *n_quad1d = 9;
                break;
        case 9: *n_quad = 25;
                *n_quad1d = 10;
                break;
    }
    // allocate integration points
    *r1_local = (float *) malloc(*n_quad * sizeof(float));
    *r2_local = (float *) malloc(*n_quad * sizeof(float));
    *w_local  =  (float *) malloc(*n_quad * sizeof(float));

    *s_r = (float *) malloc(*n_quad1d * sizeof(float));
    *oned_w_local = (float *) malloc(*n_quad1d * sizeof(float));

    // set 2D quadrature rules
    for (i = 0; i < *n_quad; i++) {
        (*r1_local)[i] = quad_2d[p][3*i];
        (*r2_local)[i] = quad_2d[p][3*i+1];
        (*w_local) [i] = quad_2d[p][3*i+2] / 2.; //weights are 2 times too big for some reason
    }

    // set 1D quadrature rules
    for (i = 0; i < *n_quad1d; i++) {
        (*s_r)[i] = quad_1d[p][2*i];
        (*oned_w_local)[i] = quad_1d[p][2*i+1];
    }
}

void checkCudaError(const char *message)
{
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
        exit(-1);
    }
}

void read_mesh(FILE *mesh_file, 
              int *num_sides,
              int num_elem,
              float *V1x, float *V1y,
              float *V2x, float *V2y,
              float *V3x, float *V3y,
              int *left_side_number, int *right_side_number,
              float *sides_x1, float *sides_y1,
              float *sides_x2, float *sides_y2,
              int *elem_s1,  int *elem_s2, int *elem_s3,
              int *left_elem, int *right_elem) {

    int i, j, s1, s2, s3, numsides;
    float J, tmpx, tmpy;
    char line[100];
    numsides = 0;
    // stores the number of sides this element has.
    int *total_sides = (int *) malloc(num_elem * sizeof(int));
    for (i = 0; i < num_elem; i++) {
        total_sides[i] = 0;
    }

    i = 0;
    while(fgets(line, sizeof(line), mesh_file) != NULL) {
        // these three vertices define the element
        sscanf(line, "%f %f %f %f %f %f", &V1x[i], &V1y[i], &V2x[i], &V2y[i], &V3x[i], &V3y[i]);

        // determine whether we should add these three sides or not
        s1 = 1;
        s2 = 1;
        s3 = 1;

        J = (V2x[i] - V1x[i]) * (V3y[i] - V1y[i]) - (V3x[i] - V1x[i]) * (V2y[i] - V1y[i]);
        if (J < 0) {
            tmpx = V1x[i];
            tmpy = V1y[i];
            V1x[i] = V2x[i];
            V1y[i] = V2y[i];
            V2x[i] = tmpx;
            V2y[i] = tmpy;
        }

        // scan through the existing sides to see if we already added it
        // TODO: yeah, there's a better way to do this.
        // TODO: Also, this is super sloppy. should be checking indices instead of float values.
        for (j = 0; j < numsides; j++) {
            // side 1
            if (s1 && ((sides_x1[j] == V1x[i] && sides_y1[j] == V1y[i]
             && sides_x2[j] == V2x[i] && sides_y2[j] == V2y[i]) 
            || (sides_x2[j] == V1x[i] && sides_y2[j] == V1y[i]
             && sides_x1[j] == V2x[i] && sides_y1[j] == V2y[i]))) {
                s1 = 0;
                // OK, we've added this side to element i
                right_elem[j] = i;
                // link the added side j to this element
                elem_s1[i] = j;
                right_side_number[j] = 0;
                break;
            }
        }
        for (j = 0; j < numsides; j++) {
            // side 2
            if (s2 && ((sides_x1[j] == V2x[i] && sides_y1[j] == V2y[i]
             && sides_x2[j] == V3x[i] && sides_y2[j] == V3y[i]) 
            || (sides_x2[j] == V2x[i] && sides_y2[j] == V2y[i]
             && sides_x1[j] == V3x[i] && sides_y1[j] == V3y[i]))) {
                s2 = 0;
                // OK, we've added this side to some element before; which one?
                right_elem[j] = i;
                elem_s2[i] = j;
                // link the added side to this element
                right_side_number[j] = 1;
                break;
            }
        }
        for (j = 0; j < numsides; j++) {
            // side 3
            if (s3 && ((sides_x1[j] == V1x[i] && sides_y1[j] == V1y[i]
             && sides_x2[j] == V3x[i] && sides_y2[j] == V3y[i]) 
            || (sides_x2[j] == V1x[i] && sides_y2[j] == V1y[i]
             && sides_x1[j] == V3x[i] && sides_y1[j] == V3y[i]))) {
                s3 = 0;
                // OK, we've added this side to some element before; which one?
                right_elem[j] = i;
                elem_s3[i] = j;
                // link the added side to this element
                right_side_number[j] = 2;
                break;
            }
        }
        // if we haven't added the side already, add it
        if (s1) {
            sides_x1[numsides] = V1x[i];
            sides_y1[numsides] = V1y[i];
            sides_x2[numsides] = V2x[i];
            sides_y2[numsides] = V2y[i];
            //third_x[numsides] = V3x[i];
            //third_y[numsides] = V3y[i];

            // link the added side to this element
            left_side_number[numsides] = 0;
            // and link the element to this side
            elem_s1[i] = numsides;

            // make this the left element
            left_elem[numsides] = i;
            numsides++;
        }
        if (s2) {
            sides_x1[numsides] = V2x[i];
            sides_y1[numsides] = V2y[i];
            sides_x2[numsides] = V3x[i];
            sides_y2[numsides] = V3y[i];

            // link the added side to this element
            left_side_number[numsides] = 1;
            // and link the element to this side
            elem_s2[i] = numsides;

            // make this the left element
            left_elem[numsides] = i;
            numsides++;
        }
        if (s3) {
            sides_x1[numsides] = V3x[i];
            sides_y1[numsides] = V3y[i];
            sides_x2[numsides] = V1x[i];
            sides_y2[numsides] = V1y[i];

            // link the added side to this element
            left_side_number[numsides] = 2;
            // and link the element to this side
            elem_s3[i] = numsides;

            // make this the left element
            left_elem[numsides] = i;
            numsides++;
        }
        i++;
    }
    //free(total_sides);
    *num_sides = numsides;
}

void time_integrate(float dt, int n_quad, int n_quad1d, int n_p, int num_elem, int num_sides, int debug) {
    int n_threads = 256;

    int n_blocks_elem    = (num_elem  / n_threads) + ((num_elem  % n_threads) ? 1 : 0);
    int n_blocks_sides   = (num_sides / n_threads) + ((num_sides % n_threads) ? 1 : 0);

    // stage 1
    checkCudaError("error before stage 1: eval_riemann");
    eval_riemann<<<n_blocks_sides, n_threads>>>
                    (d_c, d_left_riemann_rhs, d_right_riemann_rhs, d_J, 
                     d_s_length,
                     d_s_r,
                     d_V1x, d_V1y,
                     d_V2x, d_V2y,
                     d_V3x, d_V3y,
                     d_left_elem, d_right_elem,
                     d_left_side_number, d_right_side_number,
                     d_Nx, d_Ny, 
                     n_quad1d, n_p, num_sides, num_elem);
    hipDeviceSynchronize();

    if (debug) {
        printf("\n\n dt = %f -\n", dt);
        printf("-------------------------\n");
        float *left_rhs = (float *) malloc(num_sides * n_p * sizeof(float));
        float *right_rhs = (float *) malloc(num_sides * n_p * sizeof(float));
        hipMemcpy(left_rhs, d_left_riemann_rhs, num_sides * n_p * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(right_rhs, d_right_riemann_rhs, num_sides * n_p * sizeof(float), hipMemcpyDeviceToHost);
        printf(" riemann\n");
        printf(" ~~~\n");
        for (int i = 0; i < num_sides * n_p; i++) {
            if (i != 0 && i % num_sides == 0) {
                printf("   --- \n");
            }
            printf(" > (%f, %f) \n", left_rhs[i], right_rhs[i]);
        }
        free(left_rhs);
        free(right_rhs);
    }

    checkCudaError("error after stage 1: eval_riemann");

    eval_quad<<<n_blocks_elem, n_threads>>>
                    (d_c, d_quad_rhs, d_J,
                     d_xr, d_yr, d_xs, d_ys,
                     n_quad, n_p, num_elem);
    hipDeviceSynchronize();

    if (debug) {
        float *quad_rhs = (float *) malloc(num_elem * n_p * sizeof(float));
        hipMemcpy(quad_rhs, d_quad_rhs, num_elem * n_p * sizeof(float), hipMemcpyDeviceToHost);
        printf(" quad_rhs\n");
        printf(" ~~~\n");
        for (int i = 0; i < num_elem * n_p; i++) {
            if (i != 0 && i % num_elem == 0) {
                printf("   --- \n");
            }
            printf(" > %f \n", quad_rhs[i]);
            }
        free(quad_rhs);
    }

    eval_rhs<<<n_blocks_elem, n_threads>>>(d_k1, d_quad_rhs, d_left_riemann_rhs, d_right_riemann_rhs, 
                                          d_elem_s1, d_elem_s2, d_elem_s3, 
                                          d_left_elem, d_J, dt, n_p, num_sides, num_elem);
    hipDeviceSynchronize();

    if (debug) {
        float *rhs = (float *) malloc(num_elem * n_p * sizeof(float));
        hipMemcpy(rhs, d_k1, num_elem * n_p * sizeof(float), hipMemcpyDeviceToHost);
        printf(" eval_rhs\n");
        printf(" ~~~\n");
        for (int i = 0; i < num_elem * n_p; i++) {
            if (i != 0 && i % num_elem == 0) {
                printf("   --- \n");
            }
            printf(" > %f \n", rhs[i]);
            }
        free(rhs);
    }

    rk4_tempstorage<<<n_blocks_elem, n_threads>>>(d_c, d_kstar, d_k1, 0.5, n_p, num_elem);
    hipDeviceSynchronize();

    checkCudaError("error after stage 1.");

    // stage 2
    eval_riemann<<<n_blocks_sides, n_threads>>>
                    (d_kstar, d_left_riemann_rhs, d_right_riemann_rhs, d_J, 
                     d_s_length,
                     d_s_r,
                     d_V1x, d_V1y,
                     d_V2x, d_V2y,
                     d_V3x, d_V3y,
                     d_left_elem, d_right_elem,
                     d_left_side_number, d_right_side_number,
                     d_Nx, d_Ny, 
                     n_quad1d, n_p, num_sides, num_elem);
    hipDeviceSynchronize();

    eval_quad<<<n_blocks_elem, n_threads>>>
                    (d_c, d_quad_rhs, d_J,
                     d_xr, d_yr, d_xs, d_ys,
                     n_quad, n_p, num_elem);
    hipDeviceSynchronize();

    eval_rhs<<<n_blocks_elem, n_threads>>>(d_k2, d_quad_rhs, d_left_riemann_rhs, d_right_riemann_rhs,
                                          d_elem_s1, d_elem_s2, d_elem_s3, 
                                          d_left_elem, d_J, dt, n_p, num_sides, num_elem);
    hipDeviceSynchronize();

    rk4_tempstorage<<<n_blocks_elem, n_threads>>>(d_c, d_kstar, d_k2, 0.5, n_p, num_elem);
    hipDeviceSynchronize();

    checkCudaError("error after stage 2.");

    // stage 3
    eval_riemann<<<n_blocks_sides, n_threads>>>
                    (d_kstar, d_left_riemann_rhs, d_right_riemann_rhs, d_J, 
                     d_s_length,
                     d_s_r,
                     d_V1x, d_V1y,
                     d_V2x, d_V2y,
                     d_V3x, d_V3y,
                     d_left_elem, d_right_elem,
                     d_left_side_number, d_right_side_number,
                     d_Nx, d_Ny, 
                     n_quad1d, n_p, num_sides, num_elem);
    hipDeviceSynchronize();

    eval_quad<<<n_blocks_elem, n_threads>>>
                    (d_c, d_quad_rhs, d_J,
                     d_xr, d_yr, d_xs, d_ys,
                     n_quad, n_p, num_elem);
    hipDeviceSynchronize();

    eval_rhs<<<n_blocks_elem, n_threads>>>(d_k3, d_quad_rhs, d_left_riemann_rhs, d_right_riemann_rhs, 
                                          d_elem_s1, d_elem_s2, d_elem_s3, 
                                          d_left_elem, d_J, dt, n_p, num_sides, num_elem);
    hipDeviceSynchronize();

    rk4_tempstorage<<<n_blocks_elem, n_threads>>>(d_c, d_kstar, d_k3, 1.0, n_p, num_elem);
    hipDeviceSynchronize();

    checkCudaError("error after stage 3.");

    // stage 4
    eval_riemann<<<n_blocks_sides, n_threads>>>
                    (d_kstar, d_left_riemann_rhs, d_right_riemann_rhs, d_J, 
                     d_s_length,
                     d_s_r,
                     d_V1x, d_V1y,
                     d_V2x, d_V2y,
                     d_V3x, d_V3y,
                     d_left_elem, d_right_elem,
                     d_left_side_number, d_right_side_number,
                     d_Nx, d_Ny, 
                     n_quad1d, n_p, num_sides, num_elem);
    hipDeviceSynchronize();

    eval_quad<<<n_blocks_elem, n_threads>>>
                    (d_c, d_quad_rhs, d_J,
                     d_xr, d_yr, d_xs, d_ys,
                     n_quad, n_p, num_elem);
    hipDeviceSynchronize();

    eval_rhs<<<n_blocks_elem, n_threads>>>(d_k4, d_quad_rhs, d_left_riemann_rhs, d_right_riemann_rhs, 
                                          d_elem_s1, d_elem_s2, d_elem_s3, 
                                          d_left_elem, d_J, dt, n_p, num_sides, num_elem);
    hipDeviceSynchronize();

    checkCudaError("error after stage 4.");
    
    // final stage
    rk4<<<n_blocks_elem, n_threads>>>(d_c, d_k1, d_k2, d_k3, d_k4, n_p, num_elem);
    hipDeviceSynchronize();

    checkCudaError("error after final stage.");
}

void init_gpu(int num_elem, int num_sides, int n_p,
              float *V1x, float *V1y, 
              float *V2x, float *V2y, 
              float *V3x, float *V3y, 
              int *left_side_number, int *right_side_number,
              float *sides_x1, float *sides_y1,
              float *sides_x2, float *sides_y2,
              int *elem_s1, int *elem_s2, int *elem_s3,
              int *left_elem, int *right_elem) {
    checkCudaError("error before init.");
    hipDeviceReset();

    // allocate allllllllllll the memory.
    // TODO: this takes a really really long time on valor.
    hipMalloc((void **) &d_c,        num_elem * n_p * sizeof(float));
    hipMalloc((void **) &d_quad_rhs, num_elem * n_p * sizeof(float));
    hipMalloc((void **) &d_left_riemann_rhs,  num_sides * n_p * sizeof(float));
    hipMalloc((void **) &d_right_riemann_rhs, num_sides * n_p * sizeof(float));

    hipMalloc((void **) &d_kstar, num_elem * n_p * sizeof(float));
    hipMalloc((void **) &d_k1, num_elem * n_p * sizeof(float));
    hipMalloc((void **) &d_k2, num_elem * n_p * sizeof(float));
    hipMalloc((void **) &d_k3, num_elem * n_p * sizeof(float));
    hipMalloc((void **) &d_k4, num_elem * n_p * sizeof(float));

    hipMalloc((void **) &d_J, num_elem * sizeof(float));
    hipMalloc((void **) &d_s_length, num_sides * sizeof(float));

    hipMalloc((void **) &d_s_V1x, num_sides * sizeof(float));
    hipMalloc((void **) &d_s_V2x, num_sides * sizeof(float));
    hipMalloc((void **) &d_s_V1y, num_sides * sizeof(float));
    hipMalloc((void **) &d_s_V2y, num_sides * sizeof(float));

    hipMalloc((void **) &d_elem_s1, num_elem * sizeof(int));
    hipMalloc((void **) &d_elem_s2, num_elem * sizeof(int));
    hipMalloc((void **) &d_elem_s3, num_elem * sizeof(int));

    hipMalloc((void **) &d_Uv1, num_elem * sizeof(float));
    hipMalloc((void **) &d_Uv2, num_elem * sizeof(float));
    hipMalloc((void **) &d_Uv3, num_elem * sizeof(float));

    hipMalloc((void **) &d_V1x, num_elem * sizeof(float));
    hipMalloc((void **) &d_V1y, num_elem * sizeof(float));
    hipMalloc((void **) &d_V2x, num_elem * sizeof(float));
    hipMalloc((void **) &d_V2y, num_elem * sizeof(float));
    hipMalloc((void **) &d_V3x, num_elem * sizeof(float));
    hipMalloc((void **) &d_V3y, num_elem * sizeof(float));

    hipMalloc((void **) &d_xr, num_elem * sizeof(float));
    hipMalloc((void **) &d_yr, num_elem * sizeof(float));
    hipMalloc((void **) &d_xs, num_elem * sizeof(float));
    hipMalloc((void **) &d_ys, num_elem * sizeof(float));

    hipMalloc((void **) &d_s_r, n_p * sizeof(float));
    
    hipMalloc((void **) &d_left_side_number , num_sides * sizeof(int));
    hipMalloc((void **) &d_right_side_number, num_sides * sizeof(int));

    hipMalloc((void **) &d_Nx, num_sides * sizeof(float));
    hipMalloc((void **) &d_Ny, num_sides * sizeof(float));

    hipMalloc((void **) &d_right_elem, num_sides * sizeof(int));
    hipMalloc((void **) &d_left_elem , num_sides * sizeof(int));

    // set d_c to 0 not necessary
    //hipMemset(d_c, 0., num_elem * n_p * sizeof(float));
    hipMemset(d_quad_rhs, 0., num_elem * n_p * sizeof(float));

    // copy over data
    hipMemcpy(d_s_V1x, sides_x1, num_sides * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_s_V1y, sides_y1, num_sides * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_s_V2x, sides_x2, num_sides * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_s_V2y, sides_y2, num_sides * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_left_side_number , left_side_number , num_sides * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_right_side_number, right_side_number, num_sides * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_elem_s1, elem_s1, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_elem_s2, elem_s2, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_elem_s3, elem_s3, num_elem * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_V1x, V1x, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V1y, V1y, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V2x, V2x, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V2y, V2y, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V3x, V3x, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V3y, V3y, num_elem * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_left_elem , left_elem , num_sides * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_right_elem, right_elem, num_sides * sizeof(float), hipMemcpyHostToDevice);
}

void free_gpu() {
    hipFree(d_c);
    hipFree(d_quad_rhs);
    hipFree(d_left_riemann_rhs);
    hipFree(d_right_riemann_rhs);

    hipFree(d_kstar);
    hipFree(d_k1);
    hipFree(d_k2);
    hipFree(d_k3);
    hipFree(d_k4);

    hipFree(d_J);
    hipFree(d_s_length);

    hipFree(d_s_V1x);
    hipFree(d_s_V2x);
    hipFree(d_s_V1y);
    hipFree(d_s_V2y);

    hipFree(d_elem_s1);
    hipFree(d_elem_s2);
    hipFree(d_elem_s3);

    hipFree(d_Uv1);
    hipFree(d_Uv2);
    hipFree(d_Uv3);

    hipFree(d_V1x);
    hipFree(d_V1y);
    hipFree(d_V2x);
    hipFree(d_V2y);
    hipFree(d_V3x);
    hipFree(d_V3y);

    hipFree(d_xr);
    hipFree(d_yr);
    hipFree(d_xs);
    hipFree(d_ys);

    hipFree(d_s_r);
    
    hipFree(d_left_side_number);
    hipFree(d_right_side_number);

    hipFree(d_Nx);
    hipFree(d_Ny);

    hipFree(d_right_elem);
    hipFree(d_left_elem);
}

void usage_error() {
    printf("\nUsage: dgcuda [OPTIONS] [MESH] [OUTFILE]\n");
    printf(" Options: [-n] Order of polynomial approximation.\n");
    printf("          [-t] Number of timesteps.\n");
    printf("          [-d] Debug.\n");
}

int get_input(int argc, char *argv[],
               int *n, int *debug, int *timesteps,
               char **mesh_filename, char **out_filename) {

    int i;

    *timesteps = 1;
    *debug     = 0;
    // read command line input
    if (argc < 5) {
        usage_error();
        return 1;
    }
    for (i = 0; i < argc; i++) {
        // order of polynomial
        if (strcmp(argv[i], "-n") == 0) {
            if (i + 1 < argc) {
                *n = atoi(argv[i+1]);
                if (*n < 0 || *n > 8) {
                    usage_error();
                    return 1;
                }
            } else {
                usage_error();
                return 1;
            }
        }
        if (strcmp(argv[i], "-t") == 0) {
            if (i + 1 < argc) {
                *timesteps = atoi(argv[i+1]);
                if (*timesteps < 0) {
                    usage_error();
                    return 1;
                }
            } else {
                usage_error();
                return 1;
            }
        }
        if (strcmp(argv[i], "-d") == 0) {
            *debug = 1;
        }
    } 

    // second last argument is filename
    *mesh_filename = argv[argc - 2];
    // last argument is outfilename
    *out_filename  = argv[argc - 1];

    return 0;
}

int main(int argc, char *argv[]) {
    checkCudaError("error before start.");
    int num_elem, num_sides;
    int n_threads, n_blocks_elem, n_blocks_sides;
    int i, n, n_p, t, timesteps, n_quad, n_quad1d;
    int debug;

    float dt; 
    float *V1x, *V1y, *V2x, *V2y, *V3x, *V3y;
    float *sides_x1, *sides_x2;
    float *sides_y1, *sides_y2;

    float *r1_local, *r2_local, *w_local;

    float *s_r, *oned_w_local;

    int *left_elem, *right_elem;
    int *elem_s1, *elem_s2, *elem_s3;
    int *left_side_number, *right_side_number;

    FILE *mesh_file, *out_file;

    char line[100];
    char *mesh_filename;
    char *out_filename;

    float *Uv1, *Uv2, *Uv3;

    // get input 
    if (get_input(argc, argv, &n, &debug, &timesteps, &mesh_filename, &out_filename)) {
        return 1;
    }

    // set the order of the approximation & timestep
    n_p = (n + 1) * (n + 2) / 2;
    dt  = 0.01;

    // open the mesh to get num_elem for allocations
    mesh_file = fopen(mesh_filename, "r");
    out_file  = fopen(out_filename , "w");
    if (!mesh_file) {
        printf("\nERROR: mesh file not found.\n");
        return 1;
    }
    fgets(line, 100, mesh_file);
    sscanf(line, "%i", &num_elem);

    // allocate vertex points
    V1x = (float *) malloc(num_elem * sizeof(float));
    V1y = (float *) malloc(num_elem * sizeof(float));
    V2x = (float *) malloc(num_elem * sizeof(float));
    V2y = (float *) malloc(num_elem * sizeof(float));
    V3x = (float *) malloc(num_elem * sizeof(float));
    V3y = (float *) malloc(num_elem * sizeof(float));

    elem_s1 = (int *) malloc(num_elem * sizeof(int));
    elem_s2 = (int *) malloc(num_elem * sizeof(int));
    elem_s3 = (int *) malloc(num_elem * sizeof(int));

    // TODO: these are too big; should be a way to figure out how many we actually need
    left_side_number  = (int *)   malloc(3*num_elem * sizeof(int));
    right_side_number = (int *)   malloc(3*num_elem * sizeof(int));

    sides_x1    = (float *) malloc(3*num_elem * sizeof(float));
    sides_x2    = (float *) malloc(3*num_elem * sizeof(float));
    sides_y1    = (float *) malloc(3*num_elem * sizeof(float));
    sides_y2    = (float *) malloc(3*num_elem * sizeof(float)); 
    left_elem   = (int *) malloc(3*num_elem * sizeof(int));
    right_elem  = (int *) malloc(3*num_elem * sizeof(int));

    for (i = 0; i < 3*num_elem; i++) {
        right_elem[i] = -1;
    }

    // read in the mesh and make all the mappings
    read_mesh(mesh_file, &num_sides, num_elem,
                         V1x, V1y, V2x, V2y, V3x, V3y,
                         left_side_number, right_side_number,
                         sides_x1, sides_y1, 
                         sides_x2, sides_y2, 
                         elem_s1, elem_s2, elem_s3,
                         left_elem, right_elem);

    // close the file
    fclose(mesh_file);

    // initialize the gpu
    init_gpu(num_elem, num_sides, n_p,
             V1x, V1y, V2x, V2y, V3x, V3y,
             left_side_number, right_side_number,
             sides_x1, sides_y1,
             sides_x2, sides_y2, 
             elem_s1, elem_s2, elem_s3,
             left_elem, right_elem);

    n_threads        = 128;
    n_blocks_elem    = (num_elem  / n_threads) + ((num_elem  % n_threads) ? 1 : 0);
    n_blocks_sides   = (num_sides / n_threads) + ((num_sides % n_threads) ? 1 : 0);

    // pre computations
    preval_jacobian<<<n_blocks_elem, n_threads>>>(d_J, d_V1x, d_V1y, d_V2x, d_V2y, d_V3x, d_V3y, num_elem); 
    hipDeviceSynchronize();
    preval_side_length<<<n_blocks_sides, n_threads>>>(d_s_length, d_s_V1x, d_s_V1y, d_s_V2x, d_s_V2y, 
                                                      num_sides); 
    hipDeviceSynchronize();
    preval_normals<<<n_blocks_sides, n_threads>>>(d_Nx, d_Ny, 
                                                  d_s_V1x, d_s_V1y, d_s_V2x, d_s_V2y,
                                                  d_V1x, d_V1y, 
                                                  d_V2x, d_V2y, 
                                                  d_V3x, d_V3y, 
                                                  d_left_elem, d_left_side_number, num_sides); 
    hipDeviceSynchronize();
    preval_normals_direction<<<n_blocks_sides, n_threads>>>(d_Nx, d_Ny, 
                                                  d_V1x, d_V1y, 
                                                  d_V2x, d_V2y, 
                                                  d_V3x, d_V3y, 
                                                  d_left_elem, d_left_side_number, num_sides); 
    preval_partials<<<n_blocks_elem, n_threads>>>(d_V1x, d_V1y,
                                                  d_V2x, d_V2y,
                                                  d_V3x, d_V3y,
                                                  d_xr,  d_yr,
                                                  d_xs,  d_ys, num_elem);
    hipDeviceSynchronize();
    checkCudaError("error after prevals.");

    // get the correct quadrature rules for this scheme
    set_quadrature(n, &r1_local, &r2_local, &w_local, 
                   &s_r, &oned_w_local, &n_quad, &n_quad1d);

    // evaluate the basis functions at those points and store on GPU
    preval_basis(r1_local, r2_local, s_r, w_local, oned_w_local, n_quad, n_quad1d, n_p);

    // initial conditions
    init_conditions<<<n_blocks_elem, n_threads>>>(d_c, d_V1x, d_V1y, d_V2x, d_V2y, d_V3x, d_V3y,
                    n_quad, n_p, num_elem);
    checkCudaError("error after initial conditions.");

    printf("Computing...\n");
    printf(" ? %i degree polynomial interpolation (n_p = %i)\n", n, n_p);
    printf(" ? %i precomputed basis points\n", n_quad * n_p);
    printf(" ? %i elements\n", num_elem);
    printf(" ? %i sides\n", num_sides);
    printf(" ? %i timesteps\n", timesteps);
    printf(" ? 1d quadrature rules:\n");
    for (i = 0; i < n_quad1d; i++) {
        printf("     > %f - %f \n", s_r[i], oned_w_local[i]);
    }
    printf(" ? 2d quadrature rules:\n");
    for (i = 0; i < n_quad; i++) {
        printf("     > (%f, %f) - %f \n", r1_local[i], r2_local[i], w_local[i]);
    }

    if (debug) {
        float *Nx = (float *) malloc(num_sides * sizeof(float));
        float *Ny = (float *) malloc(num_sides * sizeof(float));

        hipMemcpy(Nx, d_Nx, num_sides *sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(Ny, d_Ny, num_sides *sizeof(float), hipMemcpyDeviceToHost);

        printf(" ? normals\n");
        for (i = 0; i < num_sides; i++) {
            printf("    > (%f, %f) \n", Nx[i], Ny[i]);
        }
        free(Nx);
        free(Ny);
    }


    checkCudaError("error before time integration.");
    fprintf(out_file, "View \"Exported field \" {\n");
    for (t = 0; t < timesteps; t++) {
        // time integration
        time_integrate(dt, n_quad, n_quad1d, n_p, num_elem, num_sides, debug);
    }

    if (debug) {
        float *c = (float *) malloc(num_elem * n_p * sizeof(float));
        hipMemcpy(c, d_c, num_elem * n_p * sizeof(float), hipMemcpyDeviceToHost);
        printf(" c\n");
        printf(" ~~~\n");
        for (i = 0; i < num_elem * n_p; i++) {
            if (i != 0 && i % num_elem == 0) {
                printf("   --- \n");
            }
            printf(" > %f\n", c[i]);
        }
        free(c);
    }

    // evaluate at the vertex points and copy over data
    Uv1 = (float *) malloc(num_elem * sizeof(float));
    Uv2 = (float *) malloc(num_elem * sizeof(float));
    Uv3 = (float *) malloc(num_elem * sizeof(float));
    eval_u<<<n_blocks_elem, n_threads>>>(d_c, d_V1x, d_V1y, d_V2x, d_V2y, d_V3x, d_V3y, 
                                             d_Uv1, d_Uv2, d_Uv3, num_elem, n_p);
    hipDeviceSynchronize();
    hipMemcpy(Uv1, d_Uv1, num_elem * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Uv2, d_Uv2, num_elem * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Uv3, d_Uv3, num_elem * sizeof(float), hipMemcpyDeviceToHost);

    // write data to file
    // TODO: this will output multiple vertices values. does gmsh care? i dunno...
    for (i = 0; i < num_elem; i++) {
        fprintf(out_file, "ST (%f,%f,0,%f,%f,0,%f,%f,0) {%f,%f,%f};\n", 
                               V1x[i], V1y[i], V2x[i], V2y[i], V3x[i], V3y[i],
                               Uv1[i], Uv2[i], Uv3[i]);
    }

    fprintf(out_file,"};");

    // close the output file
    fclose(out_file);

    // free variables
    free_gpu();
    
    free(Uv1);
    free(Uv2);
    free(Uv3);

    free(V1x);
    free(V1y);
    free(V2x);
    free(V2y);
    free(V3x);
    free(V3y);

    free(elem_s1);
    free(elem_s2);
    free(elem_s3);

    free(sides_x1);
    free(sides_x2);
    free(sides_y1);
    free(sides_y2);

    free(left_elem);
    free(right_elem);
    free(left_side_number);
    free(right_side_number);

    free(r1_local);
    free(r2_local);
    free(w_local);
    free(s_r);
    free(oned_w_local);

    return 0;
}
