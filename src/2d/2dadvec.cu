#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "2dadvec_kernels.cu"
/* 2dadvec.cu
 * 
 * This file calls the kernels in 2dadvec_kernels.cu for the 2D advection
 * DG method.
 */

void set_quadrature(int p, int *num_quad_oned, int *num_quad_twod,
                    float *r1, float *r2, float *w,
                    float *s1_r1, *s1_r2,
                    float *s2_r1, *s2_r2,
                    float *s3_r1, *s3_r2,
                    float *oned_r, float *oned_w) {
    switch (p) {
        case 0:
            // set number of 1d and 2d quadrature points
            *num_quad_oned= 1;
            *num_quad_twod = 1;

            // set 2d integration points
            r1[0] = 0.333333333333333;
            r2[0] = 0.333333333333333;
            w[0]  = 1.0;

            // set 1d integration points


            break;
        case 2:
            r1[0] = 0.166666666666666;
            r2[0] = 0.166666666666666;
            w[0]  = 0.333333333333333;
            r1[1] = 0.666666666666666;
            r2[1] = 0.166666666666666;
            w[1]  = 0.333333333333333;
            r1[2] = 0.166666666666666;
            r2[2] = 0.666666666666666;
            w[2]  = 0.333333333333333;
            break;
    }
}
/*             
  { {0.333333333333333,0.3333333333333333},-0.5625},
  { {0.6,0.2},.520833333333333 },
  { {0.2,0.6},.520833333333333 },
  { {0.2,0.2},.520833333333333 }
};

IntPt2d GQT4[6] = {
  { {0.816847572980459,0.091576213509771},0.109951743655322},
  { {0.091576213509771,0.816847572980459},0.109951743655322},
  { {0.091576213509771,0.091576213509771},0.109951743655322},
  { {0.108103018168070,0.445948490915965},0.223381589678011},
  { {0.445948490915965,0.108103018168070},0.223381589678011},
  { {0.445948490915965,0.445948490915965},0.223381589678011}
};

IntPt2d GQT5[7] = {
  { {0.333333333333333,0.333333333333333},0.225000000000000},
  { {0.797426985353087,0.101286507323456},0.125939180544827},
  { {0.101286507323456,0.797426985353087},0.125939180544827},
  { {0.101286507323456,0.101286507323456},0.125939180544827},
  { {0.470142064105115,0.059715871789770},0.132394152788506},
  { {0.059715871789770,0.470142064105115},0.132394152788506},
  { {0.470142064105115,0.470142064105115},0.132394152788506}
};

IntPt2d GQT6[12] = {
  { {0.873821971016996,0.063089014491502},0.050844906370207},
  { {0.063089014491502,0.873821971016996},0.050844906370207},
  { {0.063089014491502,0.063089014491502},0.050844906370207},
  { {0.501426509658179,0.249286745170910},0.116786275726379},
  { {0.249286745170910,0.501426509658179},0.116786275726379},
  { {0.249286745170910,0.249286745170910},0.116786275726379},
  { {0.636502499121399,0.310352451033784},0.082851075618374},
  { {0.310352451033784,0.636502499121399},0.082851075618374},
  { {0.636502499121399,0.053145049844816},0.082851075618374},
  { {0.310352451033784,0.053145049844816},0.082851075618374},
  { {0.053145049844816,0.310352451033785},0.082851075618374},
  { {0.053145049844816,0.636502499121399},0.082851075618374}
};

IntPt2d GQT7[13] = {
  { {0.333333333333333,0.333333333333333},-0.149570044467682},
  { {0.479308067841920,0.260345966079040},0.175615257433208},
  { {0.260345966079040,0.479308067841920},0.175615257433208},
  { {0.260345966079040,0.260345966079040},0.175615257433208},
  { {0.869739794195568,0.065130102902216},0.053347235608838},
  { {0.065130102902216,0.869739794195568},0.053347235608838},
  { {0.065130102902216,0.065130102902216},0.053347235608838},
  { {0.048690315425316,0.312865496004874},0.077113760890257},
  { {0.312865496004874,0.048690315425316},0.077113760890257},
  { {0.638444188569810,0.048690315425316},0.077113760890257},
  { {0.048690315425316,0.638444188569810},0.077113760890257},
  { {0.312865496004874,0.638444188569810},0.077113760890257},
  { {0.638444188569810,0.312865496004874},0.077113760890257}

};

IntPt2d GQT8[16] = {
  { {0.333333333333333,0.333333333333333},0.144315607677787},
  { {0.081414823414554,0.459292588292723},0.095091634267285},
  { {0.459292588292723,0.081414823414554},0.095091634267285},
  { {0.459292588292723,0.459292588292723},0.095091634267285},
  { {0.658861384496480,0.170569307751760},0.103217370534718},
  { {0.170569307751760,0.658861384496480},0.103217370534718},
  { {0.170569307751760,0.170569307751760},0.103217370534718},
  { {0.898905543365938,0.050547228317031},0.032458497623198},
  { {0.050547228317031,0.898905543365938},0.032458497623198},
  { {0.050547228317031,0.050547228317031},0.032458497623198},  
  { {0.008394777409958,0.728492392955404},0.027230314174435},
  { {0.728492392955404,0.008394777409958},0.027230314174435},
  { {0.263112829634638,0.008394777409958},0.027230314174435},
  { {0.008394777409958,0.263112829634638},0.027230314174435},
  { {0.263112829634638,0.728492392955404},0.027230314174435},
  { {0.728492392955404,0.263112829634638},0.027230314174435}
};

IntPt2d GQT9[19] = {
  { {0.333333333333333,0.333333333333333},0.097135796282799},
  { {0.020634961602525,0.489682519198738},0.031334700227139},
  { {0.489682519198738,0.020634961602525},0.031334700227139},
  { {0.489682519198738,0.489682519198738},0.031334700227139},
  { {0.125820817014127,0.437089591492937},0.077827541004774},
  { {0.437089591492937,0.125820817014127},0.077827541004774},
  { {0.437089591492937,0.437089591492937},0.077827541004774},
  { {0.623592928761935,0.188203535619033},0.079647738927210},
  { {0.188203535619033,0.623592928761935},0.079647738927210},
  { {0.188203535619033,0.188203535619033},0.079647738927210},
  { {0.910540973211095,0.044729513394453},0.025577675658698},
  { {0.044729513394453,0.910540973211095},0.025577675658698},
  { {0.044729513394453,0.044729513394453},0.025577675658698},
  { {0.036838412054736,0.221962989160766},0.043283539377289},
  { {0.221962989160766,0.036838412054736},0.043283539377289},
  { {0.036838412054736,0.741198598784498},0.043283539377289},
  { {0.741198598784498,0.036838412054736},0.043283539377289},
  { {0.741198598784498,0.221962989160766},0.043283539377289},
  { {0.221962989160766,0.741198598784498},0.043283539377289}
};

IntPt2d GQT10[25] = {
  { {0.333333333333333,0.333333333333333},0.090817990382754},
  { {0.028844733232685,0.485577633383657},0.036725957756467},
  { {0.485577633383657,0.028844733232685},0.036725957756467},
  { {0.485577633383657,0.485577633383657},0.036725957756467},
  { {0.781036849029926,0.109481575485037},0.045321059435528},
  { {0.109481575485037,0.781036849029926},0.045321059435528},
  { {0.109481575485037,0.109481575485037},0.045321059435528},
  { {0.141707219414880,0.307939838764121},0.072757916845420},
  { {0.307939838764121,0.141707219414880},0.072757916845420},
  { {0.307939838764121,0.550352941820999},0.072757916845420},
  { {0.550352941820999,0.307939838764121},0.072757916845420},
  { {0.550352941820999,0.141707219414880},0.072757916845420},
  { {0.141707219414880,0.550352941820999},0.072757916845420},
  { {0.025003534762686,0.246672560639903},0.028327242531057},
  { {0.246672560639903,0.025003534762686},0.028327242531057},
  { {0.025003534762686,0.728323904597411},0.028327242531057},
  { {0.728323904597411,0.025003534762686},0.028327242531057},
  { {0.728323904597411,0.246672560639903},0.028327242531057},
  { {0.246672560639903,0.728323904597411},0.028327242531057},
  { {0.009540815400299,0.066803251012200},0.009421666963733},
  { {0.066803251012200,0.009540815400299},0.009421666963733},
  { {0.066803251012200,0.923655933587500},0.009421666963733},
  { {0.923655933587500,0.066803251012200},0.009421666963733},
  { {0.923655933587500,0.009540815400299},0.009421666963733},
  { {0.009540815400299,0.923655933587500},0.009421666963733}
};
*/
void checkCudaError(const char *message)
{
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
        exit(-1);
    }
}

void read_mesh(FILE *mesh_file, 
              int *num_sides,
              float *V1x, float *V1y,
              float *V2x, float *V2y,
              float *V3x, float *V3y,
              int *side_number,
              float *sides_x1, float *sides_y1,
              float *sides_x2, float *sides_y2,
              float *elem_s1,  float *elem_s2, float *elem_s3,
              float *left_elem, float *right_elem) {

    int i, j, s1, s2, s3, numsides;
    char line[100];
    i = 0;
    numsides = 0;
    while(fgets(line, sizeof(line), mesh_file) != NULL) {
        // these three vertices define the element
        sscanf(line, "%f %f %f %f %f %f", &V1x[i], &V1y[i], &V2x[i], &V2y[i], &V3x[i], &V3y[i]);

        // determine whether we should add these three sides or not
        s1 = 1;
        s2 = 1;
        s3 = 1;

        // scan through the existing sides to see if we already added it
        // TODO: yeah, there's a better way to do this.
        for (j = 0; j < numsides; j++) {
            if ((sides_x1[j] == V1x[i] && sides_y1[j] == V1y[i]
             && sides_x2[j] == V2x[i] && sides_y2[j] == V2y[i]) 
            || (sides_x2[j] == V1x[i] && sides_y2[j] == V1y[i]
             && sides_x1[j] == V2x[i] && sides_y1[j] == V2y[i])) {
                s1 = 0;
                // link this element to that side
                elem_s1[i] = numsides;
                side_number[numsides] = 1;
                // and that side to this element either by left or right sided
                // if there's no left element, make this the left element otherwise, 
                // make this a right element 

                // if left element is not set, make this the left element
                if (left_elem[numsides] != -1) {
                    left_elem[numsides] = i; // something like this
                } else if (right_elem[numsides] != -1) {
                    left_elem[numsides] = i; // something like this
                }
            }
            if ((sides_x1[j] == V2x[i] && sides_y1[j] == V2y[i]
             && sides_x2[j] == V3x[i] && sides_y2[j] == V3y[i]) 
            || (sides_x2[j] == V2x[i] && sides_y2[j] == V2y[i]
             && sides_x1[j] == V3x[i] && sides_y1[j] == V3y[i])) {
                s2 = 0;
                // link this element to that side
                elem_s2[i] = numsides;
                side_number[numsides] = 2;
            }
            if ((sides_x1[j] == V2x[i] && sides_y1[j] == V2y[i]
             && sides_x2[j] == V3x[i] && sides_y2[j] == V3y[i]) 
            || (sides_x2[j] == V2x[i] && sides_y2[j] == V2y[i]
             && sides_x1[j] == V3x[i] && sides_y1[j] == V3y[i])) {
                s3 = 0;
                // link this element to that side
                elem_s3[i] = numsides;
                side_number[numsides] = 3;
            }
        }
        // if we haven't added the side already, add it
        if (s1) {
            sides_x1[numsides] = V1x[i];
            sides_y1[numsides] = V1y[i];
            sides_x2[numsides] = V2x[i];
            sides_y2[numsides] = V2y[i];
            
            // link the added side to this element
            elem_s1[i] = numsides;
            side_number[numsides] = 1;

            // if left element is not set, make this the left element
            if (left_elem[numsides] != -1) {
                left_elem[numsides] = i;
            } else {
                right_elem[numsides] = i;
            }
            numsides++;
        }
        if (s2) {
            sides_x1[numsides] = V2x[i];
            sides_y1[numsides] = V2y[i];
            sides_x2[numsides] = V3x[i];
            sides_y2[numsides] = V3y[i];

            // link the added side to this element
            elem_s2[i] = numsides;
            side_number[numsides] = 2;

            // if left element is not set, make this the left element
            if (left_elem[numsides] != -1) {
                left_elem[numsides] = i;
            } else {
                right_elem[numsides] = i;
            }
            numsides++;
        }
        if (s3) {
            sides_x1[numsides] = V3x[i];
            sides_y1[numsides] = V3y[i];
            sides_x2[numsides] = V1x[i];
            sides_y2[numsides] = V1y[i];

            // link the added side to this element
            elem_s3[i] = numsides;
            side_number[numsides] = 3;

            // if left element is not set, make this the left element
            if (left_elem[numsides] != -1) {
                left_elem[numsides] = i;
            } else {
                right_elem[numsides] = i;
            }
            numsides++;
        }
        i++;
    }

    *num_sides = numsides;
}

void time_integrate(float *c, int n_p, int num_elem, int num_sides) {
    int n_threads = 128;

    int n_blocks_quad    = (num_elem  / n_threads) + ((num_elem  % n_threads) ? 1 : 0);
    int n_blocks_riemann = (num_sides / n_threads) + ((num_sides % n_threads) ? 1 : 0);
    int n_blocks_rhs     = (((n_p + 1) * num_elem ) / n_threads) 
                           + (((n_p + 1) * num_elem % n_threads) ? 1 : 0);

    // Stage 1
    eval_riemann<<<n_blocks_riemann, n_threads>>>
                    (d_c, d_rhs, d_s1_r1, d_s1_r2,
                     d_s2_r1, d_s2_r2,
                     d_s3_r1, d_s3_r2,
                     d_oned_r, d_oned_w, 
                     d_left_idx_list, d_right_idx_list,
                     d_side_number,
                     d_Nx, d_Ny, n_p, num_sides);
    hipDeviceSynchronize();
    checkCudaError("error after stage 1: eval_riemann");

    eval_quad<<<n_blocks_quad, n_threads>>>
                    (d_c, d_rhs, d_r1, d_r2, d_w, d_J, n_p, num_elem);
    hipDeviceSynchronize();
    checkCudaError("error after stage 1: eval_quad");

    eval_rhs<<<n_blocks_rhs, n_threads>>>(d_k1, d_rhs);
    hipDeviceSynchronize();
    checkCudaError("error after stage 1: eval_rhs");

    rk4_tempstorage<<<n_blocks_rhs, n_threads>>>(d_c, d_kstar, d_k1, 0.5, n_p, num_elem);
    hipDeviceSynchronize();
    checkCudaError("error after stage 1: rk4_temp");

    checkCudaError("error after stage 1.");

    // Stage 2
    eval_riemann<<<n_blocks_riemann, n_threads>>>
                    (d_kstar, d_rhs, d_s1_r1, d_s1_r2,
                     d_s2_r1, d_s2_r2,
                     d_s3_r1, d_s3_r2,
                     d_oned_r, d_oned_w, 
                     d_left_idx_list, d_right_idx_list,
                     d_side_number,
                     d_Nx, d_Ny, n_p, num_sides);
    hipDeviceSynchronize();

    eval_quad<<<n_blocks_quad, n_threads>>>
                    (d_kstar, d_rhs, d_r1, d_r2, d_w, d_J, n_p, num_elem);
    hipDeviceSynchronize();

    eval_rhs<<<n_blocks_rhs, n_threads>>>(d_k2, d_rhs);
    hipDeviceSynchronize();

    rk4_tempstorage<<<n_blocks_rhs, n_threads>>>(d_c, d_kstar, d_k2, 0.5, n_p, num_elem);
    hipDeviceSynchronize();

    checkCudaError("error after stage 2.");

    // Stage 3
    eval_riemann<<<n_blocks_riemann, n_threads>>>
                    (d_kstar, d_rhs, d_s1_r1, d_s1_r2,
                     d_s2_r1, d_s2_r2,
                     d_s3_r1, d_s3_r2,
                     d_oned_r, d_oned_w, 
                     d_left_idx_list, d_right_idx_list,
                     d_side_number,
                     d_Nx, d_Ny, n_p, num_sides);
    hipDeviceSynchronize();

    eval_quad<<<n_blocks_quad, n_threads>>>
                    (d_kstar, d_rhs, d_r1, d_r2, d_w, d_J, n_p, num_elem);
    hipDeviceSynchronize();

    eval_rhs<<<n_blocks_rhs, n_threads>>>(d_k3, d_rhs);
    hipDeviceSynchronize();

    rk4_tempstorage<<<n_blocks_rhs, n_threads>>>(d_c, d_kstar, d_k3, 1.0, n_p, num_elem);
    hipDeviceSynchronize();

    checkCudaError("error after stage 3.");

    // Stage 4
    eval_riemann<<<n_blocks_riemann, n_threads>>>
                    (d_kstar, d_rhs, d_s1_r1, d_s1_r2,
                     d_s2_r1, d_s2_r2,
                     d_s3_r1, d_s3_r2,
                     d_oned_r, d_oned_w, 
                     d_left_idx_list, d_right_idx_list,
                     d_side_number,
                     d_Nx, d_Ny, n_p, num_sides);
    hipDeviceSynchronize();

    eval_quad<<<n_blocks_quad, n_threads>>>
                    (d_kstar, d_rhs, d_r1, d_r2, d_w, d_J, n_p, num_elem);

    hipDeviceSynchronize();

    eval_rhs<<<n_blocks_rhs, n_threads>>>(d_k4, d_rhs);
    hipDeviceSynchronize();

    checkCudaError("error after stage 4.");
    
    // final stage
    rk4<<<n_blocks_rhs, n_threads>>>(d_c, d_k1, d_k2, d_k3, d_k4, n_p, num_elem);

    hipMemcpy(c, d_c, num_elem * (n_p + 1) * sizeof(float), hipMemcpyDeviceToHost);

    checkCudaError("error after final stage.");
}

void init_gpu(int num_elem, int num_sides, int n_p,
              float *V1x, float *V1y, 
              float *V2x, float *V2y, 
              float *V3x, float *V3y, 
              int *side_number,
              float *sides_x1, float *sides_y1,
              float *sides_x2, float *sides_y2,
              float *elem_s1, float *elem_s2, float *elem_s3,
              float *left_elem, float *right_elem) {
    checkCudaError("error before init.");
    hipDeviceReset();

    // allocate allllllllllll the memory.
    // TODO: this takes a really really long time.
    hipMalloc((void **) &d_c  , num_elem * (n_p + 1) * sizeof(float));
    hipMalloc((void **) &d_rhs, num_elem * (n_p + 1) * sizeof(float));

    hipMalloc((void **) &d_r1, (n_p + 1) * sizeof(float));
    hipMalloc((void **) &d_r2, (n_p + 1) * sizeof(float));
    hipMalloc((void **) &d_w , (n_p + 1) * sizeof(float));

    hipMalloc((void **) &d_oned_r, (n_p + 1) * sizeof(float));
    hipMalloc((void **) &d_oned_w, (n_p + 1) * sizeof(float));

    hipMalloc((void **) &d_J, num_elem * sizeof(float));
    hipMalloc((void **) &d_s_len, num_sides * sizeof(float));

    hipMalloc((void **) &d_s_V1x, num_sides * sizeof(float));
    hipMalloc((void **) &d_s_V2x, num_sides * sizeof(float));
    hipMalloc((void **) &d_s_V1y, num_sides * sizeof(float));
    hipMalloc((void **) &d_s_V2y, num_sides * sizeof(float));

    hipMalloc((void **) &d_elem_s1, num_elem * sizeof(int));
    hipMalloc((void **) &d_elem_s2, num_elem * sizeof(int));
    hipMalloc((void **) &d_elem_s3, num_elem * sizeof(int));

    hipMalloc((void **) &d_V1x, num_elem * sizeof(float));
    hipMalloc((void **) &d_V1y, num_elem * sizeof(float));
    hipMalloc((void **) &d_V2x, num_elem * sizeof(float));
    hipMalloc((void **) &d_V2y, num_elem * sizeof(float));
    hipMalloc((void **) &d_V3x, num_elem * sizeof(float));
    hipMalloc((void **) &d_V3y, num_elem * sizeof(float));

    hipMalloc((void **) &d_s1_r1, (n_p + 1) * sizeof(float));
    hipMalloc((void **) &d_s1_r2, (n_p + 1) * sizeof(float));
    hipMalloc((void **) &d_s2_r1, (n_p + 1) * sizeof(float));
    hipMalloc((void **) &d_s2_r2, (n_p + 1) * sizeof(float));
    hipMalloc((void **) &d_s3_r1, (n_p + 1) * sizeof(float));
    hipMalloc((void **) &d_s3_r2, (n_p + 1) * sizeof(float));
    
    hipMalloc((void **) &d_side_number, num_sides * sizeof(int));

    hipMalloc((void **) &d_Nx, num_sides * sizeof(float));
    hipMalloc((void **) &d_Ny, num_sides * sizeof(float));

    hipMalloc((void **) &d_right_idx_list, num_sides * sizeof(int));
    hipMalloc((void **) &d_left_idx_list , num_sides * sizeof(int));

    // set d_rhs to 0
    hipMemset(d_rhs, 0, num_elem * (n_p + 1) * sizeof(float));

    // copy over data
    hipMemcpy(d_s_V1x, sides_x1, num_sides * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_s_V1y, sides_y1, num_sides * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_s_V2x, sides_x2, num_sides * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_s_V2y, sides_y2, num_sides * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_side_number, side_number, num_elem * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_elem_s1, elem_s1, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_elem_s2, elem_s2, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_elem_s3, elem_s3, num_elem * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_V1x, V1x, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V1y, V1y, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V2x, V2x, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V2y, V2y, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V3x, V3x, num_elem * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V3y, V3y, num_elem * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(d_right_idx_list, right_elem, num_sides * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_left_idx_list , left_elem , num_sides * sizeof(float), hipMemcpyHostToDevice);
}

int main() {
    checkCudaError("error before start.");
    int num_elem, num_sides;
    int i, n_p;

    float *V1x, *V1y, *V2x, *V2y, *V3x, *V3y;

    int *side_number;
    float *sides_x1, *sides_x2;
    float *sides_y1, *sides_y2;

    float *left_elem, *right_elem;
    float *elem_s1, *elem_s2, *elem_s3;

    n_p = 0;

    FILE *mesh_file;
    mesh_file = fopen("supersimple.out", "r");

    // first line should be the number of elements
    char line[100];
    fgets(line, 100, mesh_file);
    sscanf(line, "%i", &num_elem);

    // allocate vertex points
    V1x = (float *) malloc(num_elem * sizeof(float));
    V1y = (float *) malloc(num_elem * sizeof(float));
    V2x = (float *) malloc(num_elem * sizeof(float));
    V2y = (float *) malloc(num_elem * sizeof(float));
    V3x = (float *) malloc(num_elem * sizeof(float));
    V3y = (float *) malloc(num_elem * sizeof(float));

    elem_s1 = (float *) malloc(num_elem * sizeof(float));
    elem_s2 = (float *) malloc(num_elem * sizeof(float));
    elem_s3 = (float *) malloc(num_elem * sizeof(float));

    // TODO: these are too big; should be a way to figure out how many we actually need
    side_number = (int *)   malloc(3*num_elem * sizeof(int));
    sides_x1    = (float *) malloc(3*num_elem * sizeof(float));
    sides_x2    = (float *) malloc(3*num_elem * sizeof(float));
    sides_y1    = (float *) malloc(3*num_elem * sizeof(float));
    sides_y2    = (float *) malloc(3*num_elem * sizeof(float)); 
    left_elem   = (float *) malloc(3*num_elem * sizeof(float));
    right_elem  = (float *) malloc(3*num_elem * sizeof(float));

    for (i = 0; i < num_elem; i++) {
        left_elem[i] = -1;
    }

    read_mesh(mesh_file, &num_sides,
                         V1x, V1y, V2x, V2y, V3x, V3y,
                         side_number,
                         sides_x1, sides_y1, 
                         sides_x2, sides_y2, 
                         elem_s1, elem_s2, elem_s3,
                         left_elem, right_elem);

    fclose(mesh_file);
    init_gpu(num_elem, num_sides, n_p,
             V1x, V1y, V2x, V2y, V3x, V3y,
             side_number,
             sides_x1, sides_y1,
             sides_x2, sides_y2, 
             elem_s1, elem_s2, elem_s3,
             left_elem, right_elem);
    
    // pre computations
    preval_side_length<<<1, num_sides>>>(d_s_len, d_s_V1x, d_s_V1y, d_s_V2x, d_s_V2y); 
    preval_jacobian<<<1, num_elem>>>(d_J, d_V1x, d_V1y, d_V2x, d_V2y, d_V3x, d_V3y); 

    // no longer need vertices stored on the GPU
    hipFree(d_V1x);
    hipFree(d_V1y);
    hipFree(d_V2x);
    hipFree(d_V2y);
    hipFree(d_V3x);
    hipFree(d_V3y);
    hipFree(d_s_V1x);
    hipFree(d_s_V1y);
    hipFree(d_s_V2x);
    hipFree(d_s_V2y);

    float *side_len = (float *)malloc(num_sides * sizeof(float));
    float *J = (float *)malloc(num_elem * sizeof(float));

    hipMemcpy(side_len, d_s_len, num_sides * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(J, d_J, num_elem * sizeof(float), hipMemcpyDeviceToHost);

    for (i = 0; i < num_sides; i++) {
        printf("side %i = %f\n", i, side_len[i]);
    }

    float sum = 0;
    for (i = 0; i < num_elem; i++) {
        printf("J %i = %f\n", i, J[i]);
        sum += J[i];
    }

    printf("total area = %f \n", sum);

    free(side_len);
    free(J);

    float dt = 0.001;
    int t;

    float *c = (float *) malloc(num_elem * (n_p + 1) * sizeof(float));
    for (i = 0; i < num_elem * (n_p + 1); i++) {
        c[i] = 1;
    }

    hipMemcpy(d_c, c, num_elem * (n_p + 1) * sizeof(float), hipMemcpyHostToDevice);

    float *r1 = (float *) malloc(1 * sizeof(float));
    float *r2 = (float *) malloc(1 * sizeof(float));
    float *w =  (float *) malloc(1 * sizeof(float));

    set_quadrature(n_p, r1, r2, w);

    checkCudaError("error before quadrature copy.");

    hipMemcpy(d_r1, r1, 1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_r2, r2, 1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w , w , 1 * sizeof(float), hipMemcpyHostToDevice);

    checkCudaError("error before time integration.");
    // time integration
    for (t = 0; t < 1; t++) {
        time_integrate(c, n_p, num_elem, num_sides);
        for (i = 0; i < num_elem; i++) {
            printf("%f \n", c[i]);
        }
    }

    // free up memory
    free(V1x);
    free(V1y);
    free(V2x);
    free(V2y);
    free(V3x);
    free(V3y);

    free(elem_s1);
    free(elem_s2);
    free(elem_s3);

    free(sides_x1);
    free(sides_x2);
    free(sides_y1);
    free(sides_y2);

    free(left_elem);
    free(right_elem);

    return 0;
}
