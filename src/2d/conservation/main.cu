#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "conserv.cu"

extern int local_N;
extern int limiter;
extern int time_integrator;

int run_dgcuda(int argc, char *argv[]) {
    checkCudaError("error before start.");
    int num_elem, num_sides;
    int n_threads, n_blocks_elem, n_blocks_sides;
    int i, n, n_p, timesteps, n_quad, n_quad1d;

    double endtime;
    double *min_radius;
    double min_r;
    double *V1x, *V1y, *V2x, *V2y, *V3x, *V3y;
    double *sides_x1, *sides_x2;
    double *sides_y1, *sides_y2;

    double *r1_local, *r2_local, *w_local;

    double *s_r, *oned_w_local;

    int *left_elem, *right_elem;
    int *elem_s1, *elem_s2, *elem_s3;
    int *left_side_number, *right_side_number;

    FILE *mesh_file, *out_file;

    char out_filename[100];
    char *mesh_filename;

    double *Uv1, *Uv2, *Uv3;
    double *error;

    // get input 
    endtime = -1;
    if (get_input(argc, argv, &n, &timesteps, &endtime, &mesh_filename)) {
        return 1;
    }

    // set the order of the approximation & timestep
    n_p = (n + 1) * (n + 2) / 2;

    // open the mesh to get num_elem for allocations
    mesh_file = fopen(mesh_filename, "r");
    if (!mesh_file) {
        printf("\nERROR: mesh file not found.\n");
        return 1;
    }

    // read in the mesh and make all the mappings
    read_mesh(mesh_file, &num_sides, &num_elem,
                         &V1x, &V1y, &V2x, &V2y, &V3x, &V3y,
                         &left_side_number, &right_side_number,
                         &sides_x1, &sides_y1, 
                         &sides_x2, &sides_y2, 
                         &elem_s1, &elem_s2, &elem_s3,
                         &left_elem, &right_elem);

                
    // close the file
    fclose(mesh_file);

    // initialize the gpu
    init_gpu(num_elem, num_sides, n_p,
             V1x, V1y, V2x, V2y, V3x, V3y,
             left_side_number, right_side_number,
             sides_x1, sides_y1,
             sides_x2, sides_y2, 
             elem_s1, elem_s2, elem_s3,
             left_elem, right_elem);

    // set constant data
    set_N(local_N);

    checkCudaError("error after gpu init.");
    n_threads          = 256;
    n_blocks_elem      = (num_elem  / n_threads) + ((num_elem  % n_threads) ? 1 : 0);
    n_blocks_sides     = (num_sides / n_threads) + ((num_sides % n_threads) ? 1 : 0);

    // find the min inscribed circle
    preval_inscribed_circles<<<n_blocks_elem, n_threads>>>
                (d_J, d_V1x, d_V1y, d_V2x, d_V2y, d_V3x, d_V3y, num_elem);
    min_radius = (double *) malloc(num_elem * sizeof(double));

    /*
    // find the min inscribed circle. do it on the gpu if there are at least 256 elements
    if (num_elem >= 256) {
        //min_reduction<<<n_blocks_reduction, 256>>>(d_J, d_reduction, num_elem);
        hipDeviceSynchronize();
        checkCudaError("error after min_jacobian.");

        // each block finds the smallest value, so need to sort through n_blocks_reduction
        min_radius = (double *) malloc(n_blocks_reduction * sizeof(double));
        hipMemcpy(min_radius, d_reduction, n_blocks_reduction * sizeof(double), hipMemcpyDeviceToHost);
        min_r = min_radius[0];
        for (i = 1; i < n_blocks_reduction; i++) {
            min_r = (min_radius[i] < min_r) ? min_radius[i] : min_r;
        }
        free(min_radius);

    } else {
        */
        // just grab all the radii and sort them since there are so few of them
        min_radius = (double *) malloc(num_elem * sizeof(double));
        hipMemcpy(min_radius, d_J, num_elem * sizeof(double), hipMemcpyDeviceToHost);
        min_r = min_radius[0];
        for (i = 1; i < num_elem; i++) {
            min_r = (min_radius[i] < min_r) ? min_radius[i] : min_r;
        }
        free(min_radius);
    //}

    // pre computations
    preval_jacobian<<<n_blocks_elem, n_threads>>>(d_J, d_V1x, d_V1y, d_V2x, d_V2y, d_V3x, d_V3y, num_elem); 
    checkCudaError("error after preval_jacobian.");

    hipDeviceSynchronize();

    preval_side_length<<<n_blocks_sides, n_threads>>>(d_s_length, d_s_V1x, d_s_V1y, d_s_V2x, d_s_V2y, 
                                                      num_sides); 
    //hipDeviceSynchronize();
    preval_normals<<<n_blocks_sides, n_threads>>>(d_Nx, d_Ny, 
                                                  d_s_V1x, d_s_V1y, d_s_V2x, d_s_V2y,
                                                  d_V1x, d_V1y, 
                                                  d_V2x, d_V2y, 
                                                  d_V3x, d_V3y, 
                                                  d_left_side_number, num_sides); 
    hipDeviceSynchronize();

    // no longer need sides
    hipFree(d_s_V1x);
    hipFree(d_s_V2x);
    hipFree(d_s_V1y);
    hipFree(d_s_V2y);

    preval_normals_direction<<<n_blocks_sides, n_threads>>>(d_Nx, d_Ny, 
                                                  d_V1x, d_V1y, 
                                                  d_V2x, d_V2y, 
                                                  d_V3x, d_V3y, 
                                                  d_left_elem, d_left_side_number, num_sides); 

    preval_partials<<<n_blocks_elem, n_threads>>>(d_V1x, d_V1y,
                                                  d_V2x, d_V2y,
                                                  d_V3x, d_V3y,
                                                  d_xr,  d_yr,
                                                  d_xs,  d_ys, num_elem);
    hipDeviceSynchronize();
    checkCudaError("error after prevals.");

    // get the correct quadrature rules for this scheme
    set_quadrature(n, &r1_local, &r2_local, &w_local, 
                   &s_r, &oned_w_local, &n_quad, &n_quad1d);

    // evaluate the basis functions at those points and store on GPU
    preval_basis(r1_local, r2_local, s_r, w_local, oned_w_local, n_quad, n_quad1d, n_p);
    hipDeviceSynchronize();

    // initial conditions
    init_conditions<<<n_blocks_elem, n_threads>>>(d_c, d_J, d_V1x, d_V1y, d_V2x, d_V2y, d_V3x, d_V3y,
                    n_quad, n_p, num_elem);
    checkCudaError("error after initial conditions.");

    printf(" ? %i degree polynomial interpolation (n_p = %i)\n", n, n_p);
    printf(" ? %i precomputed basis points\n", n_quad * n_p);
    printf(" ? %i elements\n", num_elem);
    printf(" ? %i sides\n", num_sides);
    printf(" ? min radius = %lf\n", min_r);
    printf(" ? endtime = %lf\n", endtime);

    checkCudaError("error before time integration.");

    switch (time_integrator) {
        case RK4:
            time_integrate_rk4(n_quad, n_quad1d, n_p, n, num_elem, num_sides, endtime, min_r);
            break;
        case RK2:
            time_integrate_rk2(n_quad, n_quad1d, n_p, n, num_elem, num_sides, endtime, min_r);
            break;
        default:
            printf("Error: no time integrator selected.\n");
            exit(0);
    }

    // evaluate at the vertex points and copy over data
    Uv1 = (double *) malloc(num_elem * sizeof(double));
    Uv2 = (double *) malloc(num_elem * sizeof(double));
    Uv3 = (double *) malloc(num_elem * sizeof(double));

    // evaluate and write to file
    for (n = 0; n < local_N; n++) {
        eval_u<<<n_blocks_elem, n_threads>>>(d_c, d_Uv1, d_Uv2, d_Uv3, num_elem, n_p, n);
        hipMemcpy(Uv1, d_Uv1, num_elem * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(Uv2, d_Uv2, num_elem * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(Uv3, d_Uv3, num_elem * sizeof(double), hipMemcpyDeviceToHost);
        sprintf(out_filename, "output/U%d.msh", n);
        printf("Writing to %s...\n", out_filename);
        out_file  = fopen(out_filename , "w");
        fprintf(out_file, "View \"U%i \" {\n", n);
        for (i = 0; i < num_elem; i++) {
            fprintf(out_file, "ST (%lf,%lf,0,%lf,%lf,0,%lf,%lf,0) {%lf,%lf,%lf};\n", 
                                   V1x[i], V1y[i], V2x[i], V2y[i], V3x[i], V3y[i],
                                   Uv1[i], Uv2[i], Uv3[i]);
        }
        fprintf(out_file,"};");
        fclose(out_file);
    }

    // evaluate the u and v vectors and write to file
    //measure_error<<<n_blocks_elem, n_threads>>>(d_c, d_Uv1, d_Uv2, d_Uv3, 
                  //d_V1x, d_V1y, d_V2x, d_V2y, d_V3x, d_V3y,
                  //num_elem, n_p);

    //hipMemcpy(Uv1, d_Uv1, num_elem * sizeof(double), hipMemcpyDeviceToHost);
    //hipMemcpy(Uv2, d_Uv2, num_elem * sizeof(double), hipMemcpyDeviceToHost);
    //hipMemcpy(Uv3, d_Uv3, num_elem * sizeof(double), hipMemcpyDeviceToHost);
    //out_file  = fopen("output/p_error.out" , "w");
    //fprintf(out_file, "View \"E \" {\n");
    //for (i = 0; i < num_elem; i++) {
        //fprintf(out_file, "ST (%lf,%lf,0,%lf,%lf,0,%lf,%lf,0) {%lf,%lf,%lf};\n", 
                               //V1x[i], V1y[i], V2x[i], V2y[i], V3x[i], V3y[i],
                               //Uv1[i], Uv2[i], Uv3[i]);
    //}
    //fprintf(out_file,"};");
    //fclose(out_file);

    //error = (double *) malloc(num_elem * sizeof(double));
    //eval_error<<<n_blocks_elem, n_threads>>>(d_c, d_error, 
                                             //d_V1x, d_V1y,
                                             //d_V2x, d_V2y,
                                             //d_V3x, d_V3y,
                                             //num_elem, n_p,n_quad, 0);
    //hipMemcpy(error, d_error, num_elem * sizeof(double), hipMemcpyDeviceToHost);

    //hipMemcpy(Uv1, d_Uv1, num_elem * sizeof(double), hipMemcpyDeviceToHost);

    //double total_error = 0.;
    //for (i = 0; i < num_elem; i++) {
        //total_error += error[i];
    //}
    //total_error = sqrtf(total_error);
    //printf("error for rho = %.015lf\n", total_error);

    // free variables
    free_gpu();
    
    free(Uv1);
    free(Uv2);
    free(Uv3);

    free(V1x);
    free(V1y);
    free(V2x);
    free(V2y);
    free(V3x);
    free(V3y);

    free(elem_s1);
    free(elem_s2);
    free(elem_s3);

    free(sides_x1);
    free(sides_x2);
    free(sides_y1);
    free(sides_y2);

    free(left_elem);
    free(right_elem);
    free(left_side_number);
    free(right_side_number);

    free(r1_local);
    free(r2_local);
    free(w_local);
    free(s_r);
    free(oned_w_local);

    return 0;
}
