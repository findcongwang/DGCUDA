#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "conserv_kernels.cu"
#include "conserv_kernels_wrappers.cu"
//#include "conserv_kernels_wrappers.cu"
#include "time_integrator.cu"
#include "../quadrature.cu"
#include "../basis.cu"

extern int local_N;

/* 2dadvec_euler.cu
 * 
 * This file calls the kernels in 2dadvec_kernels_euler.cu for the 2D advection
 * DG method.
 */

/* set quadrature 
 *
 * sets the 1d quadrature integration points and weights for the boundary integrals
 * and the 2d quadrature integration points and weights for the volume intergrals.
 */
void set_quadrature(int n,
                    double **r1_local, double **r2_local, double **w_local,
                    double **s_r, double **oned_w_local, 
                    int *n_quad, int *n_quad1d) {
    int i;
    /*
     * The sides are mapped to the canonical element, so we want the integration points
     * for the boundary integrals for sides s1, s2, and s3 as shown below:

     s (r2) |\
     ^      | \
     |      |  \
     |      |   \
     |   s3 |    \ s2
     |      |     \
     |      |      \
     |      |       \
     |      |________\
     |         s1
     |
     ------------------------> r (r1)

    *
    */
    switch (n) {
        case 0: *n_quad = 1;
                *n_quad1d = 1;
                break;
        case 1: *n_quad = 3;
                *n_quad1d = 2;
                break;
        case 2: *n_quad = 6;
                *n_quad1d = 3;
                break;
        case 3: *n_quad = 12 ;
                *n_quad1d = 4;
                break;
        case 4: *n_quad = 16;
                *n_quad1d = 5;
                break;
        case 5: *n_quad = 25;
                *n_quad1d = 6;
                break;
    }
    // allocate integration points
    *r1_local = (double *)  malloc(*n_quad * sizeof(double));
    *r2_local = (double *)  malloc(*n_quad * sizeof(double));
    *w_local  = (double *) malloc(*n_quad * sizeof(double));
    printf("n_quad = %i\n", *n_quad);

    *s_r = (double *) malloc(*n_quad1d * sizeof(double));
    *oned_w_local = (double *) malloc(*n_quad1d * sizeof(double));

    // set 2D quadrature rules
    for (i = 0; i < *n_quad; i++) {
        if (n > 0) {
            (*r1_local)[i] = quad_2d[2 * n - 1][3*i];
            (*r2_local)[i] = quad_2d[2 * n - 1][3*i+1];
            (*w_local) [i] = quad_2d[2 * n - 1][3*i+2] / 2.; //weights are 2 times too big for some reason
        } else {
            (*r1_local)[i] = quad_2d[0][3*i];
            (*r2_local)[i] = quad_2d[0][3*i+1];
            (*w_local) [i] = quad_2d[0][3*i+2] / 2.; //weights are 2 times too big for some reason
        }
    }

    // set 1D quadrature rules
    for (i = 0; i < *n_quad1d; i++) {
        (*s_r)[i] = quad_1d[n][2*i];
        (*oned_w_local)[i] = quad_1d[n][2*i+1];
    }
}

void checkCudaError(const char *message)
{
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
        exit(-1);
    }
}

void read_mesh(FILE *mesh_file, 
              int *num_sides,
              int num_elem,
              double *V1x, double *V1y,
              double *V2x, double *V2y,
              double *V3x, double *V3y,
              int *left_side_number, int *right_side_number,
              double *sides_x1, double *sides_y1,
              double *sides_x2, double *sides_y2,
              int *elem_s1,  int *elem_s2, int *elem_s3,
              int *left_elem, int *right_elem) {

    int i, j, items, s1, s2, s3, numsides, boundary_side, boundary;
    double J, tmpx, tmpy;
    char line[100];
    numsides = 0;
    // stores the number of sides this element has.
    int *total_sides = (int *) malloc(num_elem * sizeof(int));
    for (i = 0; i < num_elem; i++) {
        total_sides[i] = 0;
    }

    i = 0;
    while(fgets(line, sizeof(line), mesh_file) != NULL) {
        // these three vertices define the element
        // and boundary_side tells which side is a boundary
        // while boundary determines the type of boundary
        items = sscanf(line, "%lf %lf %lf %lf %lf %lf %i %i", &V1x[i], &V1y[i], 
                                                              &V2x[i], &V2y[i], 
                                                              &V3x[i], &V3y[i], 
                                                              &boundary_side, &boundary);

        if (items != 8) {
            printf("error: not enough items (%i) while reading mesh.\n", items);
            exit(0);
        }

        // determine whether we should add these three sides or not
        s1 = 1;
        s2 = 1;
        s3 = 1;

        // enforce strictly positive jacobian
        J = (V2x[i] - V1x[i]) * (V3y[i] - V1y[i]) - (V3x[i] - V1x[i]) * (V2y[i] - V1y[i]);
        if (J < 0) {
            tmpx = V1x[i];
            tmpy = V1y[i];
            V1x[i] = V2x[i];
            V1y[i] = V2y[i];
            V2x[i] = tmpx;
            V2y[i] = tmpy;

            // need to swap boundary sides 0 and 1 since we swapped sides 0 and 1
            if (boundary_side == 0) {
                boundary_side = 1;
            } else if (boundary_side == 1) {
                boundary_side = 0;
            }
        }

        // scan through the existing sides to see if we already added it
        // TODO: yeah, there's a better way to do this.
        // TODO: Also, this is super sloppy. should be checking indices instead of double values.
        for (j = 0; j < numsides; j++) {
            // side 1
            if (s1 && ((sides_x1[j] == V1x[i] && sides_y1[j] == V1y[i]
             && sides_x2[j] == V2x[i] && sides_y2[j] == V2y[i]) 
            || (sides_x2[j] == V1x[i] && sides_y2[j] == V1y[i]
             && sides_x1[j] == V2x[i] && sides_y1[j] == V2y[i]))) {
                s1 = 0;
                // OK, we've added this side to element i
                right_elem[j] = i;
                // link the added side j to this element
                elem_s1[i] = j;
                right_side_number[j] = 0;
                break;
            }
        }
        for (j = 0; j < numsides; j++) {
            // side 2
            if (s2 && ((sides_x1[j] == V2x[i] && sides_y1[j] == V2y[i]
             && sides_x2[j] == V3x[i] && sides_y2[j] == V3y[i]) 
            || (sides_x2[j] == V2x[i] && sides_y2[j] == V2y[i]
             && sides_x1[j] == V3x[i] && sides_y1[j] == V3y[i]))) {
                s2 = 0;
                // OK, we've added this side to some element before; which one?
                right_elem[j] = i;
                elem_s2[i] = j;
                // link the added side to this element
                right_side_number[j] = 1;
                break;
            }
        }
        for (j = 0; j < numsides; j++) {
            // side 3
            if (s3 && ((sides_x1[j] == V1x[i] && sides_y1[j] == V1y[i]
             && sides_x2[j] == V3x[i] && sides_y2[j] == V3y[i]) 
            || (sides_x2[j] == V1x[i] && sides_y2[j] == V1y[i]
             && sides_x1[j] == V3x[i] && sides_y1[j] == V3y[i]))) {
                s3 = 0;
                // OK, we've added this side to some element before; which one?
                right_elem[j] = i;
                elem_s3[i] = j;
                // link the added side to this element
                right_side_number[j] = 2;
                break;
            }
        }
        // if we haven't added the side already, add it
        if (s1) {
            sides_x1[numsides] = V1x[i];
            sides_y1[numsides] = V1y[i];
            sides_x2[numsides] = V2x[i];
            sides_y2[numsides] = V2y[i];

            // link the added side to this element
            left_side_number[numsides] = 0;

            // see if this is a boundary side
            if (boundary_side == 0) {
                switch (boundary) {
                    case 10000: right_elem[numsides] = -1;
                                break;
                    case 20000: right_elem[numsides] = -2;
                                break;
                    case 30000: right_elem[numsides] = -3;
                                break;
                }
            }

            // and link the element to this side
            elem_s1[i] = numsides;

            // make this the left element
            left_elem[numsides] = i;
            numsides++;
        }
        if (s2) {
            sides_x1[numsides] = V2x[i];
            sides_y1[numsides] = V2y[i];
            sides_x2[numsides] = V3x[i];
            sides_y2[numsides] = V3y[i];

            // link the added side to this element
            left_side_number[numsides] = 1;

            // see if this is a boundary side
            if (boundary_side == 1) {
                switch (boundary) {
                    case 10000: right_elem[numsides] = -1;
                                break;
                    case 20000: right_elem[numsides] = -2;
                                break;
                    case 30000: right_elem[numsides] = -3;
                                break;
                }
            }

            // and link the element to this side
            elem_s2[i] = numsides;

            // make this the left element
            left_elem[numsides] = i;
            numsides++;
        }
        if (s3) {
            sides_x1[numsides] = V3x[i];
            sides_y1[numsides] = V3y[i];
            sides_x2[numsides] = V1x[i];
            sides_y2[numsides] = V1y[i];

            // link the added side to this element
            left_side_number[numsides] = 2;

            // see if this is a boundary side
            if (boundary_side == 2) {
                switch (boundary) {
                    case 10000: right_elem[numsides] = -1;
                                break;
                    case 20000: right_elem[numsides] = -2;
                                break;
                    case 30000: right_elem[numsides] = -3;
                                break;
                }
            }

            // and link the element to this side
            elem_s3[i] = numsides;

            // make this the left element
            left_elem[numsides] = i;
            numsides++;
        }
        i++;
    }
    //free(total_sides);
    *num_sides = numsides;
}

void init_gpu(int num_elem, int num_sides, int n_p,
              double *V1x, double *V1y, 
              double *V2x, double *V2y, 
              double *V3x, double *V3y, 
              int *left_side_number, int *right_side_number,
              double *sides_x1, double *sides_y1,
              double *sides_x2, double *sides_y2,
              int *elem_s1, int *elem_s2, int *elem_s3,
              int *left_elem, int *right_elem) {
    int reduction_size = (num_elem  / 256) + ((num_elem  % 256) ? 1 : 0);

    checkCudaError("error before init.");
    hipDeviceReset();

    hipMalloc((void **) &d_c,        local_N * num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_c_prev,   local_N * num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_quad_rhs, local_N * num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_left_riemann_rhs,  local_N * num_sides * n_p * sizeof(double));
    hipMalloc((void **) &d_right_riemann_rhs, local_N * num_sides * n_p * sizeof(double));

    hipMalloc((void **) &d_kstar, local_N * num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_k1, local_N * num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_k2, local_N * num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_k3, local_N * num_elem * n_p * sizeof(double));
    hipMalloc((void **) &d_k4, local_N * num_elem * n_p * sizeof(double));

    hipMalloc((void **) &d_J        , num_elem * sizeof(double));
    hipMalloc((void **) &d_lambda   , num_elem * sizeof(double));
    hipMalloc((void **) &d_reduction, reduction_size * sizeof(double));
    hipMalloc((void **) &d_s_length , num_sides * sizeof(double));

    hipMalloc((void **) &d_s_V1x, num_sides * sizeof(double));
    hipMalloc((void **) &d_s_V2x, num_sides * sizeof(double));
    hipMalloc((void **) &d_s_V1y, num_sides * sizeof(double));
    hipMalloc((void **) &d_s_V2y, num_sides * sizeof(double));

    hipMalloc((void **) &d_elem_s1, num_elem * sizeof(int));
    hipMalloc((void **) &d_elem_s2, num_elem * sizeof(int));
    hipMalloc((void **) &d_elem_s3, num_elem * sizeof(int));

    hipMalloc((void **) &d_Uv1, num_elem * sizeof(double));
    hipMalloc((void **) &d_Uv2, num_elem * sizeof(double));
    hipMalloc((void **) &d_Uv3, num_elem * sizeof(double));

    hipMalloc((void **) &d_error, num_elem * sizeof(double));

    hipMalloc((void **) &d_V1x, num_elem * sizeof(double));
    hipMalloc((void **) &d_V1y, num_elem * sizeof(double));
    hipMalloc((void **) &d_V2x, num_elem * sizeof(double));
    hipMalloc((void **) &d_V2y, num_elem * sizeof(double));
    hipMalloc((void **) &d_V3x, num_elem * sizeof(double));
    hipMalloc((void **) &d_V3y, num_elem * sizeof(double));

    hipMalloc((void **) &d_xr, num_elem * sizeof(double));
    hipMalloc((void **) &d_yr, num_elem * sizeof(double));
    hipMalloc((void **) &d_xs, num_elem * sizeof(double));
    hipMalloc((void **) &d_ys, num_elem * sizeof(double));

    hipMalloc((void **) &d_left_side_number , num_sides * sizeof(int));
    hipMalloc((void **) &d_right_side_number, num_sides * sizeof(int));

    hipMalloc((void **) &d_Nx, num_sides * sizeof(double));
    hipMalloc((void **) &d_Ny, num_sides * sizeof(double));

    hipMalloc((void **) &d_right_elem, num_sides * sizeof(int));
    hipMalloc((void **) &d_left_elem , num_sides * sizeof(int));

    // copy over data
    hipMemcpy(d_s_V1x, sides_x1, num_sides * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_s_V1y, sides_y1, num_sides * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_s_V2x, sides_x2, num_sides * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_s_V2y, sides_y2, num_sides * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_left_side_number , left_side_number , num_sides * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_right_side_number, right_side_number, num_sides * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_elem_s1, elem_s1, num_elem * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_elem_s2, elem_s2, num_elem * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_elem_s3, elem_s3, num_elem * sizeof(int), hipMemcpyHostToDevice);
    checkCudaError("error inside gpu init.");

    hipMemcpy(d_V1x, V1x, num_elem * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V1y, V1y, num_elem * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V2x, V2x, num_elem * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V2y, V2y, num_elem * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V3x, V3x, num_elem * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_V3y, V3y, num_elem * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(d_left_elem , left_elem , num_sides * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_right_elem, right_elem, num_sides * sizeof(int), hipMemcpyHostToDevice);
}

void free_gpu() {
    hipFree(d_c);
    hipFree(d_c_prev);
    hipFree(d_quad_rhs);
    hipFree(d_left_riemann_rhs);
    hipFree(d_right_riemann_rhs);

    hipFree(d_kstar);
    hipFree(d_k1);
    hipFree(d_k2);
    hipFree(d_k3);
    hipFree(d_k4);

    hipFree(d_J);
    hipFree(d_lambda);
    hipFree(d_reduction);
    hipFree(d_s_length);

    hipFree(d_s_V1x);
    hipFree(d_s_V2x);
    hipFree(d_s_V1y);
    hipFree(d_s_V2y);

    hipFree(d_elem_s1);
    hipFree(d_elem_s2);
    hipFree(d_elem_s3);

    hipFree(d_Uv1);
    hipFree(d_Uv2);
    hipFree(d_Uv3);

    hipFree(d_V1x);
    hipFree(d_V1y);
    hipFree(d_V2x);
    hipFree(d_V2y);
    hipFree(d_V3x);
    hipFree(d_V3y);

    hipFree(d_xr);
    hipFree(d_yr);
    hipFree(d_xs);
    hipFree(d_ys);

    hipFree(d_left_side_number);
    hipFree(d_right_side_number);

    hipFree(d_Nx);
    hipFree(d_Ny);

    hipFree(d_right_elem);
    hipFree(d_left_elem);
}

void usage_error() {
    printf("\nUsage: dgcuda [OPTIONS] [MESH] [OUTFILE]\n");
    printf(" Options: [-n] Order of polynomial approximation.\n");
    printf("          [-t] Number of timesteps.\n");
    printf("          [-d] Debug.\n");
}

int get_input(int argc, char *argv[],
               int *n, int *timesteps, 
               double *endtime,
               char **mesh_filename) {

    int i;

    *timesteps = 1;
    // read command line input
    if (argc < 5) {
        usage_error();
        return 1;
    }
    for (i = 0; i < argc; i++) {
        // order of polynomial
        if (strcmp(argv[i], "-n") == 0) {
            if (i + 1 < argc) {
                *n = atoi(argv[i+1]);
                if (*n < 0 || *n > 5) {
                    usage_error();
                    return 1;
                }
            } else {
                usage_error();
                return 1;
            }
        }
        // number of timesteps
        if (strcmp(argv[i], "-t") == 0) {
            if (i + 1 < argc) {
                *timesteps = atoi(argv[i+1]);
                if (*timesteps < 0) {
                    usage_error();
                    return 1;
                }
            } else {
                usage_error();
                return 1;
            }
        }
        if (strcmp(argv[i], "-T") == 0) {
            if (i + 1 < argc) {
                *endtime = atof(argv[i+1]);
                if (*endtime < 0) {
                    usage_error();
                    return 1;
                }
            } else {
                usage_error();
                return 1;
            }
        }
    } 

    // second last argument is filename
    *mesh_filename = argv[argc - 1];

    return 0;
}
