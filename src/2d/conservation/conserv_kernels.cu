#include "hip/hip_runtime.h"
/* 2dadvec_kernels_euler.cu
 *
 * This file contains the kernels for the 2D euler DG method.
 *
 * d_t [   rho   ] + d_x [     rho * u    ] + d_y [    rho * v     ] = 0
 * d_t [ rho * u ] + d_x [ rho * u^2 + p  ] + d_y [   rho * u * v  ] = 0
 * d_t [ rho * v ] + d_x [  rho * u * v   ] + d_y [  rho * v^2 + p ] = 0
 * d_t [    E    ] + d_x [ u * ( E +  p ) ] + d_y [ v * ( E +  p ) ] = 0
 *

 */

#include "conserv_headers.cuh"

/***********************
 *
 * DEVICE VARIABLES
 *
 ***********************/
/* These are always prefixed with d_ for "device" */
double *d_c;                 // coefficients for [rho, rho * u, rho * v, E]
double *d_c_prev;            // coefficients for [rho, rho * u, rho * v, E]
double *d_quad_rhs;          // the right hand side containing the quadrature contributions
double *d_left_riemann_rhs;  // the right hand side containing the left riemann contributions
double *d_right_riemann_rhs; // the right hand side containing the right riemann contributions

// TODO: switch to low storage runge-kutta
// runge kutta variables
double *d_kstar;
double *d_k1;
double *d_k2;
double *d_k3;
double *d_k4;

// precomputed basis functions 
// TODO: maybe making these 2^n makes sure the offsets are cached more efficiently? who knows...
// precomputed basis functions ordered like so
//
// [phi_1(r1, s1), phi_1(r2, s2), ... , phi_1(r_nq, s_nq)   ]
// [phi_2(r1, s1), phi_2(r2, s2), ... , phi_2(r_nq, s_nq)   ]
// [   .               .           .            .           ]
// [   .               .           .            .           ]
// [   .               .           .            .           ]
// [phi_np(r1, s1), phi_np(r2, s2), ... , phi_np(r_nq, s_nq)]
//
__device__ __constant__ int N;
__device__ __constant__ double basis[2048];
// note: these are multiplied by the weights
__device__ __constant__ double basis_grad_x[2048]; 
__device__ __constant__ double basis_grad_y[2048]; 

// precomputed basis functions evaluated along the sides. ordered
// similarly to basis and basis_grad_{x,y} but with one "matrix" for each side
// starting with side 0. to get to each side, offset with:
//      side_number * n_p * num_quad1d.
//__device__ __constant__ int n_p;
//__device__ __constant__ int num_elem;
//__device__ __constant__ int num_sides;
//__device__ __constant__ int n_quad;
//__device__ __constant__ int n_quad1d;

__device__ __constant__ double basis_side[1024];
__device__ __constant__ double basis_vertex[256];

// weights for 2d and 1d quadrature rules
__device__ __constant__ double w[64];
__device__ __constant__ double w_oned[16];

__device__ __constant__ double r1[32];
__device__ __constant__ double r2[32];
__device__ __constant__ double r_oned[32];

void set_N(int value) {
    hipMemcpyToSymbol(HIP_SYMBOL("N"), (void *) &value, sizeof(int));
}

void set_basis(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis"), value, size * sizeof(double));
}
void set_basis_grad_x(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_grad_x"), value, size * sizeof(double));
}
void set_basis_grad_y(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_grad_y"), value, size * sizeof(double));
}
void set_basis_side(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_side"), value, size * sizeof(double));
}
void set_basis_vertex(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_vertex"), value, size * sizeof(double));
}
void set_w(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("w"), value, size * sizeof(double));
}
void set_w_oned(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("w_oned"), value, size * sizeof(double));
}
void set_r1(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("r1"), value, size * sizeof(double));
}
void set_r2(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("r2"), value, size * sizeof(double));
}
void set_r_oned(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("r_oned"), value, size * sizeof(double));
}

// tells which side (1, 2, or 3) to evaluate this boundary integral over
int *d_left_side_number;
int *d_right_side_number;

double *d_J;         // jacobian determinant 
double *d_reduction; // for the min / maxes in the reductions 
double *d_lambda;    // stores computed lambda values for each element
double *d_s_length;  // length of sides

// the num_elem values of the x and y coordinates for the two vertices defining a side
// TODO: can i delete these after the lengths are precomputed?
//       maybe these should be in texture memory?
double *d_s_V1x;
double *d_s_V1y;
double *d_s_V2x;
double *d_s_V2y;

// the num_elem values of the x and y partials
double *d_xr;
double *d_yr;
double *d_xs;
double *d_ys;

// the K indices of the sides for each element ranged 0->H-1
int *d_elem_s1;
int *d_elem_s2;
int *d_elem_s3;

// vertex x and y coordinates on the mesh which define an element
// TODO: can i delete these after the jacobians are precomputed?
//       maybe these should be in texture memory?
double *d_V1x;
double *d_V1y;
double *d_V2x;
double *d_V2y;
double *d_V3x;
double *d_V3y;

// stores computed values at three vertices
double *d_Uv1;
double *d_Uv2;
double *d_Uv3;

// for computing the error
double *d_error;

// normal vectors for the sides
double *d_Nx;
double *d_Ny;

// index lists for sides
int *d_left_elem;  // index of left  element for side idx
int *d_right_elem; // index of right element for side idx

/* initial conditions
 *
 * computes the coefficients for the initial conditions
 * THREADS: num_elem
 */
__global__ void init_conditions(double *c, double *J,
                                double *V1x, double *V1y,
                                double *V2x, double *V2y,
                                double *V3x, double *V3y,
                                int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i, n;
    double U[4];

    if (idx < num_elem) {
        for (i = 0; i < n_p; i++) {
            // evaluate U times the i'th basis function
            evalU0(U, V1x[idx], V1y[idx], V2x[idx], V2y[idx], V3x[idx], V3y[idx], i, n_p, n_quad);

            // store the coefficients
            for (n = 0; n < 4; n++) {
                c[num_elem * n_p * n + i * num_elem + idx] = U[n];
            }
        } 
    }
}

/* min reduction function
 *
 * returns the min value from the global data J and stores in min_J
 * each block computes the min jacobian inside of that block and stores it in the
 * blockIdx.x spot of the shared min_J variable.
 * NOTE: this is fixed for 256 threads.
 */
__global__ void min_reduction(double *D, double *min_D, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;
    int i   = (blockIdx.x * 256 * 2) + threadIdx.x;

    __shared__ double s_min[256];

    if (idx < num_elem) {
        // set all of min to D[idx] initially
        s_min[tid] = D[idx];
        __syncthreads();

        // test a few
        while (i < num_elem) {
            s_min[tid] = (s_min[tid] < D[i]) ? s_min[tid] : D[i];
            s_min[tid] = (s_min[tid] < D[i + 256]) ? s_min[tid] : D[i];
            i += gridDim.x * 256 * 2;
            __syncthreads();
        }

        // first half of the warps
        __syncthreads();
        if (tid < 128) {
            s_min[tid] = (s_min[tid] < s_min[tid + 128]) ? s_min[tid] : s_min[tid + 128];
        }

        // first and second warps
        __syncthreads();
        if (tid < 64) {
            s_min[tid] = (s_min[tid] < s_min[tid + 64]) ? s_min[tid] : s_min[tid + 64];
        }

        // unroll last warp
        __syncthreads();
        if (tid < 32) {
            if (blockDim.x >= 64) {
                s_min[tid] = (s_min[tid] < s_min[tid + 32]) ? s_min[tid] : s_min[tid + 32];
            }
            if (blockDim.x >= 32) {
                s_min[tid] = (s_min[tid] < s_min[tid + 16]) ? s_min[tid] : s_min[tid + 16];
            }
            if (blockDim.x >= 16) {
                s_min[tid] = (s_min[tid] < s_min[tid + 8]) ? s_min[tid] : s_min[tid + 8];
            }
            if (blockDim.x >= 8) {
                s_min[tid] = (s_min[tid] < s_min[tid + 4]) ? s_min[tid] : s_min[tid + 4];
            }
            if (blockDim.x >= 4) {
                s_min[tid] = (s_min[tid] < s_min[tid + 2]) ? s_min[tid] : s_min[tid + 2];
            }
            if (blockDim.x >= 2) {
                s_min[tid] = (s_min[tid] < s_min[tid + 1]) ? s_min[tid] : s_min[tid + 1];
            }
        }

        __syncthreads();
        if (tid == 0) {
            min_D[blockIdx.x] = s_min[0];
        }
    }
}

/* max reduction function
 *
 * returns the max value from the global data D and stores in max
 * each block computes the max jacobian inside of that block and stores it in the
 * blockIdx.x spot of the shared max variable.
 * NOTE: this is fixed for 256 threads.
 */
__global__ void max_reduction(double *D, double *max_D, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;
    int i   = (blockIdx.x * 256 * 2) + threadIdx.x;

    __shared__ double s_max[256];

    if (idx < num_elem) {
        // set all of max to D[idx] initially
        s_max[tid] = D[idx];
        __syncthreads();

        // test a few
        while (i + 256 < num_elem) {
            s_max[tid] = (s_max[tid] > D[i]) ? s_max[tid] : D[i];
            s_max[tid] = (s_max[tid] > D[i + 256]) ? s_max[tid] : D[i];
            i += gridDim.x * 256 * 2;
            __syncthreads();
        }

        // first half of the warps
        __syncthreads();
        if (tid < 128) {
            s_max[tid] = (s_max[tid] > s_max[tid + 128]) ? s_max[tid] : s_max[tid + 128];
        }

        // first and second warps
        __syncthreads();
        if (tid < 64) {
            s_max[tid] = (s_max[tid] > s_max[tid + 64]) ? s_max[tid] : s_max[tid + 64];
        }

        // unroll last warp
        __syncthreads();
        if (tid < 32) {
            if (blockDim.x >= 64) {
                s_max[tid] = (s_max[tid] > s_max[tid + 32]) ? s_max[tid] : s_max[tid + 32];
            }
            if (blockDim.x >= 32) {
                s_max[tid] = (s_max[tid] > s_max[tid + 16]) ? s_max[tid] : s_max[tid + 16];
            }
            if (blockDim.x >= 16) {
                s_max[tid] = (s_max[tid] > s_max[tid + 8]) ? s_max[tid] : s_max[tid + 8];
            }
            if (blockDim.x >= 8) {
                s_max[tid] = (s_max[tid] > s_max[tid + 4]) ? s_max[tid] : s_max[tid + 4];
            }
            if (blockDim.x >= 4) {
                s_max[tid] = (s_max[tid] > s_max[tid + 2]) ? s_max[tid] : s_max[tid + 2];
            }
            if (blockDim.x >= 2) {
                s_max[tid] = (s_max[tid] > s_max[tid + 1]) ? s_max[tid] : s_max[tid + 1];
            }
        }

        __syncthreads();
        if (tid == 0) {
            max_D[blockIdx.x] = s_max[0];
        }
    }
}

/***********************
 *
 * PRECOMPUTING
 *
 ***********************/

/* side length computer
 *
 * precomputes the length of each side.
 * THREADS: num_sides
 */ 
__global__ void preval_side_length(double *s_length, 
                              double *s_V1x, double *s_V1y, 
                              double *s_V2x, double *s_V2y,
                              int num_sides) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        // compute and store the length of the side
        s_length[idx] = sqrtf(powf(s_V1x[idx] - s_V2x[idx],2) + powf(s_V1y[idx] - s_V2y[idx],2));
    }
}

/* inscribed circle radius computing
 *
 * computes the radius of each inscribed circle. stores in d_J to find the minumum,
 * then we reuse d_J.
 */
__global__ void preval_inscribed_circles(double *J,
                                    double *V1x, double *V1y,
                                    double *V2x, double *V2y,
                                    double *V3x, double *V3y,
                                    int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double a, b, c, k;
        a = sqrtf(powf(V1x[idx] - V2x[idx], 2) + powf(V1y[idx] - V2y[idx], 2));
        b = sqrtf(powf(V2x[idx] - V3x[idx], 2) + powf(V2y[idx] - V3y[idx], 2));
        c = sqrtf(powf(V1x[idx] - V3x[idx], 2) + powf(V1y[idx] - V3y[idx], 2));

        k = 0.5 * (a + b + c);

        // for the diameter, we multiply by 2
        J[idx] = 2 * sqrtf(k * (k - a) * (k - b) * (k - c)) / k;
    }
}

/* jacobian computing
 *
 * precomputes the jacobian determinant for each element.
 * THREADS: num_elem
 */
__global__ void preval_jacobian(double *J, 
                           double *V1x, double *V1y, 
                           double *V2x, double *V2y, 
                           double *V3x, double *V3y,
                           int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double x1, y1, x2, y2, x3, y3;

        // read vertex points
        x1 = V1x[idx];
        y1 = V1y[idx];
        x2 = V2x[idx];
        y2 = V2y[idx];
        x3 = V3x[idx];
        y3 = V3y[idx];

        // calculate jacobian determinant
        // x = x2 * r + x3 * s + x1 * (1 - r - s)
        J[idx] = (x2 - x1) * (y3 - y1) - (x3 - x1) * (y2 - y1);
    }
}

/* evaluate normal vectors
 *
 * computes the normal vectors for each element along each side.
 * THREADS: num_sides
 *
 */
__global__ void preval_normals(double *Nx, double *Ny, 
                          double *s_V1x, double *s_V1y, 
                          double *s_V2x, double *s_V2y,
                          double *V1x, double *V1y, 
                          double *V2x, double *V2y, 
                          double *V3x, double *V3y,
                          int *left_side_number, int num_sides) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        double x, y, length;
        double sv1x, sv1y, sv2x, sv2y;
    
        sv1x = s_V1x[idx];
        sv1y = s_V1y[idx];
        sv2x = s_V2x[idx];
        sv2y = s_V2y[idx];
    
        // lengths of the vector components
        x = sv2x - sv1x;
        y = sv2y - sv1y;
    
        // normalize
        length = sqrtf(powf(x, 2) + powf(y, 2));

        // store the result
        Nx[idx] = -y / length;
        Ny[idx] =  x / length;
    }
}

__global__ void preval_normals_direction(double *Nx, double *Ny, 
                          double *V1x, double *V1y, 
                          double *V2x, double *V2y, 
                          double *V3x, double *V3y,
                          int *left_elem, int *left_side_number, int num_sides) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        double new_x, new_y, dot;
        double initial_x, initial_y, target_x, target_y;
        double x, y;
        int left_idx, side;

        // get left side's vertices
        left_idx = left_elem[idx];
        side     = left_side_number[idx];

        // get the normal vector
        x = Nx[idx];
        y = Ny[idx];
    
        // make it point the correct direction by learning the third vertex point
        switch (side) {
            case 0: 
                target_x = V3x[left_idx];
                target_y = V3y[left_idx];
                initial_x = (V1x[left_idx] + V2x[left_idx]) / 2.;
                initial_y = (V1y[left_idx] + V2y[left_idx]) / 2.;
                break;
            case 1:
                target_x = V1x[left_idx];
                target_y = V1y[left_idx];
                initial_x = (V2x[left_idx] + V3x[left_idx]) / 2.;
                initial_y = (V2y[left_idx] + V3y[left_idx]) / 2.;
                break;
            case 2:
                target_x = V2x[left_idx];
                target_y = V2y[left_idx];
                initial_x = (V1x[left_idx] + V3x[left_idx]) / 2.;
                initial_y = (V1y[left_idx] + V3y[left_idx]) / 2.;
                break;
        }

        // create the vector pointing towards the third vertex point
        new_x = target_x - initial_x;
        new_y = target_y - initial_y;

        // find the dot product between the normal and new vectors
        dot = x * new_x + y * new_y;
        
        if (dot > 0) {
            Nx[idx] *= -1;
            Ny[idx] *= -1;
        }
    }
}

__global__ void preval_partials(double *V1x, double *V1y,
                                double *V2x, double *V2y,
                                double *V3x, double *V3y,
                                double *xr,  double *yr,
                                double *xs,  double *ys, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num_elem) {
        // evaulate the jacobians of the mappings for the chain rule
        // x = x2 * r + x3 * s + x1 * (1 - r - s)
        xr[idx] = V2x[idx] - V1x[idx];
        yr[idx] = V2y[idx] - V1y[idx];
        xs[idx] = V3x[idx] - V1x[idx];
        ys[idx] = V3y[idx] - V1y[idx];
    }
}

/***********************
 *
 * MAIN FUNCTIONS
 *
 ***********************/

/* limiter
 *
 * the standard limiter for coefficient values
 */
__global__ void limit_c(double *c_inner, 
                   double *c_s1, double *c_s2, double *c_s3,
                   int n_p, int num_elem) {

    //int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // get cell averages
    //avg_inner = c_inner[0];
    //avg_s1 = c_s1[0];
    //avg_s2 = c_s2[0];
    //avg_s3 = c_s3[0];

    // determine if this is a "troubled" cell

    //for (i = n_p; i > 1; i++) {
        //c_prev = c[i - 1];
    //}
}

/* left & right evaluator
 * 
 * calculate U_left and U_right at the integration point,
 * using boundary conditions when necessary.
 */
 __device__ void eval_left_right(double *C_left, double *C_right, 
                             double *U_left, double *U_right,
                             double nx, double ny,
                             double v1x, double v1y,
                             double v2x, double v2y,
                             double v3x, double v3y,
                             int j, // j, as usual, is the index of the integration point
                             int left_side, int right_side,
                             int left_idx, int right_idx,
                             int n_p, int num_elem, int n_quad1d,
                             int num_sides, double t) { 

    int i, n;

    // set U to 0
    for (n = 0; n < N; n++) {
        U_left[n]  = 0.;
        U_right[n] = 0.;
    }

    //evaluate U at the integration points
    for (i = 0; i < n_p; i++) {
        for (n = 0; n < N; n++) {
            U_left[n] += C_left[n*n_p + i] * 
                         basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
        }
    }

    // boundaries are sorted to avoid warp divergence
    switch (right_idx) {
        // reflecting 
        case -1: 
            reflecting_boundary(U_left, U_right,
                v1x, v1y, v2x, v2y, v3x, v3y, 
                nx, ny,
                j, left_side, n_quad1d);
            break;
        // outflow 
        case -2: 
            outflow_boundary(U_left, U_right,
                v1x, v1y, v2x, v2y, v3x, v3y, 
                nx, ny,
                j, left_side, n_quad1d);
            break;
        // inflow 
        case -3: 
            inflow_boundary(U_left, U_right,
                v1x, v1y, v2x, v2y, v3x, v3y, 
                nx, ny, 
                j, left_side, n_quad1d);
            break;
        // not a boundary
        default:
            // evaluate the right side at the integration point
            for (i = 0; i < n_p; i++) {
                for (n = 0; n < N; n++) {
                    U_right[n] += C_right[n*n_p + i] * 
                                  basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - j - 1];
                }
            }
            break;
    }
}

/* surface integral evaluation
 *
 * evaluate all the riemann problems for each element.
 * THREADS: num_sides
 */
__device__ void eval_surface(double *C_left, double *C_right,
                             double *left_riemann_rhs, double *right_riemann_rhs, 
                             double len, 
                             double v1x, double v1y,
                             double v2x, double v2y,
                             double v3x, double v3y,
                             int left_idx,  int right_idx,
                             int left_side, int right_side,
                             double nx, double ny, 
                             int n_quad1d, int n_quad, int n_p, int num_sides, 
                             int num_elem, double t, int idx) {
    int i, j, n;
    double s;
    double lambda;
    register double sum_left[4], sum_right[4];
    register double flux_x_l[4], flux_y_l[4];
    register double flux_x_r[4], flux_y_r[4];
    register double U_left[4], U_right[4];

    // multiply across by the i'th basis function
    for (i = 0; i < n_p; i++) {

        // initilize to zero
        for (n = 0; n < N; n++) {
            sum_left [n] = 0.;
            sum_right[n] = 0.;
        }

        for (j = 0; j < n_quad1d; j++) {
            // calculate the left and right values along the surface
            eval_left_right(C_left, C_right,
                            U_left, U_right,
                            nx, ny,
                            v1x, v1y, v2x, v2y, v3x, v3y,
                            j, left_side, right_side,
                            left_idx, right_idx,
                            n_p, num_elem, n_quad1d, num_sides, t);

            // calculate the left and right fluxes
            eval_flux(U_left, flux_x_l, flux_y_l);
            eval_flux(U_right, flux_x_r, flux_y_r);

            // calculate the max wave speed at this integration point
            lambda = eval_lambda(U_left, U_right, nx, ny);

            // calculate the riemann problem
            for (n = 0; n < N; n++) {
                s = 0.5 * ((flux_x_l[n] + flux_x_r[n]) * nx + (flux_y_l[n] + flux_y_r[n]) * ny 
                            + lambda * (U_left[n] - U_right[n]));
                sum_left[n]  += w_oned[j] * s * basis_side[left_side  * n_p * n_quad1d + i * n_quad1d + j];
                sum_right[n] += w_oned[j] * s * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
            }
        }

        // store this side's contribution in the riemann rhs vectors
        __syncthreads();
        for (n = 0; n < N; n++) {
            left_riemann_rhs[num_sides * n_p * n + i * num_sides + idx]  = -len / 2. * sum_left[n];
            right_riemann_rhs[num_sides * n_p * n + i * num_sides + idx] =  len / 2. * sum_right[n];
        }
    }
}


/* volume integrals
 *
 * evaluates and adds the volume integral to the rhs vector
 * THREADS: num_elem
 */
__device__ void eval_volume(double *C, double *quad_rhs, 
                            double x_r, double y_r, double x_s, double y_s,
                            int n_quad, int n_p, int num_elem, int idx) {
    int i, j, k, n;
    register double U[4];
    register double flux_x[4], flux_y[4];
    register double sum[4];

    // evaluate the volume integral for each coefficient
    for (i = 0; i < n_p; i++) {

        // initialize sum to 0
        for (n = 0; n < N; n++) {
            sum[n] = 0.;
        }

        // for each integration point
        for (j = 0; j < n_quad; j++) {
            // initialize to zero
            for (n = 0; n < N; n++) {
                U[n] = 0.;
            }
            // calculate at the integration point
            for (k = 0; k < n_p; k++) {
                for (n = 0; n < N; n++) {
                    U[n] += C[n*n_p + k] * basis[n_quad * k + j];
                }
            }
            // evaluate the flux
            eval_flux(U, flux_x, flux_y);
            // compute the sum
            //     [fx fy] * [y_s, -y_r; -x_s, x_r] * [phi_x phi_y]
            for (n = 0; n < N; n++) {
                sum[n] += flux_x[n] * ( basis_grad_x[n_quad * i + j] * y_s
                                       -basis_grad_y[n_quad * i + j] * y_r)
                        + flux_y[n] * (-basis_grad_x[n_quad * i + j] * x_s 
                                      + basis_grad_y[n_quad * i + j] * x_r);
            }
        }

        // store the result
        for (n = 0; n < N; n++) {
            quad_rhs[num_elem * n_p * n + i * num_elem + idx] = sum[n];
        }
    }
}

/* evaluate u
 * 
 * evaluates rho and E at the three vertex points for output
 * THREADS: num_elem
 */
__global__ void eval_u(double *C, 
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elem) {
        int i;
        double uv1, uv2, uv3;

        // calculate values at the integration points
        uv1 = 0.;
        uv2 = 0.;
        uv3 = 0.;
        for (i = 0; i < n_p; i++) {
            uv1 += C[num_elem * n_p * n + i * num_elem + idx] * basis_vertex[i * 3 + 0];
            uv2 += C[num_elem * n_p * n + i * num_elem + idx] * basis_vertex[i * 3 + 1];
            uv3 += C[num_elem * n_p * n + i * num_elem + idx] * basis_vertex[i * 3 + 2];
        }

        // store result
        Uv1[idx] = uv1;
        Uv2[idx] = uv2;
        Uv3[idx] = uv3;
    }
}

/* evaluate error
 * 
 * evaluates rho and E at the three vertex points for output
 * THREADS: num_elem
 */
__global__ void eval_error(double *C, double *error,
                       double *V1x, double *V1y,
                       double *V2x, double *V2y,
                       double *V3x, double *V3y,
                       int num_elem, int n_p, int n_quad, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elem) {
        int i, j;
        double e;
        double x, y;
        double U;

        double v1x, v1y, v2x, v2y, v3x, v3y;
        v1x = V1x[idx];
        v1y = V1y[idx];
        v2x = V2x[idx];
        v2y = V2y[idx];
        v3x = V3x[idx];
        v3y = V3y[idx];

        e = 0;
        for (j = 0; j < n_quad; j++) {
            // get the actual point on the mesh
            x = r1[j] * v2x + r2[j] * v3x + (1 - r1[j] - r2[j]) * v1x;
            y = r1[j] * v2y + r2[j] * v3y + (1 - r1[j] - r2[j]) * v1y;
            
            // evaluate U at the integration point
            U = 0.;
            for (i = 0; i < n_p; i++) {
                U += C[num_elem * n_p * n + i * num_elem + idx] * basis[i * n_quad + j];
            }

            e += w[j] * powf((U0(x, y) - U),2); 
            // evaluate exact conditions at the integration point
            //if (n == 0) {
                //e += w[j] * powf((U0(x, y) - U),2); 
            //} else if (n == 1) {
                //e += w[j] * powf((U1(x, y) - U),2);
            //} else if (n == 2) {
                //e += w[j] * powf((U2(x, y) - U),2);
            //} else if (n == 3) {
                //e += w[j] * powf((U3(x, y) - U),2);
            //}
        }

        // store the result
        error[idx] = e;
    }
}

/* evaluate u velocity
 * 
 * evaluates u and v at the three vertex points for output
 * THREADS: num_elem
 */
__device__ void eval_u_velocity(double *c, double *c_rho,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p, int idx) {
    int i;
    double uv1, uv2, uv3;
    double rhov1, rhov2, rhov3;

    // calculate values at the integration points
    rhov1 = 0.;
    rhov2 = 0.;
    rhov3 = 0.;
    for (i = 0; i < n_p; i++) {
        rhov1 += c_rho[i] * basis_vertex[i * 3 + 0];
        rhov2 += c_rho[i] * basis_vertex[i * 3 + 1];
        rhov3 += c_rho[i] * basis_vertex[i * 3 + 2];
    }

    uv1 = 0.;
    uv2 = 0.;
    uv3 = 0.;
    for (i = 0; i < n_p; i++) {
        uv1 += c[i] * basis_vertex[i * 3 + 0];
        uv2 += c[i] * basis_vertex[i * 3 + 1];
        uv3 += c[i] * basis_vertex[i * 3 + 2];
    }

    uv1 = uv1 / rhov1;
    uv2 = uv2 / rhov2;
    uv3 = uv3 / rhov3;

    // store result
    Uv1[idx] = uv1;
    Uv2[idx] = uv2;
    Uv3[idx] = uv3;
}

/* check for convergence
 *
 * see if the difference in coefficients is less than the tolerance
 */
__global__ void check_convergence(double *c_prev, double *c, int num_elem, int n_p) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    c_prev[idx] = fabs(c[idx] - c_prev[idx]);
}
