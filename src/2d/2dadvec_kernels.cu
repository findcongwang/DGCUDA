
#include <hip/hip_runtime.h>
/* 2dadvec_kernels.cu
 *
 * This file contains the kernels for the 2D advection DG method.
 * We use K = number of elements
 * and    H = number of sides
 */

/***********************
 *
 * DEVICE VARIABLES
 *
 ***********************/
/* These are always prefixed with d_ for "device" */
float *d_c;      // holds coefficients for each element
float *d_rhs;    // the right hand side: 
                 //     we reset to 0 between each and add to this to build d_c
                 //     coefficients for each time step.

// runge kutta variables
float *d_kstar;
float *d_k1;
float *d_k2;
float *d_k3;
float *d_k4;

float *d_r1;     // integration points (x) for 2d integration
float *d_r2;     // integration points (y) for 2d integration
float *d_w;      // weights for 2d integration
float *d_oned_r; // integration points (x) for 1d integration
float *d_oned_w; // weights for 2d integration

// evaluation points for the boundary integrals depending on the side
float *d_s1_r1;
float *d_s1_r2;
float *d_s2_r1;
float *d_s2_r2;
float *d_s3_r1;
float *d_s3_r2;

// tells which side (1, 2, or 3) to evaluate this boundary integral over
int *d_left_side_number;
int *d_right_side_number;

float *d_J;     // jacobian determinant 
float *d_s_length; // length of sides

// the num_elem values of the x and y coordinates for the two vertices defining a side
// TODO: can i delete these after the lengths are precomputed?
//       maybe these should be in texture memory?
float *d_s_V1x;
float *d_s_V1y;
float *d_s_V2x;
float *d_s_V2y;

// the K indices of the sides for each element ranged 0->H-1
int *d_elem_s1;
int *d_elem_s2;
int *d_elem_s3;

// vertex x and y coordinates on the mesh which define an element
// TODO: can i delete these after the jacobians are precomputed?
//       maybe these should be in texture memory?
float *d_V1x;
float *d_V1y;
float *d_V2x;
float *d_V2y;
float *d_V3x;
float *d_V3y;

// normal vectors for the sides
float *d_Nx;
float *d_Ny;

// index lists for sides
int *d_left_elem;  // index of left  element for side idx
int *d_right_elem; // index of right element for side idx

/***********************
 *
 * DEVICE FUNCTIONS
 *
 ***********************/

/* flux function
 *
 * evaluates the flux f(u) at the point u.
 */
__device__ float flux_x(float u) {
    return u;
}
__device__ float flux_y(float u) {
    return u;
}

/* basis functions
 *
 */
__device__ float basis(float x, float y, int i) {
    switch (i) {
        case 0: return 1.41421356;
    }
    return -1;
}

/* basis function gradients
 *
 */
__device__ float grad_basis_x(float x, float y, int i) {
    switch (i) {
        case 0: return 0;
    }
    return -1;
}
__device__ float grad_basis_y(float x, float y, int i) {
    switch (i) {
        case 0: return 0;
    }
    return -1;
}

/* quadrature 
 *
 * uses gaussian quadrature to evaluate the integral over the 
 * element k. takes the coefficients for u_k in c, the integration 
 * points and weights r1, r2 and w, and the jacobian J.
 */
__device__ float quad(float *c, float *r1, float *r2, float *w, float J, int idx, int k, int N) {
    int i, j;
    float sum, u;
    register float register_c[10];

    sum = 0.0;
    for (i = 0; i < N; i++) {
        // Evaluate u at the integration point.
        u = 0;
        for (j = 0; j < N; j++) {
            u += c[j] * basis(r1[i], r2[i], j);
        }
        // Add to the sum
        sum += w[i] * (  flux_x(u) * grad_basis_x(r1[i], r2[i], k) 
                       + flux_y(u) * grad_basis_y(r1[i], r2[i], k));
    }

    // Multiply in the Jacobian
    return sum;
}

/* boundary elements
 *
 * does something to handle the boundary elements.
 */
__device__ float boundary(float *c, int k, int N) {
    return 0;
}

/* riemann solver
 *
 * evaluates the riemann problem over the boundary using Gaussian quadrature
 * with Legendre polynomials as basis functions.
 */
__device__ float riemann(float u_left, float u_right) {
    return 0.5 * (u_left + u_right);
}

/***********************
 *
 * INITIAL CONDITIONS
 *
 ***********************/

/* initial condition function
 *
 * returns the value of the intial condition at point x
 */
__device__ float u0(float x, float y) {
    return 1.;
}

/* initial conditions
 *
 * computes the coefficients for the initial conditions
 * THREADS: num_elem
 */
__global__ void init_conditions(float *c, 
                                float *V1x, float *V1y,
                                float *V2x, float *V2y,
                                float *V3x, float *V3y,
                                float *r1, float *r2,
                                float *w,
                                int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i, j;
    float x, y, u;

    for (i = 0; i < n_p + 1; i++) {
        u = 0;
        for (j = 0; j < n_p + 1; j++) {
            // map from the canonical element to the actual point on the mesh
            x = (1 - r1[j] - r2[j]) * V1x[idx] + r1[j] * V2x[idx] + r2[j]*V3x[idx];
            y = (1 - r1[j] - r2[j]) * V1y[idx] + r1[j] * V2y[idx] + r2[j]*V3y[idx];
            // evaluate u there
            u += w[j] * u0(x, y) * basis(r1[j], r2[j], i);
        }
        c[i*num_elem + idx] = (2.*i + 1.) / 2. * u;
    }
}

/***********************
 *
 * PRECOMPUTING
 *
 ***********************/

/* side length computer
 *
 * precomputes the length of each side.
 * THREADS: num_sides
 */ 
__global__ void preval_side_length(float *s_length, 
                              float *s_V1x, float *s_V1y, 
                              float *s_V2x, float *s_V2y) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // compute and store the length of the side
    s_length[idx] = sqrtf(powf(s_V1x[idx] - s_V2x[idx],2) + powf(s_V1y[idx] - s_V2y[idx],2));
}

/* jacobian computing
 *
 * precomputes the jacobian determinant for each element.
 * THREADS: num_elem
 */
__global__ void preval_jacobian(float *J, 
                           float *V1x, float *V1y, 
                           float *V2x, float *V2y, 
                           float *V3x, float *V3y) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    float x1, y1, x2, y2, x3, y3;

    // read vertex points
    x1 = V1x[idx];
    y1 = V1y[idx];
    x2 = V2x[idx];
    y2 = V2y[idx];
    x3 = V3x[idx];
    y3 = V3y[idx];

    // calculate jacobian determinant
    J[idx] = (-x1 + x2) * (-y1 + y3) - (-x1 + x3) * (-y1 + y2);
}

/* evaluate normal vectors
 *
 * computes the normal vectors for each element along each side.
 * THREADS: num_sides
 *
 * TODO: what the hell direction does this point? somehow i need to always
 *       make them point out of the cell, so... remember somehow?
 */
__global__ void preval_normals(float *Nx, float *Ny, 
                          float *s_V1x, float *s_V1y, 
                          float *s_V2x, float *s_V2y,
                          float *V1x, float *V1y, 
                          float *V2x, float *V2y, 
                          float *V3x, float *V3y,
                          int *left_elem) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   float x, y, length;
   float v1x, v1y, v2x, v2y, v3x, v3y;
   float sv1x, sv1y, sv2x, sv2y;
   float dot, left_x, left_y;
   int left_idx;

   // read in global data
   left_idx = left_elem[idx];
   v1x = V1x[left_idx];
   v1y = V1y[left_idx];
   v2x = V2x[left_idx];
   v2y = V2y[left_idx];
   v3x = V3x[left_idx];
   v3y = V3y[left_idx];
   sv1x = s_V1x[idx];
   sv1y = s_V1y[idx];
   sv2x = s_V2x[idx];
   sv2y = s_V2y[idx];

   // lengths of the vector components
   x = sv1x - sv2x;
   y = sv1y - sv2y;

   // normalize
   length = sqrtf(powf(x, 2) + powf(y, 2));

   // make it point the correct direction by learning the third vertex point
   // coordinates from the left element
   if  ((v1x == sv1x && v1y == sv1y && v2x == sv2x && v2y == sv2y) ||
        (v1x == sv2x && v1y == sv2y && v2x == sv1x && v2y == sv1y)) {
       left_x = v3x;
       left_y = v3y;
   }
   else if  ((v2x == sv1x && v2y == sv1y && v3x == sv2x && v3y == sv2y) ||
             (v2x == sv2x && v2y == sv2y && v3x == sv1x && v3y == sv1y)) {
       left_x = v1x;
       left_y = v1y;
   }
   // could just be else
   else if  ((v1x == sv1x && v1y == sv1y && v3x == sv2x && v3y == sv2y) ||
             (v1x == sv2x && v1y == sv2y && v3x == sv1x && v3y == sv1y)) {
       left_x = v2x;
       left_y = v2y;
   }

   // find the dot product between the normal vector and the third vetrex point
   dot = -y*left_x + x*left_y;

   // correct the direction
   length = (dot < 0) ? -length : length;

   // store the result
   Nx[idx] = -y / length;
   Ny[idx] =  x / length;
}

/***********************
 *
 * MAIN FUNCTIONS
 *
 ***********************/

/* flux evaluation
 *
 * evaluate all the riemann problems for each element.
 * THREADS: num_sides
 */
__global__ void eval_riemann(float *c, float *rhs, 
                        float *J, float *s_length,
                        float *s1_r1, float *s1_r2,
                        float *s2_r1, float *s2_r2,
                        float *s3_r1, float *s3_r2,
                        float *oned_r, float *oned_w,
                        int *left_idx_list, int *right_idx_list,
                        int *left_side_number, int *right_side_number, 
                        float *Nx, float *Ny, int n_p, int num_sides, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        int left_idx, right_idx, right_side, left_side, i, j;
        float c_left[10], c_right[10];
        float left_r1[10], right_r1[10];
        float left_r2[10], right_r2[10];
        float nx, ny, s;
        float u_left, u_right;
        float len, left_sum, right_sum;

        // find the left and right elements
        left_idx  = left_idx_list[idx];
        right_idx = right_idx_list[idx];

        // get the length of the side
        len = s_length[idx];

        // get the normal vector for this side
        nx = Nx[idx];
        ny = Ny[idx];

        // grab the coefficients for the left & right elements
        if (right_idx != -1) {
            // not a boundary side
            for (i = 0; i < (n_p + 1); i++) {
                c_left[i]  = c[i*num_elem + left_idx];
                c_right[i] = c[i*num_elem + right_idx];
            }
        } else {
            // this is a boundary side
            for (i = 0; i < (n_p + 1); i++) {
                c_left[i]  = c[i*num_elem + left_idx];
                c_right[i] = 0;
            }
        }

        //TODO: does this speed it up
        __syncthreads();

        // need to find out what side we've got for evaluation (right, left, bottom)
        left_side  = left_side_number [idx];
        right_side = right_side_number[idx];

        // get the integration points for the left element's side
        switch (left_side) {
            case 1: 
                for (i = 0; i < (n_p + 1); i++) {
                    left_r1[i] = s1_r1[i];
                    left_r2[i] = s1_r2[i];
                }
                break;
            case 2: 
                for (i = 0; i < (n_p + 1); i++) {
                    left_r1[i] = s2_r1[i];
                    left_r2[i] = s2_r2[i];
                }
                break;
            case 3: 
                for (i = 0; i < (n_p + 1); i++) {
                    left_r1[i] = s3_r1[i];
                    left_r2[i] = s3_r2[i];
                }
                break;
        }

        // TODO: does this speed it up?
        __syncthreads();
         
        // get the integration points for the right element's side
        switch (right_side) {
            case 1: 
                for (i = 0; i < (n_p + 1); i++) {
                    right_r1[i] = s1_r1[i];
                    right_r2[i] = s1_r2[i];
                }
                break;
            case 2: 
                for (i = 0; i < (n_p + 1); i++) {
                    right_r1[i] = s2_r1[i];
                    right_r2[i] = s2_r2[i];
                }
                break;
            case 3: 
                for (i = 0; i < (n_p + 1); i++) {
                    right_r1[i] = s3_r1[i];
                    right_r2[i] = s3_r2[i];
                }
                break;
        }

        // TODO: does this speed it up?
        __syncthreads();
         
        // evaluate the polynomial over that side for both elements and add the result to rhs
        for (i = 0; i < (n_p + 1); i++) {
            u_left  = 0;
            u_right = 0;
            left_sum  = 0;
            right_sum = 0;

            // compute u evaluated over the integration point
            for (j = 0; j < (n_p + 1); j++) {
                u_left  += c_left[i]  * basis(left_r1[i], left_r2[i], j) * oned_w[i];
                u_right += c_right[i] * basis(right_r1[i], right_r2[i], j) * oned_w[i];
            }

            // solve the Riemann problem at this integration point
            s = riemann(u_left, u_right);

            // calculate the quadrature over [-1,1] for these sides
            for (j = 0; j < (n_p + 1); j++) {
                left_sum  += (nx * flux_x(s) + ny * flux_y(s)) * oned_w[i] * basis(left_r1[i],  left_r2[i],  j);
                right_sum += (nx * flux_x(s) + ny * flux_y(s)) * oned_w[i] * basis(right_r1[i], right_r2[i], j);
            }
            
            // add each side's contribution to the rhs vector
            rhs[i*num_elem + left_idx]  += nx + ny;//len / 2. * left_sum;
            // normal points from left to right
            if (right_idx != -1) {
                rhs[i*num_elem + right_idx] -= nx + ny;//len / 2. * right_sum;
            }
        }
    }
}

/* volume integrals
 *
 * evaluates and adds the volume integral to the rhs vector
 * THREADS: K
 */
 __global__ void eval_quad(float *c, float *rhs, 
                     float *r1, float *r2, float *w, float *J, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        int i;
        float quad_u, register_J;
        float register_c[10];

        // Get the coefficients for this element
        for (i = 0; i < (n_p + 1); i++) {
            register_c[i] = c[i*num_elem + idx];
        }
         
        // Grab the Jacobian
        register_J = J[idx];

        for (i = 0; i < (n_p + 1); i++) {
            // Evaluate the volume integral
            quad_u = quad(register_c, r1, r2, w, register_J, idx, i, (n_p + 1));

            // add the volume contribution result to the rhs
            rhs[i*num_elem + idx] += -quad_u;
            rhs[i*num_elem + idx] /= register_J;
        }
    }
}

/* right hand side
 *
 * stores the computed rhs vector into c and then resets it 0.
 */
__global__ void eval_rhs(float *c, float *rhs, float dt, int num_rhs) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_rhs) {
        c[idx]   = dt * rhs[idx];
        rhs[idx] = 0;
    }
}

/***********************
 *
 * TIME INTEGRATION
 *
 ***********************/

/* tempstorage for RK4
 * 
 * I need to store u + alpha * k_i into some temporary variable called k*.
 */
__global__ void rk4_tempstorage(float *c, float *kstar, float*k, float alpha, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < (n_p + 1) * num_elem) {
        kstar[idx] = c[idx] + alpha * k[idx];
    }
}

/* rk4
 *
 * computes the runge-kutta solution 
 * u_n+1 = u_n + k1/6 + k2/3 + k3/3 + k4/6
 */
__global__ void rk4(float *c, float *k1, float *k2, float *k3, float *k4, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < (n_p + 1) * num_elem) {
        c[idx] += k1[idx]/6. + k2[idx]/3. + k3[idx]/3. + k4[idx]/6.;
    }
}

