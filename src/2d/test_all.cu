#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "2dadvec.cu"

#define TOL 1E-5
#define MAX_ALPHA 3

__device__ float *d_error;

/* test_all.cu
 * 
 * this is the standard test file for dgcuda.
 */

/* initial conditions
 *
 * find the initial projection for (x - y)^alpha
 * THREADS: num_elem
 */
__global__ void init_conditions_alpha(float *c, float *J,
                                float *V1x, float *V1y,
                                float *V2x, float *V2y,
                                float *V3x, float *V3y,
                                int n_quad, int n_p, int num_elem, int alpha) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i, j;
    float x, y, u;

    if (idx < num_elem) {
        for (i = 0; i < n_p; i++) {
            u = 0.;
            // perform quadrature
            for (j = 0; j < n_quad; j++) {
                // map from the canonical element to the actual point on the mesh
                // x = x2 * r + x3 * s + x1 * (1 - r - s)
                x = r1[j] * V2x[idx] + r2[j] * V3x[idx] + (1 - r1[j] - r2[j]) * V1x[idx];
                y = r1[j] * V2y[idx] + r2[j] * V3y[idx] + (1 - r1[j] - r2[j]) * V1y[idx];

                // evaluate u there
                u += w[j] * pow(x - y, alpha) * basis[i * n_quad + j];
            }
            c[i * num_elem + idx] = u;
        } 
    }
}

/* evaluate error
 * 
 * evaluates u at the three vertex points for output
 * THREADS: num_elem
 */
__global__ void eval_error_alpha(float *c, 
                       float *V1x, float *V1y,
                       float *V2x, float *V2y,
                       float *V3x, float *V3y,
                       float *error, 
                       int num_elem, int n_p, int n_quad, float alpha) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elem) {
        int i, j;
        float u, sum;
        float x, y;

        sum = 0.;
        for (j = 0; j < n_quad; j++) {
            u = 0.;
            for (i = 0; i < n_p; i++) {
                u += c[i * num_elem + idx] * basis[n_quad * i + j];
            }
            // x = x2 * r + x3 * s + x1 * (1 - r - s)
            x = r1[j] * V2x[idx] + r2[j] * V3x[idx] + (1 - r1[j] - r2[j]) * V1x[idx];
            y = r1[j] * V2y[idx] + r2[j] * V3y[idx] + (1 - r1[j] - r2[j]) * V1y[idx];

            sum += w[j] * (pow(x - y, alpha) - u);
        }
        // store error
        error[idx] = abs(sum);
    }
}

int test_initial_projection(int n, int alpha, FILE *mesh_file, FILE *out_file) {
    checkCudaError("error before start.");
    int num_elem, num_sides;
    int n_threads, n_blocks_elem;
    int i, n_p, n_quad, n_quad1d;

    float total_error; 
    float *V1x, *V1y, *V2x, *V2y, *V3x, *V3y;
    float *sides_x1, *sides_x2;
    float *sides_y1, *sides_y2;

    float *r1_local, *r2_local, *w_local;

    float *s_r, *oned_w_local;

    int *left_elem, *right_elem;
    int *elem_s1, *elem_s2, *elem_s3;
    int *left_side_number, *right_side_number;

    char line[100];

    float *error;
    float *Uv1, *Uv2, *Uv3;

    void (*eval_u_ftn)(float*, float*, float*, float*, int, int) = NULL;

    // set the order of the approximation & timestep
    n_p = (n + 1) * (n + 2) / 2;

    // get the number of elements in the mesh
    fgets(line, 100, mesh_file);
    sscanf(line, "%i", &num_elem);

    // allocate vertex points
    V1x = (float *) malloc(num_elem * sizeof(float));
    V1y = (float *) malloc(num_elem * sizeof(float));
    V2x = (float *) malloc(num_elem * sizeof(float));
    V2y = (float *) malloc(num_elem * sizeof(float));
    V3x = (float *) malloc(num_elem * sizeof(float));
    V3y = (float *) malloc(num_elem * sizeof(float));

    elem_s1 = (int *) malloc(num_elem * sizeof(int));
    elem_s2 = (int *) malloc(num_elem * sizeof(int));
    elem_s3 = (int *) malloc(num_elem * sizeof(int));

    // TODO: these are too big; should be a way to figure out how many we actually need
    left_side_number  = (int *)   malloc(3*num_elem * sizeof(int));
    right_side_number = (int *)   malloc(3*num_elem * sizeof(int));

    sides_x1    = (float *) malloc(3*num_elem * sizeof(float));
    sides_x2    = (float *) malloc(3*num_elem * sizeof(float));
    sides_y1    = (float *) malloc(3*num_elem * sizeof(float));
    sides_y2    = (float *) malloc(3*num_elem * sizeof(float)); 
    left_elem   = (int *) malloc(3*num_elem * sizeof(int));
    right_elem  = (int *) malloc(3*num_elem * sizeof(int));

    for (i = 0; i < 3*num_elem; i++) {
        right_elem[i] = -1;
    }

    // read in the mesh and make all the mappings
    read_mesh(mesh_file, &num_sides, num_elem,
                         V1x, V1y, V2x, V2y, V3x, V3y,
                         left_side_number, right_side_number,
                         sides_x1, sides_y1, 
                         sides_x2, sides_y2, 
                         elem_s1, elem_s2, elem_s3,
                         left_elem, right_elem);

    // initialize the gpu
    init_gpu(num_elem, num_sides, n_p,
             V1x, V1y, V2x, V2y, V3x, V3y,
             left_side_number, right_side_number,
             sides_x1, sides_y1,
             sides_x2, sides_y2, 
             elem_s1, elem_s2, elem_s3,
             left_elem, right_elem);
    checkCudaError("error after gpu init.");

    n_threads        = 128;
    n_blocks_elem    = (num_elem  / n_threads) + ((num_elem  % n_threads) ? 1 : 0);

    // pre computations
    checkCudaError("error after prevals.");

    // get the correct quadrature rules for this scheme
    set_quadrature(n, &r1_local, &r2_local, &w_local, 
                   &s_r, &oned_w_local, &n_quad, &n_quad1d);

    preval_basis(r1_local, r2_local, s_r, w_local, oned_w_local, n_quad, n_quad1d, n_p);

    // initial conditions
    init_conditions_alpha<<<n_blocks_elem, n_threads>>>(d_c, d_J, d_V1x, d_V1y, d_V2x, d_V2y, d_V3x, d_V3y,
                    n_quad, n_p, num_elem, alpha);
    checkCudaError("error after initial conditions.");

    hipDeviceSynchronize();
    hipMalloc((void **) &d_error, num_elem * sizeof(float));
    eval_error_alpha<<<n_blocks_elem, n_threads>>>(d_c, d_V1x, d_V1y, 
                                                   d_V2x, d_V2y, d_V3x, d_V3y,
                                                   d_error, num_elem, n_p, n_quad, alpha);

    switch (n) {
        case 0: eval_u_ftn = eval_u_wrapper0;
                break;
        case 1: eval_u_ftn = eval_u_wrapper1;
                break;
        case 2: eval_u_ftn = eval_u_wrapper2;
                break;
        case 3: eval_u_ftn = eval_u_wrapper3;
                break;
        case 4: eval_u_ftn = eval_u_wrapper4;
                break;
        case 5: eval_u_ftn = eval_u_wrapper5;
                break;
        case 6: eval_u_ftn = eval_u_wrapper6;
                break;
        case 7: eval_u_ftn = eval_u_wrapper7;
                break;
    }

    // evaluate at the vertex points and copy over data
    Uv1 = (float *) malloc(num_elem * sizeof(float));
    Uv2 = (float *) malloc(num_elem * sizeof(float));
    Uv3 = (float *) malloc(num_elem * sizeof(float));
    eval_u_ftn<<<n_blocks_elem, n_threads>>>(d_c, d_Uv1, d_Uv2, d_Uv3, num_elem, n_p);
    hipMemcpy(Uv1, d_Uv1, num_elem * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Uv2, d_Uv2, num_elem * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Uv3, d_Uv3, num_elem * sizeof(float), hipMemcpyDeviceToHost);

    // get the L1 error
    error = (float *) malloc(num_elem * sizeof(float));
    hipMemcpy(error, d_error, num_elem * sizeof(float), hipMemcpyDeviceToHost);

    // get the total L1 error
    total_error = 0.;
    for (i = 0; i < num_elem; i++) {
        total_error += error[i];
    }

    // write data to file
    // TODO: this will output multiple vertices values. does gmsh care? i dunno...
    fprintf(out_file, "View \"Exported field \" {\n");
    for (i = 0; i < num_elem; i++) {
        fprintf(out_file, "ST (%lf,%lf,0,%lf,%lf,0,%lf,%lf,0) {%lf,%lf,%lf};\n", 
                               V1x[i], V1y[i], V2x[i], V2y[i], V3x[i], V3y[i],
                               Uv1[i], Uv2[i], Uv3[i]);
    }

    // close the output file
    fprintf(out_file,"};");

    // free variables
    free_gpu();
    
    free(error);

    free(V1x);
    free(V1y);
    free(V2x);
    free(V2y);
    free(V3x);
    free(V3y);

    free(elem_s1);
    free(elem_s2);
    free(elem_s3);

    free(sides_x1);
    free(sides_x2);
    free(sides_y1);
    free(sides_y2);

    free(left_elem);
    free(right_elem);
    free(left_side_number);
    free(right_side_number);

    free(r1_local);
    free(r2_local);
    free(w_local);
    free(s_r);
    free(oned_w_local);
    
    printf(" (%lf) ", total_error);
    if (total_error < TOL) {
        return 0;
    } else {
        return 1;
    }
}

int test_timestep(int n, int alpha, int timesteps, float dt, FILE *mesh_file, FILE *out_file) {
    checkCudaError("error before start.");
    int num_elem, num_sides;
    int n_threads, n_blocks_elem, n_blocks_sides;
    int i, n_p, t, n_quad, n_quad1d;

    float *V1x, *V1y, *V2x, *V2y, *V3x, *V3y;
    float *sides_x1, *sides_x2;
    float *sides_y1, *sides_y2;

    float *r1_local, *r2_local, *w_local;

    float *s_r, *oned_w_local;

    int *left_elem, *right_elem;
    int *elem_s1, *elem_s2, *elem_s3;
    int *left_side_number, *right_side_number;

    char line[100];

    float total_error;
    float *error;
    float *Uv1, *Uv2, *Uv3;

    void (*eval_u_ftn)(float*, float*, float*, float*, int, int) = NULL;

    // set the order of the approximation & timestep
    n_p = (n + 1) * (n + 2) / 2;

    // open the mesh to get num_elem for allocations
    if (!mesh_file) {
        printf("\nERROR: mesh file not found.\n");
        return 1;
    }
    fgets(line, 100, mesh_file);
    sscanf(line, "%i", &num_elem);

    // allocate vertex points
    V1x = (float *) malloc(num_elem * sizeof(float));
    V1y = (float *) malloc(num_elem * sizeof(float));
    V2x = (float *) malloc(num_elem * sizeof(float));
    V2y = (float *) malloc(num_elem * sizeof(float));
    V3x = (float *) malloc(num_elem * sizeof(float));
    V3y = (float *) malloc(num_elem * sizeof(float));

    elem_s1 = (int *) malloc(num_elem * sizeof(int));
    elem_s2 = (int *) malloc(num_elem * sizeof(int));
    elem_s3 = (int *) malloc(num_elem * sizeof(int));

    // TODO: these are too big; should be a way to figure out how many we actually need
    left_side_number  = (int *)   malloc(3*num_elem * sizeof(int));
    right_side_number = (int *)   malloc(3*num_elem * sizeof(int));

    sides_x1    = (float *) malloc(3*num_elem * sizeof(float));
    sides_x2    = (float *) malloc(3*num_elem * sizeof(float));
    sides_y1    = (float *) malloc(3*num_elem * sizeof(float));
    sides_y2    = (float *) malloc(3*num_elem * sizeof(float)); 
    left_elem   = (int *) malloc(3*num_elem * sizeof(int));
    right_elem  = (int *) malloc(3*num_elem * sizeof(int));

    for (i = 0; i < 3*num_elem; i++) {
        right_elem[i] = -1;
    }

    // read in the mesh and make all the mappings
    read_mesh(mesh_file, &num_sides, num_elem,
                         V1x, V1y, V2x, V2y, V3x, V3y,
                         left_side_number, right_side_number,
                         sides_x1, sides_y1, 
                         sides_x2, sides_y2, 
                         elem_s1, elem_s2, elem_s3,
                         left_elem, right_elem);

    // initialize the gpu
    init_gpu(num_elem, num_sides, n_p,
             V1x, V1y, V2x, V2y, V3x, V3y,
             left_side_number, right_side_number,
             sides_x1, sides_y1,
             sides_x2, sides_y2, 
             elem_s1, elem_s2, elem_s3,
             left_elem, right_elem);

    checkCudaError("error after gpu init.");
    n_threads        = 128;
    n_blocks_elem    = (num_elem  / n_threads) + ((num_elem  % n_threads) ? 1 : 0);
    n_blocks_sides   = (num_sides / n_threads) + ((num_sides % n_threads) ? 1 : 0);

    // pre computations
    preval_jacobian<<<n_blocks_elem, n_threads>>>(d_J, d_V1x, d_V1y, d_V2x, d_V2y, d_V3x, d_V3y, num_elem); 
    hipDeviceSynchronize();
    preval_side_length<<<n_blocks_sides, n_threads>>>(d_s_length, d_s_V1x, d_s_V1y, d_s_V2x, d_s_V2y, 
                                                      num_sides); 
    hipDeviceSynchronize();
    preval_normals<<<n_blocks_sides, n_threads>>>(d_Nx, d_Ny, 
                                                  d_s_V1x, d_s_V1y, d_s_V2x, d_s_V2y,
                                                  d_V1x, d_V1y, 
                                                  d_V2x, d_V2y, 
                                                  d_V3x, d_V3y, 
                                                  d_left_side_number, num_sides); 
    hipDeviceSynchronize();
    preval_normals_direction<<<n_blocks_sides, n_threads>>>(d_Nx, d_Ny, 
                                                  d_V1x, d_V1y, 
                                                  d_V2x, d_V2y, 
                                                  d_V3x, d_V3y, 
                                                  d_left_elem, d_left_side_number, num_sides); 
    preval_partials<<<n_blocks_elem, n_threads>>>(d_V1x, d_V1y,
                                                  d_V2x, d_V2y,
                                                  d_V3x, d_V3y,
                                                  d_xr,  d_yr,
                                                  d_xs,  d_ys, num_elem);
    hipDeviceSynchronize();
    checkCudaError("error after prevals.");

    // get the correct quadrature rules for this scheme
    set_quadrature(n, &r1_local, &r2_local, &w_local, 
                   &s_r, &oned_w_local, &n_quad, &n_quad1d);

    // evaluate the basis functions at those points and store on GPU
    preval_basis(r1_local, r2_local, s_r, w_local, oned_w_local, n_quad, n_quad1d, n_p);
    hipDeviceSynchronize();

    // initial conditions
    init_conditions_alpha<<<n_blocks_elem, n_threads>>>(d_c, d_J, d_V1x, d_V1y, d_V2x, d_V2y, d_V3x, d_V3y,
                    n_quad, n_p, num_elem, alpha);
    checkCudaError("error after initial conditions.");

    fprintf(out_file, "View \"Exported field \" {\n");

    time_integrate_rk4(dt, n_quad, n_quad1d, n_p, n, num_elem, num_sides, 0, alpha, timesteps);
    t = timesteps * dt;

    // evaluate at the vertex points and copy over data
    Uv1 = (float *) malloc(num_elem * sizeof(float));
    Uv2 = (float *) malloc(num_elem * sizeof(float));
    Uv3 = (float *) malloc(num_elem * sizeof(float));

    switch (n) {
        case 0: eval_u_ftn = eval_u_wrapper0;
                break;
        case 1: eval_u_ftn = eval_u_wrapper1;
                break;
        case 2: eval_u_ftn = eval_u_wrapper2;
                break;
        case 3: eval_u_ftn = eval_u_wrapper3;
                break;
        case 4: eval_u_ftn = eval_u_wrapper4;
                break;
        case 5: eval_u_ftn = eval_u_wrapper5;
                break;
        case 6: eval_u_ftn = eval_u_wrapper6;
                break;
        case 7: eval_u_ftn = eval_u_wrapper7;
                break;
    }

    // get the L1 error
    error = (float *) malloc(num_elem * sizeof(float));
    hipMalloc((void **) &d_error, num_elem * sizeof(float));
    eval_error_alpha<<<n_blocks_elem, n_threads>>>(d_c, d_V1x, d_V1y, 
                                                   d_V2x, d_V2y, d_V3x, d_V3y,
                                                   d_error, num_elem, n_p, n_quad, alpha);

    hipMemcpy(error, d_error, num_elem * sizeof(float), hipMemcpyDeviceToHost);

    // get the total L1 error
    total_error = 0.;
    for (i = 0; i < num_elem; i++) {
        total_error += error[i];
    }

 
    eval_u_ftn<<<n_blocks_elem, n_threads>>>(d_c, d_Uv1, d_Uv2, d_Uv3, num_elem, n_p);
    hipDeviceSynchronize();
    hipMemcpy(Uv1, d_Uv1, num_elem * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Uv2, d_Uv2, num_elem * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Uv3, d_Uv3, num_elem * sizeof(float), hipMemcpyDeviceToHost);

    // write data to file
    // TODO: this will output multiple vertices values. does gmsh care? i dunno...
    for (i = 0; i < num_elem; i++) {
        fprintf(out_file, "ST (%lf,%lf,0,%lf,%lf,0,%lf,%lf,0) {%lf,%lf,%lf};\n", 
                               V1x[i], V1y[i], V2x[i], V2y[i], V3x[i], V3y[i],
                               Uv1[i], Uv2[i], Uv3[i]);
    }

    fprintf(out_file,"};");

    // free variables
    free_gpu();
    
    free(Uv1);
    free(Uv2);
    free(Uv3);

    free(V1x);
    free(V1y);
    free(V2x);
    free(V2y);
    free(V3x);
    free(V3y);

    free(elem_s1);
    free(elem_s2);
    free(elem_s3);

    free(sides_x1);
    free(sides_x2);
    free(sides_y1);
    free(sides_y2);

    free(left_elem);
    free(right_elem);
    free(left_side_number);
    free(right_side_number);

    free(error);

    printf(" (%lf) ", total_error);
    if (total_error < TOL) {
        return 0;
    } else {
        return 1;
    }
}

void run_initial_projection_tests() {
    int alpha, n;
    FILE *mesh, *out;
    printf("*************************\n");
    printf("* INITIAL PROJECTION\n");
    printf("*************************\n");
    ///////////////
    // CANONICAL
    ///////////////
    for (alpha = 0; alpha < MAX_ALPHA; alpha++) {
        printf("*************************\n");
        printf("* u(x, y) = (x - y)^%i\n", alpha);
        printf("*************************\n");
        printf("    Testing canonical...\n");

        char outfilename[100];
        for (n = alpha; n < 6; n++) {
            sprintf(outfilename, "tests/t0/alpha%i_canonical%i.out", alpha, n);
            printf(outfilename);
            out  = fopen(outfilename, "w");
            mesh = fopen("mesh/canonical.pmsh", "r");
            printf("     > n = %i : ", n);

            if (!test_initial_projection(n, alpha, mesh, out)) {
                printf(" pass\n");
            } else {
                printf(" FAIL\n");
            }
            fclose(mesh);
            fclose(out);
        }
    }

    ///////////////
    // BOX
    ///////////////
    for (alpha = 0; alpha < MAX_ALPHA; alpha++) {
        printf("*************************\n");
        printf("* u(x, y) = (x - y)^%i\n", alpha);
        printf("*************************\n");
        printf("    Testing box...\n");

        char outfilename[100];
        for (n = alpha; n < 6; n++) {
            sprintf(outfilename, "tests/t0/alpha%i_box%i.out", alpha, n);
            printf(outfilename);
            out  = fopen(outfilename, "w");
            mesh = fopen("mesh/box.pmsh", "r");
            printf("     > n = %i : ", n);

            if (!test_initial_projection(n, alpha, mesh, out)) {
                printf(" pass\n");
            } else {
                printf(" FAIL\n");
            }
            fclose(mesh);
            fclose(out);
        }
    }

    ///////////////
    // SUPERSIMPLE
    ///////////////
    for (alpha = 0; alpha < MAX_ALPHA; alpha++) {
        printf("*************************\n");
        printf("* u(x, y) = (x - y)^%i\n", alpha);
        printf("*************************\n");
        printf("    Testing supersimple...\n");

        char outfilename[100];
        for (n = alpha; n < 6; n++) {
            sprintf(outfilename, "tests/t0/alpha%i_supersimple%i.out", alpha, n);
            printf(outfilename);
            out  = fopen(outfilename, "w");
            mesh = fopen("mesh/supersimple.pmsh", "r");
            printf("     > n = %i : ", n);

            if (!test_initial_projection(n, alpha, mesh, out)) {
                printf(" pass\n");
            } else {
                printf(" FAIL\n");
            }
            fclose(mesh);
            fclose(out);
        }
    }

    ///////////////
    // UNIFORM
    ///////////////
    for (alpha = 0; alpha < MAX_ALPHA; alpha++) {
        printf("*************************\n");
        printf("* u(x, y) = (x - y)^%i\n", alpha);
        printf("*************************\n");
        printf("    Testing uniform...\n");

        char outfilename[100];
        for (n = alpha; n < 6; n++) {
            sprintf(outfilename, "tests/t0/alpha%i_uniform%i.out", alpha, n);
            printf(outfilename);
            out  = fopen(outfilename, "w");
            mesh = fopen("mesh/uniform.pmsh", "r");
            printf("     > n = %i : ", n);

            if (!test_initial_projection(n, alpha, mesh, out)) {
                printf(" pass\n");
            } else {
                printf(" FAIL\n");
            }
            fclose(mesh);
            fclose(out);
        }
    }
}

void run_timestep_tests() {
    int alpha, n, timesteps;
    float dt;
    FILE *mesh, *out;

    for (timesteps = 1; timesteps < 1000; timesteps *= 10) {
        printf("*************************\n");
        printf("* TIMESTEPS : %i\n", timesteps);
        printf("*************************\n");

        ///////////////
        // CANONICAL
        ///////////////
        for (alpha = 0; alpha < MAX_ALPHA; alpha++) {
            printf("*************************\n");
            printf("* u(x, y) = (x - y)^%i\n", alpha);
            printf("*************************\n");
            printf("    Testing canonical...\n");
            dt = 0.001;

            char outfilename[100];
            for (n = alpha; n < 6; n++) {
                sprintf(outfilename, "tests/t%i/alpha%i_canonical%i.out", timesteps, alpha, n);
                printf(outfilename);
                out  = fopen(outfilename, "w");
                mesh = fopen("mesh/canonical.pmsh", "r");
                printf(" > n = %i : ", n);

                if (!test_timestep(n, alpha, timesteps, dt, mesh, out)) {
                    printf(" pass\n");
                } else {
                    printf(" FAIL\n");
                }
                fclose(mesh);
                fclose(out);
            }
        }
        ///////////////
        // BOX
        ///////////////
        for (alpha = 0; alpha < MAX_ALPHA; alpha++) {
            printf("*************************\n");
            printf("* u(x, y) = (x - y)^%i\n", alpha);
            printf("*************************\n");
            printf("    Testing box...\n");
            dt = 0.01;

            char outfilename[100];
            for (n = alpha; n < 6; n++) {
                sprintf(outfilename, "tests/t%i/alpha%i_box%i.out", timesteps, alpha, n);
                printf(outfilename);
                out  = fopen(outfilename, "w");
                mesh = fopen("mesh/box.pmsh", "r");
                printf(" > n = %i : ", n);

                if (!test_timestep(n, alpha, timesteps, dt, mesh, out)) {
                    printf(" pass\n");
                } else {
                    printf(" FAIL\n");
                }
                fclose(mesh);
                fclose(out);
            }
        }
        ///////////////
        // UNIFORM
        ///////////////
        for (alpha = 0; alpha < MAX_ALPHA; alpha++) {
            printf("*************************\n");
            printf("* u(x, y) = (x - y)^%i\n", alpha);
            printf("*************************\n");
            printf("    Testing uniform...\n");
            dt = 0.01;

            char outfilename[100];
            for (n = alpha; n < 6; n++) {
                sprintf(outfilename, "tests/t%i/alpha%i_uniform%i.out", timesteps, alpha, n);
                printf(outfilename);
                out  = fopen(outfilename, "w");
                mesh = fopen("mesh/uniform.pmsh", "r");
                printf(" > n = %i : ", n);

                if (!test_timestep(n, alpha, timesteps, dt, mesh, out)) {
                    printf(" pass\n");
                } else {
                    printf(" FAIL\n");
                }
                fclose(mesh);
                fclose(out);
            }
        }
    }
}

int main() {
    //run_initial_projection_tests();
    run_timestep_tests();
}
