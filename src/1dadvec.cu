#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include "1dadvec_kernels.cu"

void checkCudaError(const char *message)
{
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
        exit(-1);
    }
}

/* integrate in time
 *
 * take one time step; calls the kernel functions to compute in parallel.
 */
void timeIntegrate(float *u, float a, int K, float dt, float dx, double t, int Np) {
    int size = K * Np;

    int nThreads = 128;

    int nBlocksRHS   = K / nThreads + ((K % nThreads) ? 1 : 0);
    int nBlocksFlux  = (K + 1) / nThreads + (((K + 1) % nThreads) ? 1 : 0);
    int nBlocksRK    = (Np*K) / nThreads + (((Np* K) % nThreads) ? 1 : 0);

    // Stage 1
    // f <- flux(u)
    calcFlux<<<nBlocksFlux, nThreads>>>(d_u, d_f, a, t, K, Np);
    hipDeviceSynchronize();
    // k1 <- dt*rhs(u)
    rhs<<<nBlocksRHS, nThreads>>>(d_u, d_k1, d_f, d_w, d_r, a, dt, dx, K, Np);
    hipDeviceSynchronize();
    // k* <- u + k1/2
    rk4_tempstorage<<<nBlocksRK, nThreads>>>(d_u, d_kstar, d_k1, 0.5, dt, Np, K);
    hipDeviceSynchronize();

    // Stage 2
    // f <- flux(k*)
    calcFlux<<<nBlocksFlux, nThreads>>>(d_kstar, d_f, a, t, K, Np);
    hipDeviceSynchronize();
    // k2 <- dt*rhs(k*)
    rhs<<<nBlocksRHS, nThreads>>>(d_kstar, d_k2, d_f, d_r, d_x, a, dt, dx, K, Np);
    hipDeviceSynchronize();
    // k* <- u + k2/2
    rk4_tempstorage<<<nBlocksRK, nThreads>>>(d_u, d_kstar, d_k2, 0.5, dt, Np, K);
    hipDeviceSynchronize();

    // Stage 3
    // f <- flux(k*)
    calcFlux<<<nBlocksFlux, nThreads>>>(d_kstar, d_f, a, t, K, Np);
    hipDeviceSynchronize();
    // k3 <- dt*rhs(k*)
    rhs<<<nBlocksRHS, nThreads>>>(d_kstar, d_k3, d_f, d_r, d_x, a, dt, dx, K, Np);
    hipDeviceSynchronize();
    // k* <- u + k3
    rk4_tempstorage<<<nBlocksRK, nThreads>>>(d_u, d_kstar, d_k3, 1, dt, Np, K);
    hipDeviceSynchronize();

    // Stage 4
    // f <- flux(k*)
    calcFlux<<<nBlocksFlux, nThreads>>>(d_kstar, d_f, a, t, K, Np);
    hipDeviceSynchronize();
    // k4 <- dt*rhs(k*)
    rhs<<<nBlocksRHS, nThreads>>>(d_kstar, d_k4, d_f, d_r, d_x, a, dt, dx, K, Np);
    hipDeviceSynchronize();

    checkCudaError("error after rk4");

    rk4<<<nBlocksRK, nThreads>>>(d_u, d_k1, d_k2, d_k3, d_k4, Np, K);

    hipMemcpy(u, d_u, size * sizeof(float), hipMemcpyDeviceToHost);
}

/* allocate memory on the GPU
 */
void initGPU(int K, int Np) {
    int size = K * Np;
    hipDeviceReset();
    checkCudaError("error after reset?");

    // Main variables
    hipMalloc((void **) &d_u , size * sizeof(float));
    hipMalloc((void **) &d_f,  (K + 1) * sizeof(float));
    hipMalloc((void **) &d_rx, size * sizeof(float));
    hipMalloc((void **) &d_mesh, K * sizeof(float));
    hipMalloc((void **) &d_x, size * sizeof(float));
    hipMalloc((void **) &d_r, Np * sizeof(float));
    hipMalloc((void **) &d_w, Np * sizeof(float));

    // Runge-Kutta storage
    hipMalloc((void **) &d_kstar , size * sizeof(float));
    hipMalloc((void **) &d_k1 , size * sizeof(float));
    hipMalloc((void **) &d_k2 , size * sizeof(float));
    hipMalloc((void **) &d_k3 , size * sizeof(float));
    hipMalloc((void **) &d_k4 , size * sizeof(float));

    checkCudaError("error in init");
}

int main() {
    int i, size, t, timesteps;
    float *u;     // the computed result
    float *r;     // the GLL points
    float *w;     // Gaussian integration weights
    
    int Np  = 2;              // polynomial order of the approximation
    int K   = 2*40;           // the mesh size
    float a = 0;              // left boundary
    float b = 2*3.14159;      // right boundary
    float dx = (b - a) / K;    // size of cell
    float aspeed = 2*3.14159; // the wave speed

    float CFL = .75;  // CFL number (duh)
    float dt = 0.5* (CFL/aspeed * dx); // timestep
    timesteps = 1000; 

    size = Np * K;  // size of u

    u = (float *) malloc(K * Np * sizeof(float));
    r = (float *) malloc(Np * sizeof(float));
    w = (float *) malloc(Np * sizeof(float));

    int nThreads    = 128;
    int nBlocksMesh = (K + 1) / nThreads + (((K + 1) % nThreads) ? 1 : 0);
    int nBlocksU    = K / nThreads + ((size % nThreads) ? 1 : 0);

    // Allocate space on the GPU
    initGPU(K, Np);

    // Init the mesh's endpoints
    initMesh<<<nBlocksMesh, nThreads>>>(d_mesh, d_x, dx, a, K);
    hipDeviceSynchronize();

    // Copy over r and w
    r[0] = -1/sqrt(3);
    r[1] =  1/sqrt(3);
    w[0] = 1;
    w[1] = 1;
    hipMemcpy(d_r, r, Np * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w, w, Np * sizeof(float), hipMemcpyHostToDevice);

    // Init the mesh
    initX<<<nBlocksMesh, nThreads>>>(d_mesh, d_x, d_r, dx, K, Np);
    hipDeviceSynchronize();

    // Initialize u0
    initU<<<nBlocksU, nThreads>>>(d_u, d_x, d_w, d_r, K, Np);
    hipDeviceSynchronize();

    checkCudaError("error after initialization");
    // File for output
    FILE *data;
    data = fopen("data.txt", "w");

    hipMemcpy(u, d_u, size * sizeof(float), hipMemcpyDeviceToHost);

    // Run the integrator 
    for (t = 0; t < timesteps; t++) {
        for (i = 0; i < size; i++) {
            fprintf(data," %f ", u[i]);
        }
        fprintf(data, "\n");
        timeIntegrate(u, aspeed, K, dt, dx, dt*t, Np);
    }
    fclose(data);

    // Free host data
    free(u);
    free(r);

    // Free GPU data
    hipFree(d_u);
    hipFree(d_f);
    hipFree(d_rx);
    hipFree(d_mesh);
    hipFree(d_x);
    hipFree(d_r);

    hipFree(d_kstar);
    hipFree(d_k1);
    hipFree(d_k2);
    hipFree(d_k3);
    hipFree(d_k4);
}
